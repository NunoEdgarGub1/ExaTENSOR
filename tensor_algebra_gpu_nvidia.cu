#include "hip/hip_runtime.h"
/** Tensor Algebra Library for NVidia GPU: NV-TAL (CUDA based).
REVISION: 2015/11/02
Copyright (C) 2015 Dmitry I. Lyakh (email: quant4me@gmail.com)
Copyright (C) 2015 Oak Ridge National Laboratory (UT-Battelle)

This source file is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
-------------------------------------------------------------------------------
OPTIONS:
 # -D CUDA_ARCH=350: target GPU compute capability (default is 130);
 # -D NO_GPU: disables GPU usage;
 # -D NO_BLAS: disables cuBLAS calls, they will be replaced by in-house routines (slower);
 # -D DEBUG_GPU: collection of debugging information will be activated;
NOTES:
 # Minimal required compute capability is 1.1 (1.3 for double precision);
 # cuBLAS.v2 is required when BLAS is enabled;
 # Functions without underscores at the end of their names are blocking functions;
   Functions with one underscore at the end of their names are non-blocking functions;
   Functions with two underscores at the end of their names are (non-blocking) CUDA kernels.
 # Non-blocking tensor algebra functions carry an additional input argument <coherence_ctrl>
   which controls the tensor argument synchronization accross different devices.
 # Non-blocking tensor algebra functions carry an additional output argument <cuda_task> (task handle).
 # Seems like hipEventRecord() issued in different streams can serialize the stream
   execution for some older compute capabilities. EVENT_RECORD=0 will disable event recording.
   If GPU timing is needed, event recording has to be enabled (EVENT_RECORD=1).
FOR DEVELOPERS ONLY:
 # Current device resources:
    - Global memory pointer (any device);
    - Argument buffer entry handle (any device);
    - Multi-index-entry pointer (any device, entry length = MAX_TENSOR_RANK);
    - Constant-memory-entry handle (Nvidia GPU);
    - CUDA stream handle (Nvidia GPU);
    - CUDA event handle (Nvidia GPU).
 # A life cycle of a C object (for example, tensBlck_t):
    a) Allocate memory for the object, if needed: Suffix _alloc or _create (includes cleaning);
    b) Clean (initialize to null) an allocated (empty) object: Suffix _clean (normally included in _create);
    c) Construct (define or redefine) an existing object (resources will be acquired/released): Suffix _construct;
    d) Destruct a defined object (resources will be released, the object will be initialized to null): Suffix _destruct;
    e) Free the memory occupied by a clean (initialized to null or destructed) object: Suffix _free or _destroy (may include _destruct).
   Thus, as a rule, the device resource acquisition/release occurs solely in _construct and _destruct functions.
 # A state of a C object:
    a) Undefined: After the memory allocation (either dynamic or static);
    b) Defined nullified: After cleaning or destruction;
    c) Defined to a value: After construction;
    d) Dead: After memory deallocation (if it was allocated dynamically).
 # Resource acquisition/release:
    - Tensor block constructor/destructor acquires/releases global memory resources, including
      both pointers and buffer handles, as well as multi-index bank entries (pinned Host memory).
    - Tensor operation scheduling functions acquire GPU global memory resources,
      GPU constant memory resources, multi-index entries, CUDA stream and event handles (Nvidia GPU).
    - CUDA task completion/error check functions release GPU global memory resources,
      GPU constant memory resources, multi-index entries, CUDA stream and event handles (Nvidia GPU).
 # Functions which construct tensor blocks or perform asynchronous operations on them
   allocate resources (global/constant memory, etc). In case the corresponding resource
   allocator returns TRY_LATER or DEVICE_UNABLE (or an error), the corresponding function
   must clean the partially created tensor block or the CUDA task before returning:
   Thus, the corresponding object will be kept in its initial state if no SUCCESS.
TO BE FIXED:
 # The pinned multi-index slab is only initialized when NVidia GPU is enabled (in <init_gpus>).
   Consequently, I should probably move this initialization into <talsh_init> instead.
**/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#ifndef NO_GPU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef NO_BLAS
#include <hipblas.h>
#endif

#endif

#include "tensor_algebra.h"

#ifndef NO_GPU
//PARAMETERS:
#define GPU_DEBUG_DUMP_SIZE 128 //size of the GPU debug dump (int array)
#endif
//----------------------------------------------------------------------
//FUNCTION PROTOTYPES:
// IMPORTED:
#ifdef __cplusplus
extern "C" {
#endif
 void get_contr_permutations(int lrank, int rrank, const int *cptrn, int *dprm, int *lprm, int *rprm,
                             int *ncd, int *nlu, int *nru, int *ierr);
#ifdef __cplusplus
}
#endif
// LOCAL (PRIVATE):
static int tens_valid_data_kind(int datk);
static int prmn_convert(int n, const int *o2n, int *n2o);
static int non_trivial_prmn(int n, const int *prm);
static int tensDevRsc_create(talsh_dev_rsc_t **drsc);
static int tensDevRsc_clean(talsh_dev_rsc_t * drsc);
static int tensDevRsc_empty(talsh_dev_rsc_t * drsc);
static int tensDevRsc_attach_mem(talsh_dev_rsc_t * drsc, int dev_id, void * mem_p, int buf_entry);
static int tensDevRsc_detach_mem(talsh_dev_rsc_t * drsc);
static int tensDevRsc_allocate_mem(talsh_dev_rsc_t * drsc, int dev_id, size_t mem_size, int in_arg_buf);
static int tensDevRsc_free_mem(talsh_dev_rsc_t * drsc);
#ifndef NO_GPU
static int tensDevRsc_get_const_entry(talsh_dev_rsc_t * drsc, int dev_id);
static int tensDevRsc_release_const_entry(talsh_dev_rsc_t * drsc);
#endif
static int tensDevRsc_release_all(talsh_dev_rsc_t * drsc);
static int tensDevRsc_destroy(talsh_dev_rsc_t * drsc);
static int mi_entry_get(int ** mi_entry);
static int mi_entry_release(int * mi_entry);
static int mi_entry_pinned(int * mi_entry);
#ifndef NO_GPU
static int cuda_stream_get(int gpu_num, int * cuda_stream_handle);
static int cuda_stream_release(int gpu_num, int cuda_stream_handle);
static int cuda_event_get(int gpu_num, int * cuda_event_handle);
static int cuda_event_release(int gpu_num, int cuda_event_handle);
static void limit_cuda_blocks2d(int max_blocks, int *bx, int *by);
static int cuda_task_finalize(cudaTask_t *cuda_task, int err_code, int gpu_num);
static int cuda_task_record(cudaTask_t *cuda_task, int err_code, int gpu_num, hipStream_t hip_stream,
            hipEvent_t cuda_start, hipEvent_t cuda_comput, hipEvent_t cuda_output, hipEvent_t cuda_finish,
            int scr_entry_cnt, int *scr_entries);
// CUDA KERNELS:
__global__ void gpu_array_2norm2_r4__(size_t arr_size, const float *arr, float *bnorm2);
__global__ void gpu_array_2norm2_r8__(size_t arr_size, const double *arr, double *bnorm2);
__global__ void gpu_array_init_r4__(size_t tsize, float *arr, float val);
__global__ void gpu_array_init_r8__(size_t tsize, double *arr, double val);
__global__ void gpu_array_scale_r4__(size_t tsize, float *arr, float val);
__global__ void gpu_array_scale_r8__(size_t tsize, double *arr, double val);
__global__ void gpu_array_add_r4__(size_t tsize, float* __restrict__ arr0, const float* __restrict__ arr1, float val);
__global__ void gpu_array_add_r8__(size_t tsize, double* __restrict__ arr0, const double* __restrict__ arr1, double val);
__global__ void gpu_array_dot_product_r4__(size_t tsize, const float *arr1, const float *arr2, volatile float *dprod);
__global__ void gpu_array_dot_product_r8__(size_t tsize, const double *arr1, const double *arr2, volatile double *dprod);
__global__ void gpu_array_product_r4__(size_t tsize1, const float* __restrict__ arr1, size_t tsize2,
                                       const float* __restrict__ arr2, float* __restrict__ arr0);
__global__ void gpu_array_product_r8__(size_t tsize1, const double* __restrict__ arr1, size_t tsize2,
                                       const double* __restrict__ arr2, double* __restrict__ arr0);
__global__ void gpu_tensor_block_copy_dlf_r4__(int dmo, int drc, int dim_num, int const_args_pos,
                                               const float* __restrict__ tens_in, float* __restrict__ tens_out);
__global__ void gpu_tensor_block_copy_dlf_r8__(int dmo, int drc, int dim_num, int const_args_pos,
                                               const double* __restrict__ tens_in, double* __restrict__ tens_out);
__global__ void gpu_tensor_block_copy_scatter_dlf_r4__(int dmo, int drc, int dim_num, int const_args_pos,
                                                       const float* __restrict__ tens_in, float* __restrict__ tens_out);
__global__ void gpu_tensor_block_copy_scatter_dlf_r8__(int dmo, int drc, int dim_num, int const_args_pos,
                                                       const double* __restrict__ tens_in, double* __restrict__ tens_out);
__global__ void gpu_matrix_multiply_tn_r4__(size_t ll, size_t lr, size_t lc, const float* __restrict__ arg1,
                                            const float* __restrict__ arg2, float* __restrict__ arg0);
__global__ void gpu_matrix_multiply_tn_r8__(size_t ll, size_t lr, size_t lc, const double* __restrict__ arg1,
                                            const double* __restrict__ arg2, double* __restrict__ arg0);
#endif
//------------------------------------------------------------------------------------------------------
//PARAMETERS:
static int VERBOSE=1; //verbosity for error messages
#ifndef NO_GPU
//GLOBAL DATA:
// GPU control on the current MPI process:
static int gpu_up[MAX_GPUS_PER_NODE]; //GPU_OFF(0): GPU is disabled; GPU_MINE(1): GPU is enabled; GPU_MINE_CUBLAS(2): GPU is BLAS enabled
static hipDeviceProp_t gpu_prop[MAX_GPUS_PER_NODE]; //properties of all GPUs present on the node
static talsh_stats_t gpu_stats[MAX_GPUS_PER_NODE]; //runtime statistics for all GPUs present on the node
#ifndef NO_BLAS
// Infrastructure for CUBLAS:
static hipblasHandle_t cublas_handle[MAX_GPUS_PER_NODE]; //each GPU present on a node obtains its own cuBLAS context handle
#endif
#endif
// Slab for the multi-index storage (will be pinned):
static int miBank[MAX_GPU_ARGS*MAX_MLNDS_PER_TENS][MAX_TENSOR_RANK]; //All active .dims[], .divs[], .grps[] will be stored here
static int miFreeHandle[MAX_GPU_ARGS*MAX_MLNDS_PER_TENS]; //free entries for storing multi-indices
static int miFFE; //number of free handles left in miBank

#ifndef NO_GPU
// Slabs for the GPU asynchronous resources:
//  CUDA stream handles:
static hipStream_t CUDAStreamBank[MAX_GPUS_PER_NODE][MAX_CUDA_TASKS]; //pre-allocated CUDA stream handles (for each CUDA device)
static int CUDAStreamFreeHandle[MAX_GPUS_PER_NODE][MAX_CUDA_TASKS]; //free CUDA stream handles
static int CUDAStreamFFE[MAX_GPUS_PER_NODE]; //number of free handles left in CUDAStreamFreeHandle
//  CUDA event handles:
static hipEvent_t CUDAEventBank[MAX_GPUS_PER_NODE][MAX_CUDA_EVENTS]; //pre-allocated CUDA event handles (for each CUDA device)
static int CUDAEventFreeHandle[MAX_GPUS_PER_NODE][MAX_CUDA_EVENTS]; //free CUDA event handles
static int CUDAEventFFE[MAX_GPUS_PER_NODE]; //number of free handles left in CUDAEventFreeHandle

// Slab of GPU constant memory arguments for each GPU (managed by "mem_manager.cu"):
__device__ __constant__ int const_args_dims[MAX_GPU_ARGS][MAX_TENSOR_RANK]; //storage for device constant memory arguments: dimension extents
__device__ __constant__ int const_args_prmn[MAX_GPU_ARGS][MAX_TENSOR_RANK]; //storage for device constant memory arguments: permutation

// GPU error control and debugging for each GPU:
__device__ int gpu_error_count=0; //total number of CUDA errors registered on device till the current moment
__device__ int gpu_debug_dump[GPU_DEBUG_DUMP_SIZE]; //debug dump

// Global CUDA event recording policy:
static int EVENT_RECORD=1; //non-zero value enables CUDA event recording
static int PRINT_TIMING=1; //non-zero value enables time printing statements
// Infrastructure for function <gpu_tensor_block_copy_dlf> (blocking and non-blocking):
static int TRANS_SHMEM=1; //switch between shared-memory tensor transpose (1) and scatter tensor transpose (0)
// Infrastructure for <gpu_tensor_block_contract_dlf_> (non-blocking):
#ifndef NO_BLAS
static int DISABLE_BLAS=0; //non-zero value will disable cuBLAS usage (if it had been cuBLAS compiled/linked)
#else
static int DISABLE_BLAS=1; //non-zero value will disable cuBLAS usage (if it had been cuBLAS compiled/linked)
#endif
static cudaTask_t * LastTask[MAX_GPUS_PER_NODE]; //last CUDA task successfully scheduled on each GPU
__device__ __constant__ float sgemm_alpha=1.0f; //default alpha constant for SGEMM
__device__ __constant__ float sgemm_beta=1.0f;  //default beta constant SGEMM
__device__ __constant__ double dgemm_alpha=1.0; //default alpha constant for DGEMM
__device__ __constant__ double dgemm_beta=1.0;  //default beta constant DGEMM
// Infrastructure for functions <gpu_array_norm2_XX>:
__device__ int norm2_wr_lock=0; //write lock (shared by all <gpu_array_norm2_XX> running on device)
// Infrastructure for kernels <gpu_array_dot_product_XX__>:
__device__ int dot_product_wr_lock=0; //write lock (shared by all <gpu_array_dot_product_XX__> running on device)

// Infrastructure for functions <gpu_array_2norm2_XX> (blocking)`Obsolete:
__device__ float gpu_blck_norms2_r4[MAX_CUDA_BLOCKS]; //`Obsolete
__device__ double gpu_blck_norms2_r8[MAX_CUDA_BLOCKS]; //`Obsolete
static float blck_norms2_r4[MAX_CUDA_BLOCKS];  //`Obsolete `Not multi-GPU safe
static double blck_norms2_r8[MAX_CUDA_BLOCKS]; //`Obsolete `Not multi-GPU safe
#endif
//-------------------------------------------------------------------------------------
//DEVICE ID CONVERSION:
int encode_device_id(int dev_kind, int dev_num)
/** Given a device ID <dev_num> within its kind <dev_kind>, returns the flat device ID.
    DEV_MAX value on return means that the arguments were invalid. **/
{
 int dev_id=DEV_MAX; //Return of this value (= outside devices range) will mean that the arguments were invalid
 switch(dev_kind){
  case DEV_HOST: if(dev_num == 0) dev_id=0; break;
  case DEV_NVIDIA_GPU: if(dev_num >= 0 && dev_num < MAX_GPUS_PER_NODE) dev_id=1+dev_num; break;
  case DEV_INTEL_MIC: if(dev_num >= 0 && dev_num < MAX_MICS_PER_NODE) dev_id=1+MAX_GPUS_PER_NODE+dev_num; break;
  case DEV_AMD_GPU: if(dev_num >= 0 && dev_num < MAX_AMDS_PER_NODE) dev_id=1+MAX_GPUS_PER_NODE+MAX_MICS_PER_NODE+dev_num; break;
  default: dev_id=DEV_MAX; //unknown device kind
 }
 return dev_id;
}

int decode_device_id(int dev_id, int *dev_kind)
/** Given a flat device ID <dev_id>, returns the device kind <dev_kind>
    and its kind-specific ID (>=0) as the return value.
    A negative return status means an invalid <dev_id> was passed. **/
{
 int dvn=-1; //Negative return value will correspond to an invalid <dev_id>
 int dvid=abs(dev_id); //flat device id is defined up to a sign
 if(dvid == 0){ //Host
  *dev_kind=DEV_HOST; dvn=0;
 }else if(dvid >= 1 && dvid <= MAX_GPUS_PER_NODE){ //Nvidia GPU
  *dev_kind=DEV_NVIDIA_GPU; dvn=dvid-1;
 }else if(dvid >= 1+MAX_GPUS_PER_NODE && dvid <= MAX_GPUS_PER_NODE+MAX_MICS_PER_NODE){ //Intel MIC
  *dev_kind=DEV_INTEL_MIC; dvn=dvid-1-MAX_GPUS_PER_NODE;
 }else if(dvid >= 1+MAX_GPUS_PER_NODE+MAX_MICS_PER_NODE && dvid <= MAX_GPUS_PER_NODE+MAX_MICS_PER_NODE+MAX_AMDS_PER_NODE){ //AMD GPU
  *dev_kind=DEV_AMD_GPU; dvn=dvid-1-MAX_GPUS_PER_NODE-MAX_MICS_PER_NODE;
 }
 return dvn; //ID of the device within its kind
}

#ifndef NO_GPU
//GPU DEBUG FUNCTIONS:
__host__ int gpu_get_error_count()
/** Returns the total number of CUDA errors occured on current GPU.
    A negative return status means an error occurred. **/
{
 int i;
 hipError_t err=hipMemcpyFromSymbol((void*)&i,gpu_error_count,sizeof(gpu_error_count),0,hipMemcpyDeviceToHost);
 if(err == hipSuccess){return i;}else{return -1;}
}

__host__ int gpu_get_debug_dump(int *dump)
/** Returns the debug dump (int array) from current GPU.
    A positive return status is the length of the debug dump.
    A negative return status means an error occurred. **/
{
 hipError_t err=hipMemcpyFromSymbol((void*)dump,gpu_debug_dump,sizeof(int)*GPU_DEBUG_DUMP_SIZE,0,hipMemcpyDeviceToHost);
 if(err == hipSuccess){return GPU_DEBUG_DUMP_SIZE;}else{return -1;}
}
#endif

//AUXILIARY FUNCTIONS:
static int tens_valid_data_kind(int datk)
/** Returns YEP if the data kind <datk> is valid in TAL-SH, NOPE otherwise. **/
{
 if(datk == R4 || datk == R8 || datk == C8 || datk == NO_TYPE) return YEP;
 return NOPE;
}

static int prmn_convert(int n, const int *o2n, int *n2o)
/** Converts an O2N permutation into N2O (length = n). Both permutations
    are sign-free and the numeration starts from 1. **/
{
 int i,j;
 if(n >= 0){
  for(i=0;i<n;i++){j=o2n[i]-1; if(j >= 0 && j < n){n2o[j]=i+1;}else{return 1;}}
 }else{
  return 2;
 }
 return 0;
}

static int non_trivial_prmn(int n, const int *prm)
/** Returns 0 if the permutation prm[0:n-1] is trivial, 1 otherwise.
    The permutation is sign-free and the numeration starts from 1. No error check. **/
{
 int i,f=0;
 for(i=0;i<n;i++){if(prm[i] != i+1){f=1; break;}}
 return f;
}

//DEVICE RESOURCE MANAGEMENT:
static int tensDevRsc_create(talsh_dev_rsc_t **drsc)
/** Creates a new device resource descriptor and inits it to null. **/
{
 int errc;
 *drsc=(*talsh_dev_rsc_t)malloc(sizeof(talsh_dev_rsc_t)); if(*drsc == NULL) return 1;
 errc=tensDevRsc_clean(*drsc);
 return 0;
}

static int tensDevRsc_clean(talsh_dev_rsc_t * drsc)
/** Cleans (initializes to null) a device resource for tensBlck_t. **/
{
 if(drsc != NULL){
  drsc->dev_id=DEV_NULL;    //flat device id
  drsc->gmem_p=NULL;        //device global memory pointer (any device)
  drsc->buf_entry=-1;       //device argument buffer entry (any device)
  drsc->const_mem_entry=-1; //constant memory bank entry (Nvidia GPU only)
 }else{
  return -1;
 }
 return 0;
}

static int tensDevRsc_empty(talsh_dev_rsc_t * drsc)
/** Returns YEP if the device resource descriptor is empty, NOPE otherwise.
    Negative return status means an error. **/
{
 if(drsc == NULL) return -1;
 if(drsc->gmem_p != NULL || drsc->buf_entry >= 0 || drsc->const_mem_entry >= 0) return NOPE;
 drsc->dev_id=DEV_NULL;
 return YEP;
}

static int tensDevRsc_attach_mem(talsh_dev_rsc_t * drsc, int dev_id, void * mem_p, int buf_entry = -1)
/** Attaches a chunk of existing global memory to a device resource descriptor.
    If <buf_entry> >= 0, that means that the global memory is in the argument buffer.
    If the resource descriptor had already been assigned a device, the <dev_id>
    argument must match that one. **/
{
 if(drsc == NULL) return -1;
 if(dev_id < 0) return -2;
 if(mem_p == NULL) return -3;
 if(drsc->dev_id >= 0 && drsc->dev_id != dev_id) return 1; //resource was assigned to a different device
 if(drsc->gmem_p != NULL) return 2; //resource already has global memory attached
 drsc->dev_id=dev_id; drsc->gmem_p=mem_p; drsc->buf_entry=buf_entry;
 return 0;
}

static int tensDevRsc_detach_mem(talsh_dev_rsc_t * drsc)
/** Detaches a chunk of external memory from a device resource descriptor.
    Regardless of the origin of that memory, it is not released. **/
{
 int errc;

 if(drsc == NULL) return -1;
 if(drsc->dev_id < 0) return -2; //empty resource descriptor
 if(drsc->gmem_p == NULL) return 1; //no global memory attached
 drsc->gmem_p=NULL; drsc->buf_entry=-1;
 errc=tensDevRsc_empty(drsc);
 return 0;
}

static int tensDevRsc_allocate_mem(talsh_dev_rsc_t * drsc, int dev_id, size_t mem_size, int in_arg_buf = NOPE)
/** Allocates global memory on device <dev_id> and attaches it to a device resource descriptor.
    If <in_arg_buf> = YEP, the memory will be allocated via that device's argument buffer.
    A return status TRY_LATER or DEVICE_UNABLE indicates the resource shortage and is not an error. **/
{
 int i,devk,devn,errc;
 char *byte_ptr;

 if(drsc == NULL) return -1;
 if(dev_id < 0) return -2;
 if(mem_size <= 0) return -3;
 devn=decode_device_id(dev_id,&devk);
 if(devn < 0) return -4; //invalid flat device id
 if(drsc->dev_id >= 0 && drsc->dev_id != dev_id) return 1; //resource was assigned to a different device
 if(drsc->gmem_p != NULL) return 2; //resource already has global memory attached
 switch(devk){
  case DEV_HOST:
   if(in_arg_buf == NOPE){
    errc=host_mem_alloc_pin(&(drsc->gmem_p),mem_size); if(errc != 0){drsc->gmem_p = NULL; return 3;}
   }else{
    errc=get_buf_entry_host(mem_size,&byte_ptr,&i);
    if(errc != 0){if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 4;}}
    drsc->gmem_p=(void*)byte_ptr; drsc->buf_entry=i;
   }
   break;
  case DEV_NVIDIA_GPU:
#ifndef NO_GPU
   if(in_arg_buf == NOPE){
    errc=gpu_mem_alloc(&(drsc->gmem_p),mem_size,devn); if(errc != 0){drsc->gmem_p = NULL; return 5;}
   }else{
    errc=get_buf_entry_gpu(devn,mem_size,&byte_ptr,&i);
    if(errc != 0){if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 6;}}
    drsc->gmem_p=(void*)byte_ptr; drsc->buf_entry=i;
   }
   break;
#else
   return -5;
#endif
  case DEV_INTEL_MIC:
#ifndef NO_MIC
   //`Future
   break;
#else
   return -6;
#endif
  case DEV_AMD_GPU:
#ifndef NO_AMD
   //`Future
   break;
#else
   return -7;
#endif
  default:
   return -8; //unknown device kind
 }
 drsc->dev_id=dev_id;
 return 0;
}

static int tensDevRsc_free_mem(talsh_dev_rsc_t * drsc)
/** Releases global memory referred to by a device resource descriptor.
    An unsuccessful release of the global memory is marked with
    an error status NOT_CLEAN, but the corresponding components of
    the resource descriptor are cleared anyway. **/
{
 int n,devn,devk,errc;

 n=0;
 if(drsc == NULL) return -1;
 if(drsc->dev_id < 0) return -2;
 if(drsc->gmem_p == NULL) return -3;
 devn=decode_device_id(drsc->dev_id,&devk);
 if(devn < 0) return -4; //invalid flat device id
 switch(devk){
  case DEV_HOST:
   if(drsc->buf_entry >= 0){
    errc=free_buf_entry_host(drsc->buf_entry); if(errc != 0) n=NOT_CLEAN;
    drsc->gmem_p=NULL; drsc->buf_entry=-1;
   }else{
    if(drsc->gmem_p != NULL){
     errc=host_mem_free_pin(drsc->gmem_p); if(errc != 0) n=NOT_CLEAN;
     drsc->gmem_p=NULL;
    }
   }
   break;
  case DEV_NVIDIA_GPU:
#ifndef NO_GPU
   if(drsc->buf_entry >= 0){
    errc=free_buf_entry_gpu(devn,drsc->buf_entry); if(errc != 0) n=NOT_CLEAN;
    drsc->gmem_p=NULL; drsc->buf_entry=-1;
   }else{
    if(drsc->gmem_p != NULL){
     errc=gpu_mem_free(drsc->gmem_p,devn); if(errc != 0) n=NOT_CLEAN;
     drsc->gmem_p=NULL;
    }
   }
   break;
#else
   return -5;
#endif
  case DEV_INTEL_MIC:
#ifndef NO_MIC
   //`Future
   break;
#else
   return -6;
#endif
  case DEV_AMD_GPU:
#ifndef NO_AMD
   //`Future
   break;
#else
   return -7;
#endif
  default:
   return -8; //invalid device kind
 }
 errc=tensDevRsc_empty(drsc);
 return n;
}

#ifndef NO_GPU
static int tensDevRsc_get_const_entry(talsh_dev_rsc_t * drsc, int dev_id)
/** Acquires a constant memory entry on an Nvidia GPU. A return status TRY_LATER or
    DEVICE_UNABLE indicate a temporary or permanent shortage of the corresponding resource. **/
{
 int i,devn,devk,errc;

 if(drsc == NULL) return -1;
 if(dev_id < 0) return -2;
 devn=decode_device_id(dev_id,&devk);
 if(devn < 0) return -3; //invalid flat device id
 if(drsc->dev_id >= 0 && drsc->dev_id != dev_id) return 1; //resource was assigned to a different device
 if(drsc->const_mem_entry >= 0) return 2; //resource already has constant memory entry attached
 if(devk == DEV_NVIDIA_GPU){
  errc=const_args_entry_get(devn,&i);
  if(errc != 0){if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 3;}}
  drsc->const_mem_entry=i;
 }else{
  return 4;
 }
 drsc->dev_id=dev_id;
 return 0;
}

static int tensDevRsc_release_const_entry(talsh_dev_rsc_t * drsc)
/** Releases a GPU constant memory entry from a given device resource descriptor.
    An unsuccessful release of the GPU constant memory entry is marked with
    an error status NOT_CLEAN, but the corresponding component of the resource
    descriptor is cleared anyway. **/
{
 int n,devn,devk,errc;

 n=0;
 if(drsc == NULL) return -1;
 if(drsc->dev_id < 0) return -2;
 devn=decode_device_id(drsc->dev_id,&devk);
 if(devn < 0) return -3; //invalid flat device id
 if(drsc->const_mem_entry < 0) return 1;
 if(devk == DEV_NVIDIA_GPU){
  errc=const_args_entry_free(devn,drsc->const_mem_entry); if(errc != 0) n=NOT_CLEAN;
  drsc->const_mem_entry=-1;
 }else{
  return 2;
 }
 errc=tensDevRsc_empty(drsc);
 return n;
}
#endif

static int tensDevRsc_release_all(talsh_dev_rsc_t * drsc)
/** Releases all device resources in <drsc>. An unsuccessful release
    of one or more resources is marked with an error status NOT_CLEAN,
    but the corresponding components of the device resource descriptor
    are cleared anyway. **/
{
 int n,devk,devn,errc;

 n=0;
 if(drsc == NULL) return -1;
 if(drsc->dev_id >= 0){ //resource handle is not empty
  devn=decode_device_id(drsc->dev_id,&devk);
  if(devn < 0) return -2; //invalid flat device id
  switch(devk){
   case DEV_HOST:
    if(drsc->buf_entry >= 0){
     errc=free_buf_entry_host(drsc->buf_entry); if(errc != 0) n=NOT_CLEAN;
     drsc->gmem_p=NULL; drsc->buf_entry=-1;
    }else{
     if(drsc->gmem_p != NULL){
      errc=host_mem_free_pin(drsc->gmem_p); i(errc != 0) n=NOT_CLEAN;
      drsc->gmem_p=NULL;
     }
    }
    break;
   case DEV_NVIDIA_GPU:
#ifndef NO_GPU
    if(drsc->buf_entry >= 0){
     errc=free_buf_entry_gpu(devn,drsc->buf_entry); if(errc != 0) n=NOT_CLEAN;
     drsc->gmem_p=NULL; drsc->buf_entry=-1;
    }else{
     if(drsc->gmem_p != NULL){
      errc=gpu_mem_free(drsc->gmem_p,devn); if(errc != 0) n=NOT_CLEAN;
      drsc->gmem_p=NULL;
     }
    }
    if(drsc->const_mem_entry >= 0){
     errc=const_args_entry_free(devn,drsc->const_mem_entry); if(errc != 0) n=NOT_CLEAN;
     drsc->const_mem_entry=-1;
    }
    break;
#else
    return -3;
#endif
   case DEV_INTEL_MIC:
#ifndef NO_MIC
    //`Future
    break;
#else
    return -4;
#endif
   case DEV_AMD_GPU:
#ifndef NO_AMD
    //`Future
    break;
#else
    return -5;
#endif
   default:
    return -6; //invalid device kind
  }
  errc=tensDevRsc_clean(drsc);
 }
 return n;
}

static int tensDevRsc_destroy(talsh_dev_rsc_t * drsc)
/** Completely destroys a device resource descriptor. A return status NOT_CLEAN
    means that certain resources have not been released cleanly,
    but it is not a critical error in general (however, a leak can occur). **/
{
 int n,errc;

 n=0;
 if(drsc == NULL) return -1;
 errc=tensDevRsc_release_all(drsc); if(errc != 0) n=NOT_CLEAN;
 free(drsc);
 return n;
}

static int mi_entry_get(int ** mi_entry)
/** Obtains a pointer to an entry in the multi-index storage slab.
    The entry can fit an <int> multi-index up to MAX_TENSOR_RANK length.
    Returns TRY_LATER if no free handles are currently available. **/
{
 int m;
 *mi_entry=NULL;
 if(miFFE > 0){ //number of free handles left
  m=miFreeHandle[--miFFE];
  *mi_entry=&miBank[m][0];
 }else{
  return TRY_LATER; //currently no free handles left
 }
 return 0;
}

static int mi_entry_release(int * mi_entry)
/** Releases an entry back to the multi-index storage slab. **/
{
 int m;
 if(mi_entry != NULL){
  if(miFFE >= 0){
   m=(int)(mi_entry-&miBank[0][0]);
   if(m%MAX_TENSOR_RANK == 0){
    m/=MAX_TENSOR_RANK;
    miFreeHandle[miFFE++]=m;
   }else{
    return 1;
   }
  }else{
   return 2;
  }
 }else{
  return 3;
 }
 return 0;
}

static int mi_entry_pinned(int * mi_entry)
/** Returns YEP if the multi-index is in the multi-index bank,
    NOPE othewise. **/
{
 int n;

 n=NOPE;
 if(mi_entry != NULL){
  if((unsigned int)(mi_entry-miBank[0][0]) < MAX_GPU_ARGS*MAX_MLNDS_PER_TENS*MAX_TENSOR_RANK) n=YEP;
 }
 return n;
}

#ifndef NO_GPU
__host__ static int cuda_stream_get(int gpu_num, int * cuda_stream_handle)
/** For GPU#gpu_num, returns a usable CUDA stream handle <cuda_stream_handle>.
Non-zero return status means an error, except the return status TRY_LATER means
no free resources are currently available (not an error). **/
{
 *cuda_stream_handle=-1;
 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  if(gpu_up[gpu_num] >= GPU_MINE){
   if(CUDAStreamFFE[gpu_num] > 0){ //number of free handles left on GPU#gpu_num
    *cuda_stream_handle=CUDAStreamFreeHandle[gpu_num][--CUDAStreamFFE[gpu_num]];
    if(*cuda_stream_handle < 0 || *cuda_stream_handle >= MAX_CUDA_TASKS){
     *cuda_stream_handle=-1; return 3; //invalid handle: corruption
    }
   }else{
    return TRY_LATER; //all handles are currently busy
   }
  }else{
   return 2;
  }
 }else{
  return 1;
 }
 return 0;
}

__host__ static int cuda_stream_release(int gpu_num, int cuda_stream_handle)
/** For GPU#gpu_num, releases a CUDA stream handle <cuda_stream_handle>.
Non-zero return status means an error. **/
{
 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  if(gpu_up[gpu_num] >= GPU_MINE){
   if(cuda_stream_handle >= 0 && cuda_stream_handle < MAX_CUDA_TASKS){
    if(CUDAStreamFFE[gpu_num] < 0 || CUDAStreamFFE[gpu_num] > MAX_CUDA_TASKS) return 5; //corrupted
    if(CUDAStreamFFE[gpu_num] < MAX_CUDA_TASKS){
     CUDAStreamFreeHandle[gpu_num][CUDAStreamFFE[gpu_num]++]=cuda_stream_handle;
    }else{
     return 4; //an attempt to release a non-existing handle
    }
   }else{
    return 3;
   }
  }else{
   return 2;
  }
 }else{
  return 1;
 }
 return 0;
}

__host__ static int cuda_event_get(int gpu_num, int * cuda_event_handle)
/** For GPU#gpu_num, returns a usable CUDA event handle <cuda_event_handle>.
Non-zero return status means an error, except the return status TRY_LATER means
no free resources are currently available (not an error). **/
{
 *cuda_event_handle=-1;
 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  if(gpu_up[gpu_num] >= GPU_MINE){
   if(CUDAEventFFE[gpu_num] > 0){ //number of free handles left on GPU#gpu_num
    *cuda_event_handle=CUDAEventFreeHandle[gpu_num][--CUDAEventFFE[gpu_num]];
    if(*cuda_event_handle < 0 || *cuda_event_handle >= MAX_CUDA_EVENTS){
     *cuda_event_handle=-1; return 3; //invalid handle: corruption
    }
   }else{
    return TRY_LATER; //all handles are currently busy
   }
  }else{
   return 2;
  }
 }else{
  return 1;
 }
 return 0;
}

__host__ static int cuda_event_release(int gpu_num, int cuda_event_handle)
/** For GPU#gpu_num, releases a CUDA event handle <cuda_event_handle>.
Non-zero return status means an error. **/
{
 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  if(gpu_up[gpu_num] >= GPU_MINE){
   if(cuda_event_handle >= 0 && cuda_event_handle < MAX_CUDA_EVENTS){
    if(CUDAEventFFE[gpu_num] < 0 || CUDAEventFFE[gpu_num] > MAX_CUDA_EVENTS) return 5; //corrupted
    if(CUDAEventFFE[gpu_num] < MAX_CUDA_EVENTS){
     CUDAEventFreeHandle[gpu_num][CUDAEventFFE[gpu_num]++]=cuda_event_handle;
    }else{
     return 4; //an attempt to release a non-existing handle
    }
   }else{
    return 3;
   }
  }else{
   return 2;
  }
 }else{
  return 1;
 }
 return 0;
}

__host__ static void limit_cuda_blocks2d(int max_blocks, int *bx, int *by)
/** Limits the number of CUDA blocks in a 2d grid to <max_blocks>.
    No argument validity check! **/
{
 if(max_blocks > 1){
  double rdc = ((double)max_blocks)/(((double)(*bx))*((double)(*by)));
  if(rdc < 1.0){
   rdc=sqrt(rdc);
   if(*bx > *by){
    *by=(int)(rdc*((double)(*by))); if(*by < 1){*by=1; *bx=max_blocks; return;}
    *bx=(int)(rdc*((double)(*bx)));
   }else{
    *bx=(int)(rdc*((double)(*bx))); if(*bx < 1){*bx=1; *by=max_blocks; return;}
    *by=(int)(rdc*((double)(*by)));
   }
   if((*bx)*(*by) > max_blocks){
    if(*bx > *by){(*bx)--;}else{(*by)--;}
   }
  }
 }else{
  *bx=1; *by=1;
 }
 return;
}

//NV-TAL INITIALIZATION/SHUTDOWN (internal use only):
__host__ int init_gpus(int gpu_beg, int gpu_end)
/** Initializes all GPU contexts for the current MPI process. Returned positive value is
the number of initialized GPUs. A negative return status means an error occured.
Each enabled GPU from the range [gpu_beg:gpu_end] will obtain its own cublasHandle as well.
The first GPU from the given range will be left active at the end. If <gpu_beg> > <gpu_end>,
no GPU will be initialized. **/
{
 size_t m;
 int i,j,n,errc;
 hipError_t err;
#ifndef NO_BLAS
 hipblasStatus_t err_cublas;
#endif
 n=0; for(i=0;i<MAX_GPUS_PER_NODE;i++) gpu_up[i]=GPU_OFF; //initial GPU status
//Multi-index bank:
 miFFE=MAX_GPU_ARGS*MAX_MLNDS_PER_TENS;
 for(j=0;j<miFFE;j++) miFreeHandle[j]=j;
 m=(size_t)(miFFE*MAX_TENSOR_RANK*sizeof(int));
 errc=host_mem_register(&miBank[0][0],m);
 if(errc != 0){
  if(VERBOSE) printf("#ERROR(tensor_algebra_gpu_nvidia:init_gpus): Unable to register the multi-index bank: Error %d\n",errc);
  return -3;
 }
 if(gpu_beg >= 0 && gpu_end >= gpu_beg){
  err=hipGetDeviceCount(&i); if(err != hipSuccess) return -1;
  if(gpu_end >= MAX_GPUS_PER_NODE || gpu_end >= i) return -2;
  for(i=gpu_end;i>=gpu_beg;i--){
   err=hipSetDevice(i);
   if(err == hipSuccess){
    gpu_up[i]=GPU_MINE; err=hipGetDeviceProperties(&(gpu_prop[i]),i); if(err != hipSuccess) gpu_up[i]=GPU_OFF;
    if(gpu_up[i] > GPU_OFF){
//SHMEM width:
     errc=gpu_set_shmem_width(GPU_SHMEM_WIDTH);
     if(errc != 0 && VERBOSE) printf("#WARNING(tensor_algebra_gpu_nvidia:init_gpus): Unable to set GPU SHMEM width %d: Error %d \n",GPU_SHMEM_WIDTH,errc);
//cuBLAS.v2 context:
#ifndef NO_BLAS
     err_cublas=hipblasCreate(&(cublas_handle[i]));
     if(err_cublas == HIPBLAS_STATUS_SUCCESS){
      gpu_up[i]=GPU_MINE_CUBLAS;
      err_cublas=hipblasSetPointerMode(cublas_handle[i],HIPBLAS_POINTER_MODE_DEVICE);
      if(err_cublas != HIPBLAS_STATUS_SUCCESS) gpu_up[i]=GPU_MINE;
     }
#endif
    }
//CUDA stream bank:
    if(gpu_up[i] > GPU_OFF){
     for(j=0;j<MAX_CUDA_TASKS;j++) CUDAStreamFreeHandle[i][j]=j; CUDAStreamFFE[i]=MAX_CUDA_TASKS;
     for(j=0;j<MAX_CUDA_TASKS;j++){
      err=hipStreamCreate(&(CUDAStreamBank[i][j])); if(err != hipSuccess){gpu_up[i]=GPU_OFF; break;};
     }
    }
//CUDA event bank:
    if(gpu_up[i] > GPU_OFF){
     for(j=0;j<MAX_CUDA_EVENTS;j++) CUDAEventFreeHandle[i][j]=j; CUDAEventFFE[i]=MAX_CUDA_EVENTS;
     for(j=0;j<MAX_CUDA_EVENTS;j++){
      err=hipEventCreate(&(CUDAEventBank[i][j])); if(err != hipSuccess){gpu_up[i]=GPU_OFF; break;};
     }
    }
//Last task:
    LastTask[i]=NULL;
//Clear GPU statistics:
    gpu_stats[i].tasks_submitted=0;
    gpu_stats[i].tasks_completed=0;
    gpu_stats[i].tasks_deferred=0;
    gpu_stats[i].tasks_failed=0;
    gpu_stats[i].flops=0.0;
    gpu_stats[i].traffic_in=0.0;
    gpu_stats[i].traffic_out=0.0;
    gpu_stats[i].time_active=0.0;
    gpu_stats[i].time_start=clock();
//Accept GPU as ready (active):
    if(gpu_up[i] > GPU_OFF) n++;
   }
  }
 }
 return n; //number of initialized GPU's
}

__host__ int free_gpus(int gpu_beg, int gpu_end)
/** Destroys all GPU/CUBLAS contexts on all GPU devices belonging to the MPI process.
A positive value returned is the number of failed GPUs; a negative one is an error.
If <gpu_beg> > <gpu_end>, nothing wil be done. **/
{
 int i,j,n,failure;
 hipError_t err;
#ifndef NO_BLAS
 hipblasStatus_t err_cublas;
#endif
 failure=0; n=0;
//Multi-index bank:
 miFFE=MAX_GPU_ARGS*MAX_MLNDS_PER_TENS;
 for(j=0;j<miFFE;j++) miFreeHandle[j]=j;
 j=host_mem_unregister(&miBank[0][0]); //`This is probably not needed
 if(j != 0){
  failure++;
  if(VERBOSE) printf("#WARNING(tensor_algebra_gpu_nvidia:free_gpus): Unable to unregister the multi-index bank: Error %d\n",j);
 }
 if(gpu_beg >= 0 && gpu_end >= gpu_beg){
  err=hipGetDeviceCount(&i); if(err != hipSuccess) return -1;
  if(gpu_end >= MAX_GPUS_PER_NODE || gpu_end >= i) return -2;
  for(i=gpu_beg;i<=gpu_end;i++){
   if(gpu_up[i] > GPU_OFF){
    n++; err=hipSetDevice(i);
    if(err == hipSuccess){
#ifndef NO_BLAS
     if(gpu_up[i] >= GPU_MINE_CUBLAS){err_cublas=hipblasDestroy(cublas_handle[i]); if(err_cublas == HIPBLAS_STATUS_SUCCESS) gpu_up[i]=GPU_MINE;}
#endif
//CUDA stream bank:
     if(gpu_up[i] > GPU_OFF){
      for(j=0;j<MAX_CUDA_TASKS;j++) CUDAStreamFreeHandle[i][j]=j; CUDAStreamFFE[i]=MAX_CUDA_TASKS;
      for(j=0;j<MAX_CUDA_TASKS;j++){err=hipStreamDestroy(CUDAStreamBank[i][j]); if(err != hipSuccess) failure++;}
     }
//CUDA event bank:
     if(gpu_up[i] > GPU_OFF){
      for(j=0;j<MAX_CUDA_EVENTS;j++) CUDAEventFreeHandle[i][j]=j; CUDAEventFFE[i]=MAX_CUDA_EVENTS;
      for(j=0;j<MAX_CUDA_EVENTS;j++){err=hipEventDestroy(CUDAEventBank[i][j]); if(err != hipSuccess) failure++;}
     }
//Last task:
     LastTask[i]=NULL;
     n--; err=hipDeviceReset();
    }
    gpu_up[i]=GPU_OFF; //GPU is taken out of use regardless of its status!
   }
  }
 }
 if(failure && VERBOSE) printf("#WARNING(tensor_algebra_gpu_nvidia:free_gpus): Resource deallocation was not fully successful!");
 return n;
}

__host__ int gpu_is_mine(int gpu_num)
/** Positive return: GPU is mine; 0: GPU is not mine; -1: invalid <gpu_num>. **/
{if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){return gpu_up[gpu_num];}else{return -1;}}

__host__ int gpu_busy_least()
/** Returns the ID of the least busy GPU (non-negative) or -1 (no GPU found). **/
{
 int i,j,m,n;
 m=-1; n=-1;
 for(i=0;i<MAX_GPUS_PER_NODE;i++){
  if(gpu_up[i] != GPU_OFF){
   j=gpu_stats[i].tasks_submitted-(gpu_stats[i].tasks_completed+gpu_stats[i].tasks_deferred+gpu_stats[i].tasks_failed);
   if(m >= 0){
    if(j < m){m=j; n=i;};
   }else{
    m=j; n=i;
   }
  }
 }
 return n;
}

__host__ int gpu_activate(int gpu_num)
/** If GPU is enabled (mine), does hipSetDevice; returns non-zero otherwise (error). **/
{
 hipError_t err;
 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  if(gpu_up[gpu_num] > GPU_OFF){err=hipSetDevice(gpu_num); if(err != hipSuccess) return 3;}else{return 2;}
 }else{
  return 1; //invalid <gpu_num>
 }
 return 0;
}

//NV-TAL INTERNAL CONTROL:
__host__ int gpu_set_shmem_width(int width){
/** Sets the GPU shared memory bank width:
    <width> = R4: 4 bytes;
    <width> = R8: 8 bytes. **/
 hipError_t cerr;
 if(width == R8){
  cerr=hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
 }else if(width == R4){
  cerr=hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
 }else{
  return 1; //invalid <width> passed
 }
 if(cerr != hipSuccess) return 2;
 return 0;
}

__host__ void gpu_set_event_policy(int alg)
/** Turns on/off timing CUDA events (1/0). **/
{if(alg == EVENTS_OFF){EVENT_RECORD=EVENTS_OFF;}else{EVENT_RECORD=EVENTS_ON;}; return;}

__host__ void gpu_set_transpose_algorithm(int alg)
/** Activates either the scatter or the shared-memory based tensor transpose algorithm. **/
{if(alg == EFF_TRN_OFF){TRANS_SHMEM=EFF_TRN_OFF;}else{TRANS_SHMEM=EFF_TRN_ON;}; return;}

__host__ void gpu_set_matmult_algorithm(int alg){
/** Activates either cuBLAS (fast) or my own (slow) BLAS CUDA kernels. **/
#ifndef NO_BLAS
 if(alg == BLAS_ON){DISABLE_BLAS=BLAS_ON;}else{DISABLE_BLAS=BLAS_OFF;};
#endif
 return;
}

__host__ int gpu_print_stats(int gpu_num = -1)
/** Prints GPU statistics for GPU#<gpu_num>. If <gpu_num>=-1,
    prints GPU statistics for all active GPUs.
    A negative return status means invalid <gpu_num>. **/
{
 int i,b,f;
 clock_t ctm;

 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  b=gpu_num; f=gpu_num; //select a specific GPU
 }else if(gpu_num == -1){
  b=0; f=MAX_GPUS_PER_NODE-1; //select all GPUs
 }else{
  return -1; //invalid GPU number
 }
 for(i=b;i<=f;i++){
  if(gpu_is_mine(i) != GPU_OFF){
   ctm=clock();
   gpu_stats[i].time_active=((double)(ctm-gpu_stats[i].time_start))/CLOCKS_PER_SEC;
   printf("\n#MSG(TAL-SH::NV-TAL): Statistics on GPU #%d:\n",i);
   printf(" Number of tasks submitted: %llu\n",gpu_stats[i].tasks_submitted);
   printf(" Number of tasks completed: %llu\n",gpu_stats[i].tasks_completed);
   printf(" Number of tasks deferred : %llu\n",gpu_stats[i].tasks_deferred);
   printf(" Number of tasks failed   : %llu\n",gpu_stats[i].tasks_failed);
   printf(" Number of Flops processed: %G\n",gpu_stats[i].flops);
   printf(" Number of Bytes received : %G\n",gpu_stats[i].traffic_in);
   printf(" Number of Bytes sent     : %G\n",gpu_stats[i].traffic_out);
   printf(" Time active (sec)        : %f\n",gpu_stats[i].time_active);
  }else{
   printf("\n#MSG(TAL-SH::NV-TAL): Statistics on GPU #%d: GPU is OFF\n",i);
  }
 }
 return 0;
}
#endif

//TENSOR BLOCK API:
int tensShape_clean(talsh_tens_shape_t * tshape)
/** Cleans a tensor shape. A clean (initialized to null) tensor shape has .num_dim=-1.
    A further defined tensor shape has .num_dim >= 0. **/
{
 if(tshape != NULL){
  tshape->num_dim=-1; //tensor rank
  tshape->dims=NULL;  //tensor dimension extents
  tshape->divs=NULL;  //tensor dimension dividers (segment sizes)
  tshape->grps=NULL;  //tensor dimension groups
 }else{
  return -1;
 }
 return 0;
}

int tensShape_construct(talsh_tens_shape_t * tshape, int pinned, int rank, const int * dims = NULL,
                                                  const int * divs = NULL, const int * grps = NULL)
/** (Re-)defines a tensor shape. It is errorneous to pass an uninitialized tensor shape here,
    that is, the tensor shape *(tshape) must be either clean or previously defined. If <rank> > 0,
    <dims[rank]> must be supplied, whereas <divs[rank]> and <grps[rank]> are always optional.
    If <pinned> = YEP, then the multi-indices will be allocated via the multi-index bank (pinned),
    otherwise a regular malloc will be called. TRY_LATER or DEVICE_UNABLE return statuses are not
    errors and in this case the input tensor shape will stay unchanged. A return status NOT_CLEAN
    indicates an unsuccessful resource release that can be tolerated in general
    (the construction will still occur). **/
{
 int i,errc;
 int *mi_dims,*mi_divs,*mi_grps;

 errc=0;
//Check arguments:
 if(tshape == NULL) return -1;
 if(rank < 0) return -2;
 if(dims != NULL){for(i=0;i<rank;i++){if(dims[i] < 0) return -3;}}
 if(divs != NULL){for(i=0;i<rank;i++){if(divs[i] < 0) return -4;}}
 if(grps != NULL){for(i=0;i<rank;i++){if(grps[i] < 0) return -5;}}
 if(rank > 0 && dims == NULL) return -6; //dimension extents must be present for rank>0
//Acquire/release resources if needed:
 mi_dims=NULL; mi_divs=NULL; mi_grps=NULL;
 if(rank > 0 && tshape->num_dim <= 0){ //acquire multi-index resources
  if(pinned == NOPE){
   mi_dims=(int*)malloc(3*MAX_TENSOR_RANK*sizeof(int));
   if(mi_dims == NULL) return TRY_LATER;
   mi_divs=mi_dims+MAX_TENSOR_RANK;
   mi_grps=mi_divs+MAX_TENSOR_RANK;
  }else{
 //Multi-index "Dimension extents":
   errc=mi_entry_get(&mi_dims); //acquire a mi resource
   if(errc != 0){
    if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 1;}
   }
 //Multi-index "Dimension dividers":
   errc=mi_entry_get(&mi_divs); //acquire a mi resource
   if(errc != 0){
    i=mi_entry_release(mi_dims);
    if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 2;}
   }
 //Multi-index "Dimension groups":
   errc=mi_entry_get(&mi_grps); //acquire a mi resource
   if(errc != 0){
    i=mi_entry_release(mi_divs); i=mi_entry_release(mi_dims);
    if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 3;}
   }
  }
  tshape->dims=mi_dims; tshape->divs=mi_divs; tshape->grps=mi_grps;
  errc=0;
 }else if(rank == 0 && tshape->num_dim > 0){ //release multi-index resources
  errc=tensShape_destruct(tshape); if(errc != 0 && errc != NOT_CLEAN) return 4;
 }
//Define the new tensor shape:
 tshape->num_dim=rank;
 if(dims != NULL){
  for(i=0;i<rank;i++) tshape->dims[i]=dims[i];
 }
 if(divs != NULL){
  for(i=0;i<rank;i++) tshape->divs[i]=divs[i];
 }else{
  for(i=0;i<rank;i++) tshape->divs[i]=tshape->dims[i]; //default dividers (one segment per dimension)
 }
 if(grps != NULL){
  for(i=0;i<rank;i++) tshape->grps[i]=grps[i];
 }else{
  for(i=0;i<rank;i++) tshape->grps[i]=0; //default groups (all indices belong to the unrestricted group)
 }
 return errc; //either 0 or NOT_CLEAN
}

int tensShape_destruct(talsh_tens_shape_t * tshape)
/** Destructs a defined tensor shape (releases resources and cleans it).
    If the input tensor shape is initialized to null, nothing happens.
    In case of an unsuccessful resource release, a return status NOT_CLEAN
    will be returned, which can be considered as a tolerable error since
    the tensor shape will be cleaned anyway (although a leak can occur). **/
{
 int n,pinned,errc;

 n=0; //will be incremented upon an unsuccessful resource release
 if(tshape == NULL) return -1;
 if(tshape->num_dim > 0){ //need to release resources
  if(tshape->dims != NULL){
   pinned=mi_entry_pinned(tshape->dims);
   if(pinned == NOPE){
    free(tshape->dims); //will free all {dims,divs,grps}
    tshape->dims=NULL; tshape->divs=NULL; tshape->grps=NULL;
   }else{
    if(tshape->dims != NULL){errc=mi_entry_release(tshape->dims); if(errc != 0) n++; tshape->dims=NULL;} //release a mi resource
    if(tshape->divs != NULL){errc=mi_entry_release(tshape->divs); if(errc != 0) n++; tshape->divs=NULL;} //release a mi resource
    if(tshape->grps != NULL){errc=mi_entry_release(tshape->grps); if(errc != 0) n++; tshape->grps=NULL;} //release a mi resource
   }
  }else{
   return -2;
  }
 }
 if(n != 0) n=NOT_CLEAN;
 errc=tensShape_clean(tshape);
 return n; //either 0 or NOT_CLEAN
}

static size_t tensShape_volume(const talsh_tens_shape_t * tshape)
/** Returns the volume of a defined tensor shape, or 0 otherwise. **/
{
 int i;
 size_t vol;

 vol=0;
 if(tshape->num_dim >= 0 && tshape->num_dim <= MAX_TENSOR_RANK){
  vol=1;
  for(i=0;i<tshape->num_dim;i++){
   if(tshape->dims[i] > 0){
    vol*=tshape->dims[i];
   }else{
    return 0;
   }
  }
 }
 return vol;
}

int tensBlck_create(tensBlck_t **ctens)
/** Creates an empty instance of tensBlck_t and initializes it to null (on Host). **/
{
 int errc;

 *ctens=(tensBlck_t*)malloc(sizeof(tensBlck_t)); if(*ctens == NULL) return TRY_LATER;
 (*ctens)->data_kind=NO_TYPE;
 errc=tensShape_clean(&((*ctens)->shape)); if(errc != 0) return 1;
 (*ctens)->src_rsc=NULL; //source memory resource (where the tensor body is before the operation)
 (*ctens)->dst_rsc=NULL; //destination memory resource (where the tensor body will be after the operation)
 (*ctens)->tmp_rsc=NULL; //temporary memory resource (where the tensor body can be during the operation)
 (*ctens)->prmn_h=NULL;  //dimension permutation (internal use only)
 return 0;
}

int tensBlck_destroy(tensBlck_t *ctens)
/** Destroys a defined instance of tensBlck_t (either nullified or shape-defined).
    A return status NOT_CLEAN indicates an unsuccessful resource release, which
    can be considered as a tolerable error (the object will still be destroyed). **/
{
 int errc;

 errc=0;
 if(ctens == NULL) return -1;
 errc=tensBlck_destruct(ctens); if(errc != 0 && errc != NOT_CLEAN) errc=1;
 free(ctens);
 return errc;
}

int tensBlck_construct(tensBlck_t *ctens,      //pointer to defined tensor block (either nullified or defined to a value)
                       int pinned,             //YEP: tensor shape multi-indices will be pinned (for GPU), NOPE: regular malloc (not pinned)
                       int trank,              //tensor rank
                       const int *dims = NULL, //tensor dimension extents (when trank > 0)
                       const int *divs = NULL, //tensor dimension dividers (when trank > 0, optional)
                       const int *grps = NULL) //tensor dimension groups (when trank > 0, optional)
/** Constructs (defines/redefines) a tensor block without attaching its body (only the shape).
    If the tensor block is to be used on Nvidia GPUs or other asynchronous devices,
    argument <pinned> must be set to YEP (NOPE will not use pinned memory).
    A return status NOT_CLEAN indicates an unsuccessful resource release, which,
    can be considered as a tolerable error (the object will still be constructed). **/
{
 int n,errc;

 n=0;
 if(ctens == NULL) return -1;
 if(trank < 0 || trank > MAX_TENSOR_RANK) return -2; //invalid tensor rank
 if(trank > 0 && dims == NULL) return -3; //dimension extents must be present for rank>0 tensors
 errc=tensBlck_destruct(ctens); if(errc != 0){if(errc == NOT_CLEAN){n=errc;}else{return 1;}}
 errc=tensShape_construct(&(ctens->shape),pinned,trank,dims,divs,grps);
 if(errc != 0){if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 2;}}
 return n; //either 0 or NOT_CLEAN
}

int tensBlck_attach_body(tensBlck_t *ctens,     //pointer to a shape-defined (constructed) tensor block
                         int data_kind,         //data kind (R4,R8,C8)
                         int dev_id,            //flat device id where the body resides (or should reside)
                         void *body_ptr = NULL, //pointer to the tensor body (global memory of device <dev_id>)
                         int buf_entry = -1)    //argument buffer entry handle corresponding to the <body_ptr> (optional)
/** Attaches a body to a shape-defined tensor block. If both <body_ptr> and <buf_entry> are absent,
    a resource will be allocated on device <dev_id> in the device argument buffer (if available).
    If <buf_entry> is absent, a defined <body_ptr> points to an external memory (either pinned or not).
    If both <body_ptr> and <buf_entry> are defined, the external memory is assumed to be within that
    argument buffer entry. In all cases, the memory resource will be associated with the .src_rsc component
    of tensBlck_t. It is forbidden to attempt allocating/attaching a memory resource when an existing memory
    resource is still in use (this will result in an error). A return status of TRY_LATER or DEVICE_UNABLE
    indicates the current or permanent shortage in the necessary resources and is not an error. **/
{
 int errc;
 size_t vol,body_size;

 if(ctens == NULL) return -1;
 if(tens_valid_data_kind(data_kind) != YEP || data_kind == NO_TYPE) return -2;
 if(ctens->shape.num_dim < 0 || ctens->shape.num_dim > MAX_TENSOR_RANK) return -3; //tensor block must be shape-defined
 if(body_ptr == NULL && buf_entry >= 0) return -4; //a defined argument buffer entry must be supplied with the corresponding pointer
 vol=tensShape_volume(&(ctens->shape)); //tensor body volume (number of elements)
 body_size=vol*data_kind; //tensor body size in bytes
 if(body_ptr == NULL){ //allocate memory in the argument buffer
  errc=tensDevRsc_allocate_mem(ctens->src_rsc,dev_id,body_size,YEP);
  if(errc != 0){if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 1;}}
 }else{ //associate memory
  errc=tensDevRsc_attach_mem(ctens->src_rsc,dev_id,body_ptr,buf_entry);
  if(errc != 0){if(errc == TRY_LATER || errc == DEVICE_UNABLE){return errc;}else{return 2;}}
 }
 return 0;
}

int tensBlck_destruct(tensBlck_t *ctens, int release_body = YEP, int which_body = EVERYTHING)
/** Destructs a defined tensor block (releases all resources and initializes the tensor block to null).
    If <release_body> == YEP/NOPE, the global memory resources will be released/kept. Argument <which_body>
    can further regulate which tensor body to be released/kept (SOURCE, DESTINATION, TEMPORARY, EVERYTHING).
    A return status NOT_CLEAN indicates an unsuccessful resource release that may be
    considered as a tolerable error since the tensor block will be nullified anyway. **/
{
 int n,errc;

 n=0;
 if(ctens == NULL) return -1;
 if(ctens->shape.num_dim >= 0){ //shape-defined tensor block
  if(ctens->shape.num_dim > MAX_TENSOR_RANK) return -2;
//Dimension permutation (temporary resource used by Nvidia GPU):
  if(ctens->prmn_h != NULL){errc=mi_entry_release(ctens->prmn_h); if(errc != 0) n=NOT_CLEAN; ctens->prmn_h=NULL;} //release a mi resource
//Release the TEMPORARY resource:
  if(ctens->tmp_rsc != NULL &&
     ((release_body == YEP && (which_body == EVERYTHING || which_body == TEMPORARY)) ||
      (release_body == NOPE && (which_body != EVERYTHING && which_body != TEMPORARY)))){
   errc=tensDevRsc_release(ctens->tmp_rsc); if(errc != 0) n=NOT_CLEAN;
  }
  ctens->tmp_rsc=NULL;
//Release the DESTINATION resource:
  if(ctens->dst_rsc != NULL &&
     ((release_body == YEP && (which_body == EVERYTHING || which_body == DESTINATION)) ||
      (release_body == NOPE && (which_body != EVERYTHING && which_body != DESTINATION)))){
   errc=tensDevRsc_release(ctens->dst_rsc); if(errc != 0) n=NOT_CLEAN;
  }
  ctens->dst_rsc=NULL;
//Release the SOURCE resource:
  if(ctens->src_rsc != NULL &&
     ((release_body == YEP && (which_body == EVERYTHING || which_body == SOURCE)) ||
      (release_body == NOPE && (which_body != EVERYTHING && which_body != SOURCE)))){
   errc=tensDevRsc_release(ctens->src_rsc); if(errc != 0) n=NOT_CLEAN;
  }
  ctens->src_rsc=NULL;
 }else{ //nullified tensor block: All resources must have been released already
  if(ctens->src_rsc != NULL){ctens->src_rsc=NULL; n=NOT_CLEAN;}
  if(ctens->dst_rsc != NULL){ctens->dst_rsc=NULL; n=NOT_CLEAN;}
  if(ctens->tmp_rsc != NULL){ctens->tmp_rsc=NULL; n=NOT_CLEAN;}
  if(ctens->prmn_h != NULL){ctens->prmn_h=NULL; n=NOT_CLEAN;}
 }
 if(tens_valid_data_kind(ctens->data_kind) != YEP) n=NOT_CLEAN; ctens->data_kind=NO_TYPE;
 errc=tensShape_destruct(&(ctens->shape)); if(errc != 0){if(errc == NOT_CLEAN){n=NOT_CLEAN;}else{return 1;}}
 return n;
}

int tensBlck_acc_id(const tensBlck_t *ctens, int *dev_kind, int *entry_gpu, int *entry_const, int *data_kind,
                             int *there)
/** Returns Accelerator ID on which the tensor block data resides or will reside (negative return means Host residence).
INPUT:
 # ctens - pointer to an instance of tensBlck_t;
OUTPUT:
 # tensBlck_acc_id - ACC ID (non-negative) OR -1 (Host residence of the tensor block);
 # dev_kind - device (accelerator) kind (GPU, MIC, etc.);
 # entry_gpu - device global memory argument-buffer entry number;
 # entry_const - GPU constant memory entry number;
 # data_kind - 4:float, 8:double, 16:double_complex;
 # there - 0 means that tensor block elements are not in the ACC memory yet, 1 means opposite.
**/
{
 int dev_num;
 if(ctens != NULL){
  *data_kind=ctens->data_kind; *there=tensBlck_present(ctens);
  *dev_kind=DEV_HOST; dev_num=decode_device_id(ctens->device_id,dev_kind);
  if(*dev_kind == DEV_NVIDIA_GPU){ //GPU residence
   *entry_gpu=ctens->buf_entry_gpu; *entry_const=ctens->const_args_entry;
  }else{ //Not GPU residence
   *entry_gpu=-1; *entry_const=-1;
  }
 }else{
  dev_num=-1; *dev_kind=-1; *entry_gpu=-1; *entry_const=-1; *data_kind=0; *there=0;
 }
 return dev_num;
}

int tensBlck_set_presence(tensBlck_t *ctens) //Marks tensor block data as residing on GPU
{if(ctens != NULL){if(ctens->device_id < 0) ctens->device_id=-(ctens->device_id); return 0;}else{return 1;}}

int tensBlck_set_absence(tensBlck_t *ctens) //Unmarks tensor block data as residing on GPU
{if(ctens != NULL){if(ctens->device_id > 0) ctens->device_id=-(ctens->device_id); return 0;}else{return 1;}}

int tensBlck_present(const tensBlck_t *ctens) //Checks presence of the block data on Device (or Host)
{if(ctens != NULL){if(ctens->device_id >= 0){return 1;}else{return 0;}}else{return -1;}}

int tensBlck_hab_free(tensBlck_t *ctens){
/** For tensor blocks simultaneously residing on Host and GPU, frees the Host copy.
The data does not have to be present on GPU, in which case the tensor block
becomes uninitialized but still usable on the GPU. If the Host-residing tensor body
had been allocated in HAB, it frees that HAB buffer entry, otherwise deallocates
the corresponding pinned memory explicitly via <host_mem_free_pin>. **/
 int i,dev_kind,dev_num,errc;

 errc=0;
 if(ctens != NULL){
  dev_num=decode_device_id(ctens->device_id,&dev_kind);
  if(dev_num >= 0){
   if(dev_kind == DEV_NVIDIA_GPU){
    if(ctens->elems_h != NULL){
     if(ctens->buf_entry_host >= 0){
      i=free_buf_entry_host(ctens->buf_entry_host); errc+=i;
      if(i == 0) ctens->buf_entry_host=-1;
     }else{
      i=host_mem_free_pin(ctens->elems_h); errc+=i;
     }
     ctens->buf_entry_host=-1; ctens->elems_h=NULL;
    }else{
     errc+=333;
    }
   }else{
    errc+=555;
   }
  }else{
   errc+=777;
  }
 }else{
  errc+=999;
 }
 return errc;
}

size_t tensBlck_volume(const tensBlck_t *ctens) //Number of elements in a tensor block (volume)
{size_t tvol=1; for(int i=0;i<ctens->rank;i++){tvol*=(ctens->dims_h[i]);}; return tvol;}

#ifndef NO_GPU
//CUDA TASK API:
__host__ int cuda_task_create(cudaTask_t **cuda_task)
/** Creates an instance of cudaTask_t **/
{
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:cuda_task_create): sizeof(cudaTask_t) = %d",sizeof(cudaTask_t)); //debug
 *cuda_task=(cudaTask_t*)malloc(sizeof(cudaTask_t)); if(*cuda_task == NULL) return 1;
 (*cuda_task)->task_error=-1; (*cuda_task)->gpu_id=-1; (*cuda_task)->scr_entry_count=0;
 return 0;
}

__host__ int cuda_task_clean(cudaTask_t *cuda_task)
/** Cleans an existing cudaTask_t for reuse. **/
{
 int i,j,cur_gpu,err_code;
 hipError_t err;

 err_code=0;
 if(cuda_task != NULL){
  i=cuda_task_complete(cuda_task);
  if(i == CUDA_TASK_COMPLETED){ //task has completed (successfully or not)
   if(cuda_task->gpu_id >= 0 && cuda_task->gpu_id < MAX_GPUS_PER_NODE){
    cur_gpu=-1; err=hipGetDevice(&cur_gpu); if(err != hipSuccess){cur_gpu=-1; err_code+=1;}
    err=hipSuccess; if(cur_gpu != cuda_task->gpu_id) err=hipSetDevice(cuda_task->gpu_id);
    if(err == hipSuccess){
     err=hipEventDestroy(cuda_task->task_finish);
     err=hipEventDestroy(cuda_task->task_output);
     err=hipEventDestroy(cuda_task->task_comput);
     err=hipEventDestroy(cuda_task->task_start);
     err=hipStreamDestroy(cuda_task->task_stream); if(err != hipSuccess) err_code+=5;
    }else{
     err_code+=20;
    }
    if(cuda_task->scr_entry_count <= MAX_SCR_ENTRY_COUNT){
     for(i=cuda_task->scr_entry_count-1;i>=0;i--){
      j=free_buf_entry_gpu(cuda_task->gpu_id,cuda_task->scr_entry[i]); err_code+=j*100;
     }
    }else{
     err_code+=10000;
    }
    if(cur_gpu >= 0 && cur_gpu != cuda_task->gpu_id){err=hipSetDevice(cur_gpu); if(err != hipSuccess) err_code+=30000;}
   }else{
    err_code+=50000;
   }
   cuda_task->task_error=-1; cuda_task->gpu_id=-1; cuda_task->scr_entry_count=0;
  }else if(i == CUDA_TASK_EMPTY || i == CUDA_TASK_ERROR){ //empty task or cuda_task_complete() failed
   cuda_task->task_error=-1; cuda_task->gpu_id=-1; cuda_task->scr_entry_count=0;
  }else{ //task has not completed yet, thus cannot be destroyed
   err_code+=100000;
  }
 }else{
  err_code+=300000;
 }
 return err_code;
}

__host__ int cuda_task_destroy(cudaTask_t *cuda_task)
/** Destroys an instance of cudaTask_t if the CUDA task has completed or empty. **/
{
 int i,j,cur_gpu,err_code;
 hipError_t err;

 err_code=0;
 if(cuda_task != NULL){
  if(EVENT_RECORD != 0){ //`This should be done atomically for thread safety
   if(cuda_task->gpu_id >= 0 && cuda_task->gpu_id < MAX_GPUS_PER_NODE){
    if(LastTask[cuda_task->gpu_id] == cuda_task) LastTask[cuda_task->gpu_id]=NULL;
   }
  }
  i=cuda_task_complete(cuda_task);
  if(i == CUDA_TASK_COMPLETED){ //task has completed (successfully or not)
   if(cuda_task->gpu_id >= 0 && cuda_task->gpu_id < MAX_GPUS_PER_NODE){
    cur_gpu=-1; err=hipGetDevice(&cur_gpu); if(err != hipSuccess){cur_gpu=-1; err_code+=1;}
    err=hipSuccess; if(cur_gpu != cuda_task->gpu_id) err=hipSetDevice(cuda_task->gpu_id);
    if(err == hipSuccess){
     err=hipEventDestroy(cuda_task->task_finish);
     err=hipEventDestroy(cuda_task->task_output);
     err=hipEventDestroy(cuda_task->task_comput);
     err=hipEventDestroy(cuda_task->task_start);
     err=hipStreamDestroy(cuda_task->task_stream); if(err != hipSuccess) err_code+=5;
    }else{
     err_code+=20;
    }
    if(cuda_task->scr_entry_count <= MAX_SCR_ENTRY_COUNT){
     for(i=cuda_task->scr_entry_count-1;i>=0;i--){
      j=free_buf_entry_gpu(cuda_task->gpu_id,cuda_task->scr_entry[i]); err_code+=j*100;
     }
    }else{
     err_code+=10000;
    }
    if(cur_gpu >= 0 && cur_gpu != cuda_task->gpu_id){err=hipSetDevice(cur_gpu); if(err != hipSuccess) err_code+=30000;}
   }else{
    err_code+=50000;
   }
   free(cuda_task); cuda_task=NULL;
  }else if(i == CUDA_TASK_EMPTY || i == CUDA_TASK_ERROR){ //empty task or cuda_task_complete() failed
   free(cuda_task); cuda_task=NULL;
  }else{ //task has not completed yet, thus cannot be destroyed
   err_code+=100000;
  }
 }else{
  err_code+=300000;
 }
 return err_code;
}

__host__ static int cuda_task_finalize(cudaTask_t *cuda_task, int err_code, int gpu_num=-1)
/** Finalizes a CUDA task: gpu_num=-1: on Host; gpu_num>=0: on GPU#gpu_num. **/
{if(cuda_task != NULL){cuda_task->task_error=err_code; cuda_task->gpu_id=gpu_num; return 0;}else{return 1;}}

__host__ static int cuda_task_record(cudaTask_t *cuda_task, int err_code, int gpu_num, hipStream_t hip_stream,
                     hipEvent_t cuda_start, hipEvent_t cuda_comput, hipEvent_t cuda_output, hipEvent_t cuda_finish,
                     int scr_entry_cnt, int *scr_entries)
/** Registers a CUDA task: Launch-error-free tasks are recorded with .task_error=-1 (in normal progress). **/
{
 int i;
 if(cuda_task != NULL){
  if(err_code == 0){ //No error occured during the task scheduling
   cuda_task->task_error=-1;            //error code (<0: In progress; 0: Success; >0: Launch error (may be in progress))
  }else{
   cuda_task->task_error=err_code;      //error code (<0: In progress; 0: Success; >0: Launch error (may be in progress))
  }
  if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
   cuda_task->gpu_id=gpu_num;            //GPU number on which the task was scheduled
   cuda_task->task_stream=hip_stream;   //CUDA stream assinged to the task
   cuda_task->task_start=cuda_start;     //CUDA event recorded at the beginning of the task
   cuda_task->task_comput=cuda_comput;   //CUDA event recorded when the computing kernel starts (all the input data is on device)
   cuda_task->task_output=cuda_output;   //CUDA event recorded when the computing kernel finishes (before output is copied back)
   cuda_task->task_finish=cuda_finish;   //CUDA event recorded at the end of the task
   if(scr_entry_cnt >= 0 && scr_entry_cnt <= MAX_SCR_ENTRY_COUNT){
    cuda_task->scr_entry_count=scr_entry_cnt; //number of additional GPU argument buffer entries allocated by the task
    for(i=0;i<scr_entry_cnt;i++) cuda_task->scr_entry[i]=scr_entries[i]; //additional GPU argument buffer entries allocated by the task
   }else{
    return 3;
   }
  }else{
   return 2;
  }
 }else{
  return 1;
 }
 return 0;
}

__host__ int cuda_task_gpu_id(const cudaTask_t *cuda_task){return cuda_task->gpu_id;}

__host__ int cuda_task_status(cudaTask_t *cuda_task)
/** Checks the status of a CUDA task. Possible status values are listed in
tensor_algebra.h and tensor_algebra.inc: Keep them consistent! **/
{
 int task_stat,cur_gpu;
 hipError_t err;
 if(cuda_task != NULL){
  if(cuda_task->task_error == 0) return CUDA_TASK_COMPLETED; //task has successfully completed
  cur_gpu=-1; err=hipGetDevice(&cur_gpu); if(err != hipSuccess) return CUDA_TASK_ERROR;
  if(cur_gpu != cuda_task->gpu_id){
   err=hipSetDevice(cuda_task->gpu_id); if(err != hipSuccess){err=hipSetDevice(cur_gpu); return CUDA_TASK_ERROR;}
  }
  err=hipEventQuery(cuda_task->task_finish);
  if(err == hipSuccess){
   if(cuda_task->task_error < 0) cuda_task->task_error=0; //task completed successfully
   task_stat=CUDA_TASK_COMPLETED; //task completed
  }else{
   err=hipEventQuery(cuda_task->task_output);
   if(err == hipSuccess){
    task_stat=CUDA_TASK_OUTPUT_THERE; //computing kernel has finished
   }else{
    err=hipEventQuery(cuda_task->task_comput);
    if(err == hipSuccess){
     task_stat=CUDA_TASK_INPUT_THERE; //computation started, input data is on device (can be reused later)
    }else{
     err=hipEventQuery(cuda_task->task_start);
     if(err == hipSuccess){
      task_stat=CUDA_TASK_STARTED; //task started
     }else{
      task_stat=CUDA_TASK_SCHEDULED; //task has not started yet
     }
    }
   }
  }
  if(cur_gpu != cuda_task->gpu_id){err=hipSetDevice(cur_gpu); if(err != hipSuccess) return CUDA_TASK_ERROR;}
 }else{
  task_stat=CUDA_TASK_EMPTY;
 }
 return task_stat;
}

__host__ int cuda_task_complete(cudaTask_t *cuda_task)
/** Returns CUDA_TASK_COMPLETED if an existing CUDA task <cuda_task> has completed.
Note that having cuda_task->task_error=0 suggests completion without further querying!
Other possible outputs: CUDA_TASK_EMPTY, CUDA_TASK_SCHEDULED, CUDA_TASK_COMPLETED, CUDA_TASK_ERROR.
**/
{
 int err_code,cur_gpu;
 hipError_t err;
 err_code=CUDA_TASK_EMPTY;
 if(cuda_task != NULL){
  if(cuda_task->task_error != 0){ //Negative: Task in progress or empty; Positive: Task scheduling error occured
   cur_gpu=-1; err=hipGetDevice(&cur_gpu); if(err != hipSuccess) return CUDA_TASK_ERROR;
   if(cur_gpu != cuda_task->gpu_id){
    err=hipSetDevice(cuda_task->gpu_id); if(err != hipSuccess){err=hipSetDevice(cur_gpu); return CUDA_TASK_ERROR;}
   }
   err=hipStreamQuery(cuda_task->task_stream);
   if(err != hipSuccess && err != hipErrorInvalidHandle){ //task is still in progress
    err_code=CUDA_TASK_SCHEDULED;
   }else{ //task completed successfully or has never been scheduled
    if(err == hipErrorInvalidHandle){ //stream does not exist
     err_code=CUDA_TASK_EMPTY;
    }else{
     err_code=CUDA_TASK_COMPLETED; if(cuda_task->task_error < 0) cuda_task->task_error=0;
    }
   }
   if(cur_gpu != cuda_task->gpu_id){err=hipSetDevice(cur_gpu); if(err != hipSuccess) return CUDA_TASK_ERROR;}
  }else{
   err_code=CUDA_TASK_COMPLETED;
  }
 }
 return err_code;
}

__host__ int cuda_task_wait(cudaTask_t *cuda_task)
/** Waits on accomplishment of a CUDA task: Returns the output of cuda_task_complete().
Possible values are CUDA_TASK_COMPLETED, CUDA_TASK_ERROR, CUDA_TASK_EMPTY. **/
{
 int i,j;
 i=CUDA_TASK_SCHEDULED; j=1;
 while(j>0){
  i=cuda_task_complete(cuda_task);
  if(i == CUDA_TASK_COMPLETED || i == CUDA_TASK_ERROR || i == CUDA_TASK_EMPTY) j--;
 }
 return i;
}

__host__ int cuda_tasks_wait(int num_tasks, cudaTask_t **cuda_tasks, int* task_stats)
/** Waits upon completion of a series of CUDA tasks. Returns 0 on success. **/
{
 int i,j,n;
 if(num_tasks >= 0){
  if(num_tasks > 0){
   if(cuda_tasks != NULL && task_stats != NULL){
    for(i=0;i<num_tasks;i++){task_stats[i]=CUDA_TASK_SCHEDULED;}
    n=num_tasks;
    while(n>0){
     for(i=0;i<num_tasks;i++){
      j=task_stats[i];
      if(j != CUDA_TASK_COMPLETED && j != CUDA_TASK_ERROR && j != CUDA_TASK_EMPTY){
       if(cuda_tasks[i] != NULL){
        j=cuda_task_complete(cuda_tasks[i]); task_stats[i]=j;
        if(j == CUDA_TASK_COMPLETED || j == CUDA_TASK_ERROR || j == CUDA_TASK_EMPTY) n--;
       }else{
        return 1;
       }
      }
     }
    }
   }else{
    return 2;
   }
  }
 }else{
  return 3;
 }
 return 0;
}

__host__ float cuda_task_time(const cudaTask_t *cuda_task, float *in_copy, float *out_copy, float *comp)
/** Returns the time (in seconds) the CUDA task took to complete (only when EVENT_RECORD != 0).
Also, in_copy is input copying time, out_copy is output copying time, and comp is computing time in sec. **/
{
 int cur_gpu;
 float time_ms;
 hipError_t err;
 if(cuda_task != NULL){
  if(EVENT_RECORD != 0){
   cur_gpu=-1; err=hipGetDevice(&cur_gpu); if(err != hipSuccess) return -2.0f;
   if(cur_gpu != cuda_task->gpu_id){err=hipSetDevice(cuda_task->gpu_id); if(err != hipSuccess) return -3.0f;}
   err=hipEventElapsedTime(&time_ms,cuda_task->task_start,cuda_task->task_comput); //time in miliseconds
   if(err == hipSuccess){*in_copy=time_ms/1000.0f;}else{*in_copy=-1.0f;}
   err=hipEventElapsedTime(&time_ms,cuda_task->task_comput,cuda_task->task_output); //time in miliseconds
   if(err == hipSuccess){*comp=time_ms/1000.0f;}else{*comp=-1.0f;}
   err=hipEventElapsedTime(&time_ms,cuda_task->task_output,cuda_task->task_finish); //time in miliseconds
   if(err == hipSuccess){*out_copy=time_ms/1000.0f;}else{*out_copy=-1.0f;}
   err=hipEventElapsedTime(&time_ms,cuda_task->task_start,cuda_task->task_finish); //time in miliseconds
   if(err == hipSuccess){time_ms/=1000.0f;}else{time_ms=-1.0f;} //time in seconds
   if(cur_gpu != cuda_task->gpu_id){err=hipSetDevice(cur_gpu); if(err != hipSuccess) return -4.0f;}
   return time_ms;
  }else{ //timing events are disabled
   return -5.0f;
  }
 }else{
  return -13.0f; //empty task
 }
}

//----------------------------------------------------------------------------------------
//EXPORTED FUNCTIONS (callable from Fortran):
//----------------------------------------------------
// CPU->GPU TENSOR BLOCK COPY (blocking):
__host__ int gpu_put_arg(tensBlck_t *ctens) //Blocking
/** This function copies a tensor block from the Host argument buffer into a GPU argument buffer **/
{
 int i,dev_kind,dev_num,gpu_num;
 size_t tsize;
 hipError_t err;
 if(ctens != NULL){
  if(ctens->elems_h != NULL && ctens->elems_d != NULL){
   if(ctens->rank >= 0 && ctens->data_kind > 0){ //data_kind = {4|8|16}
    err=hipGetLastError(); err=hipSuccess;
    dev_kind=DEV_HOST; dev_num=decode_device_id(ctens->device_id,&dev_kind);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0){
     err=hipGetDevice(&gpu_num); if(err != hipSuccess) return 1;
     if(dev_num != gpu_num){err=hipSetDevice(dev_num); if(err != hipSuccess){err=hipSetDevice(gpu_num); return 2;}}
     tsize=tensBlck_volume(ctens); //tensor block size (elements)
     if(tsize > 0){
      err=hipMemcpy(ctens->elems_d,ctens->elems_h,tsize*(ctens->data_kind),hipMemcpyHostToDevice);
      if(err != hipSuccess){err=hipSetDevice(gpu_num); return 3;}
      i=tensBlck_set_presence(ctens); if(i != 0){err=hipSetDevice(gpu_num); return 4;}
      if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess) return 5;}
     }else{
      err=hipSetDevice(gpu_num); return 6;
     }
    }else{
     return 7;
    }
   }else{
    return 8;
   }
  }else{
   return 9;
  }
 }else{
  return 10;
 }
 return 0;
}
//----------------------------------------------------
// GPU->CPU TENSOR BLOCK COPY (blocking):
__host__ int gpu_get_arg(tensBlck_t *ctens) //Blocking
/** This function copies a tensor block from a GPU argument buffer into the Host argument buffer **/
{
 int dev_kind,dev_num,gpu_num;
 size_t tsize;
 hipError_t err;
 if(ctens != NULL){
  if(ctens->elems_h != NULL && ctens->elems_d != NULL){
   if(ctens->rank >= 0 && ctens->data_kind > 0){ //data_kind = {4|8|16}
    err=hipGetLastError(); err=hipSuccess;
    dev_kind=DEV_HOST; dev_num=decode_device_id(ctens->device_id,&dev_kind);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0){
     if(ctens->device_id > 0){ //tensor block must be present on GPU
      err=hipGetDevice(&gpu_num); if(err != hipSuccess) return 1;
      if(dev_num != gpu_num){err=hipSetDevice(dev_num); if(err != hipSuccess){err=hipSetDevice(gpu_num); return 2;}}
      tsize=tensBlck_volume(ctens); //tensor block size (elements)
      if(tsize > 0){
       err=hipMemcpy(ctens->elems_h,ctens->elems_d,tsize*(ctens->data_kind),hipMemcpyDeviceToHost);
       if(err != hipSuccess){err=hipSetDevice(gpu_num); return 3;}
       if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess) return 4;}
      }else{
       err=hipSetDevice(gpu_num); return 5;
      }
     }else{
      return 6;
     }
    }else{
     return 7;
    }
   }else{
    return 8;
   }
  }else{
   return 9;
  }
 }else{
  return 10;
 }
 return 0;
}
//-----------------------------------------------------------------
// CPU->GPU TENSOR BLOCK COPY (non-blocking):
__host__ int gpu_put_arg_(tensBlck_t *ctens, cudaTask_t *cuda_task) //Non-blocking
/** This function copies a tensor block from the Host argument buffer into a GPU argument buffer **/
{
 int i,dev_kind,dev_num,gpu_num;
 size_t tsize;
 hipStream_t hip_stream;
 hipEvent_t cuda_start,cuda_finish;
 hipError_t err;
 if(ctens != NULL && cuda_task != NULL){
  if(ctens->elems_h != NULL && ctens->elems_d != NULL){
   if(ctens->rank >= 0 && ctens->data_kind > 0){ //data_kind = {4|8|16}
    err=hipGetLastError(); err=hipSuccess;
    dev_kind=DEV_HOST; dev_num=decode_device_id(ctens->device_id,&dev_kind);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0){
     err=hipGetDevice(&gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,1); return 1;}
     if(dev_num != gpu_num){ //set new GPU
      err=hipSetDevice(dev_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,2); err=hipSetDevice(gpu_num); return 2;}
     }
     i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_start); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_finish); if(err != hipSuccess) i++;
     if(i != 0){
      i=cuda_task_record(cuda_task,3,dev_num,hip_stream,cuda_start,cuda_finish,cuda_finish,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 3;
     }
     tsize=tensBlck_volume(ctens); //tensor block size (elements)
     if(tsize > 0){
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,4,dev_num,hip_stream,cuda_start,cuda_finish,cuda_finish,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 4;
       }
      }
      err=hipMemcpyAsync(ctens->elems_d,ctens->elems_h,tsize*(ctens->data_kind),hipMemcpyHostToDevice,hip_stream);
      if(err != hipSuccess){
       i=cuda_task_record(cuda_task,5,dev_num,hip_stream,cuda_start,cuda_finish,cuda_finish,cuda_finish,0,NULL);
       err=hipSetDevice(gpu_num); return 5;
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,6,dev_num,hip_stream,cuda_start,cuda_finish,cuda_finish,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 6;
       }
      }
      i=cuda_task_record(cuda_task,0,dev_num,hip_stream,cuda_start,cuda_finish,cuda_finish,cuda_finish,0,NULL);
      if(i!=0){i=cuda_task_finalize(cuda_task,7,dev_num); err=hipSetDevice(gpu_num); return 7;}
      if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,8,dev_num); return 8;}}
     }else{
      i=cuda_task_record(cuda_task,9,dev_num,hip_stream,cuda_start,cuda_finish,cuda_finish,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 9;
     }
    }else{
     i=cuda_task_finalize(cuda_task,10); return 10;
    }
   }else{
    i=cuda_task_finalize(cuda_task,11); return 11;
   }
  }else{
   i=cuda_task_finalize(cuda_task,12); return 12;
  }
 }else{
  i=cuda_task_finalize(cuda_task,13); return 13;
 }
 return 0;
}
//-----------------------------------------------------------------
// GPU->CPU TENSOR BLOCK COPY (non-blocking):
__host__ int gpu_get_arg_(tensBlck_t *ctens, cudaTask_t *cuda_task) //Non-blocking
/** This function copies a tensor block from a GPU argument buffer into the Host argument buffer **/
{
 int i,dev_kind,dev_num,gpu_num;
 size_t tsize;
 hipStream_t hip_stream;
 hipEvent_t cuda_start,cuda_finish;
 hipError_t err;
 if(ctens != NULL && cuda_task != NULL){
  if(ctens->elems_h != NULL && ctens->elems_d != NULL){
   if(ctens->rank >= 0 && ctens->data_kind > 0){ //data_kind = {4|8|16}
    err=hipGetLastError(); err=hipSuccess;
    dev_kind=DEV_HOST; dev_num=decode_device_id(ctens->device_id,&dev_kind);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0){
     if(ctens->device_id > 0){ //tensor block must be present on GPU
      err=hipGetDevice(&gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,1); return 1;}
      if(dev_num != gpu_num){
       err=hipSetDevice(dev_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,2); err=hipSetDevice(gpu_num); return 2;}
      }
      i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
      err=hipEventCreate(&cuda_start); if(err != hipSuccess) i++;
      err=hipEventCreate(&cuda_finish); if(err != hipSuccess) i++;
      if(i != 0){
       i=cuda_task_record(cuda_task,3,dev_num,hip_stream,cuda_start,cuda_start,cuda_start,cuda_finish,0,NULL);
       err=hipSetDevice(gpu_num); return 3;
      }
      tsize=tensBlck_volume(ctens); //tensor block size (elements)
      if(tsize > 0){
       if(EVENT_RECORD != 0){
        err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
         i=cuda_task_record(cuda_task,4,dev_num,hip_stream,cuda_start,cuda_start,cuda_start,cuda_finish,0,NULL);
         err=hipSetDevice(gpu_num); return 4;
        }
       }
       err=hipMemcpyAsync(ctens->elems_h,ctens->elems_d,tsize*(ctens->data_kind),hipMemcpyDeviceToHost,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,5,dev_num,hip_stream,cuda_start,cuda_start,cuda_start,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 5;
       }
       if(EVENT_RECORD != 0){
        err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
         i=cuda_task_record(cuda_task,6,dev_num,hip_stream,cuda_start,cuda_start,cuda_start,cuda_finish,0,NULL);
         err=hipSetDevice(gpu_num); return 6;
        }
       }
       i=cuda_task_record(cuda_task,0,dev_num,hip_stream,cuda_start,cuda_start,cuda_start,cuda_finish,0,NULL);
       if(i!=0){i=cuda_task_finalize(cuda_task,7,dev_num); err=hipSetDevice(gpu_num); return 7;}
       if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,8,dev_num); return 8;}}
      }else{
       i=cuda_task_record(cuda_task,9,dev_num,hip_stream,cuda_start,cuda_start,cuda_start,cuda_finish,0,NULL);
       err=hipSetDevice(gpu_num); return 9;
      }
     }else{
      i=cuda_task_finalize(cuda_task,10); return 10;
     }
    }else{
     i=cuda_task_finalize(cuda_task,11); return 11;
    }
   }else{
    i=cuda_task_finalize(cuda_task,12); return 12;
   }
  }else{
   i=cuda_task_finalize(cuda_task,13); return 13;
  }
 }else{
  i=cuda_task_finalize(cuda_task,14); return 14;
 }
 return 0;
}
//-------------------------------------------------------------------------------
// SQUARED 2-NORM OF AN ARRAY (R4) RESIDING ON GPU (blocking):
__host__ int gpu_array_2norm2_r4(size_t arr_size, const float *arr, float *norm2)
/** This function computes the sum of squared elements of a <float>
array arr(0:arr_size-1) which already resides on GPU.
Executed on the currently set GPU device. **/
{
 int i,bx;
 size_t l,mthr,sz;
 float *bnorm2;
 const char *err_msg;
 hipError_t err;
 if(arr != NULL && norm2 != NULL && arr_size > 0){
  err=hipGetLastError(); err=hipSuccess;
  mthr=MAX_CUDA_BLOCKS*THRDS_ARRAY_NORM2; *norm2=0.0f;
  err=hipGetSymbolAddress((void**)&bnorm2,gpu_blck_norms2_r4); if(err != hipSuccess) return 1;
  for(l=0;l<arr_size;l+=mthr){
   if(l+mthr > arr_size){sz=arr_size-l;}else{sz=mthr;}; bx=1+(int)((sz-1)/THRDS_ARRAY_NORM2);
   gpu_array_2norm2_r4__<<<bx,THRDS_ARRAY_NORM2,THRDS_ARRAY_NORM2*sizeof(float)>>>(sz,&arr[l],bnorm2);
   err=hipDeviceSynchronize();
   if(err != hipSuccess){
    err_msg=hipGetErrorString(err);
    if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_array_2norm2_r4): Kernel sync error: %s\n",err_msg);
    return 2;
   }
   err=hipGetLastError();
   if(err != hipSuccess){
    err_msg=hipGetErrorString(err);
    if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_array_2norm2_r4): Kernel error: %s\n",err_msg);
    return 3;
   }
   err=hipMemcpyFromSymbol((void*)blck_norms2_r4,gpu_blck_norms2_r4,bx*sizeof(float),0,hipMemcpyDeviceToHost);
   if(err != hipSuccess){
    err_msg=hipGetErrorString(err);
    if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_array_2norm2_r4): Copy error: %s\n",err_msg);
    return 4;
   }
   for(i=0;i<bx;i++) *norm2+=blck_norms2_r4[i];
  }
 }else{
  return 5;
 }
 return 0;
}
//---------------------------------------------------------------------------------
// SQUARED 2-NORM OF AN ARRAY (R8) RESIDING ON GPU (blocking):
__host__ int gpu_array_2norm2_r8(size_t arr_size, const double *arr, double *norm2)
/** This function computes the sum of squared elements of a <double>
 array arr(0:arr_size-1) which already resides on GPU.
Executed on the currently set GPU device. **/
{
 int i,bx;
 size_t l,mthr,sz;
 double *bnorm2;
 const char *err_msg;
 hipError_t err;
 if(arr != NULL && norm2 != NULL && arr_size > 0){
  err=hipGetLastError(); err=hipSuccess;
  mthr=MAX_CUDA_BLOCKS*THRDS_ARRAY_NORM2; *norm2=0.0;
  err=hipGetSymbolAddress((void**)&bnorm2,gpu_blck_norms2_r8); if(err != hipSuccess) return 1;
  for(l=0;l<arr_size;l+=mthr){
   if(l+mthr > arr_size){sz=arr_size-l;}else{sz=mthr;}; bx=1+(int)((sz-1)/THRDS_ARRAY_NORM2);
   gpu_array_2norm2_r8__<<<bx,THRDS_ARRAY_NORM2,THRDS_ARRAY_NORM2*sizeof(double)>>>(sz,&arr[l],bnorm2);
   err=hipDeviceSynchronize();
   if(err != hipSuccess){
    err_msg=hipGetErrorString(err);
    if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_array_2norm2_r8): Kernel sync error: %s\n",err_msg);
    return 2;
   }
   err=hipGetLastError();
   if(err != hipSuccess){
    err_msg=hipGetErrorString(err);
    if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_array_2norm2_r8): Kernel error: %s\n",err_msg);
    return 3;
   }
   err=hipMemcpyFromSymbol((void*)blck_norms2_r8,gpu_blck_norms2_r8,bx*sizeof(double),0,hipMemcpyDeviceToHost);
   if(err != hipSuccess){
    err_msg=hipGetErrorString(err);
    if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_array_2norm2_r8): Copy error: %s\n",err_msg);
    return 4;
   }
   for(i=0;i<bx;i++) *norm2+=blck_norms2_r8[i];
  }
 }else{
  return 5;
 }
 return 0;
}
//---------------------------------------------------------------------------------------
// MATRIX MULTIPLICATION 'TN' (R4) (blocking):
__host__ int gpu_matrix_multiply_tn_r4(size_t ll, size_t lr, size_t lc,
                                       const float *lmat, const float *rmat, float *dmat)
/** dmat(0:ll-1,0:lr-1)+=lmat(0:lc-1,0:ll-1)*rmat(0:lc-1,0:lr-1)
All matrices are in Host memory. Executed on the currently set GPU device. **/
{
 size_t dsize,lsize,rsize;
 float *dptr,*lptr,*rptr;
 int bx,by,err_code;
 const char *err_msg;
 hipError_t err;
 if(lc > 0 && ll > 0 && lr > 0 && lmat != NULL && rmat != NULL && dmat != NULL){
  err=hipGetLastError(); err=hipSuccess;
  dsize=ll*lr*sizeof(float); lsize=lc*ll*sizeof(float); rsize=lc*lr*sizeof(float);
  err_code=gpu_mem_alloc((void**)&dptr,dsize); if(err_code != 0) return 1;
  err_code=gpu_mem_alloc((void**)&lptr,lsize); if(err_code != 0) return 2;
  err_code=gpu_mem_alloc((void**)&rptr,rsize); if(err_code != 0) return 3;
  err=hipMemcpy((void*)dptr,(void*)dmat,dsize,hipMemcpyHostToDevice); if(err != hipSuccess) return 4;
  err=hipMemcpy((void*)lptr,(void*)lmat,lsize,hipMemcpyHostToDevice); if(err != hipSuccess) return 5;
  err=hipMemcpy((void*)rptr,(void*)rmat,rsize,hipMemcpyHostToDevice); if(err != hipSuccess) return 6;
  err_code=gpu_get_error_count();
  bx=1+(ll-1)/MAT_MULT_TILE_DIMX; by=1+(lr-1)/MAT_MULT_TILE_DIMY; limit_cuda_blocks2d(MAX_CUDA_BLOCKS,&bx,&by);
  dim3 blcks(bx,by); dim3 thrds(MAT_MULT_TILE_DIMX,MAT_MULT_TILE_DIMY);
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_matrix_multiply_tn_r4): Running GPU kernel ..."); //debug
  gpu_matrix_multiply_tn_r4__<<<blcks,thrds>>>(ll,lr,lc,lptr,rptr,dptr);
  err=hipDeviceSynchronize(); if(err != hipSuccess) return 7;
  err=hipGetLastError();
  if(err!=hipSuccess){
   err_msg=hipGetErrorString(err);
   if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_matrix_multiply_tn_r4): Kernel error: %s\n",err_msg);
   return 8;
  }
  if(gpu_get_error_count() > err_code) return 9;
//printf("Done: %d",err); //debug
  err=hipMemcpy((void*)dmat,(void*)dptr,dsize,hipMemcpyDeviceToHost); if(err != hipSuccess) return 10;
  err=hipDeviceSynchronize(); if(err != hipSuccess) return 11;
  err_code=gpu_mem_free((void*)rptr); if(err_code != 0) return 12;
  err_code=gpu_mem_free((void*)lptr); if(err_code != 0) return 13;
  err_code=gpu_mem_free((void*)dptr); if(err_code != 0) return 14;
  err=hipDeviceSynchronize(); if(err != hipSuccess) return 15;
 }else{
  return 16;
 }
 return 0;
}
//------------------------------------------------------------------------------------------
// MATRIX MULTIPLICATION 'TN' (R8) (blocking):
__host__ int gpu_matrix_multiply_tn_r8(size_t ll, size_t lr, size_t lc,
                                       const double *lmat, const double *rmat, double *dmat)
/** dmat(0:ll-1,0:lr-1)+=lmat(0:lc-1,0:ll-1)*rmat(0:lc-1,0:lr-1)
All matrices are in Host memory. Executed on the currently set GPU device. **/
{
 size_t dsize,lsize,rsize;
 double *dptr,*lptr,*rptr;
 int bx,by,err_code;
 const char *err_msg;
 hipError_t err;
 if(lc > 0 && ll > 0 && lr > 0 && lmat != NULL && rmat != NULL && dmat != NULL){
  err=hipGetLastError(); err=hipSuccess;
  dsize=ll*lr*sizeof(double); lsize=lc*ll*sizeof(double); rsize=lc*lr*sizeof(double);
  err_code=gpu_mem_alloc((void**)&dptr,dsize); if(err_code != 0) return 1;
  err_code=gpu_mem_alloc((void**)&lptr,lsize); if(err_code != 0) return 2;
  err_code=gpu_mem_alloc((void**)&rptr,rsize); if(err_code != 0) return 3;
  err=hipMemcpy((void*)dptr,(void*)dmat,dsize,hipMemcpyHostToDevice); if(err != hipSuccess) return 4;
  err=hipMemcpy((void*)lptr,(void*)lmat,lsize,hipMemcpyHostToDevice); if(err != hipSuccess) return 5;
  err=hipMemcpy((void*)rptr,(void*)rmat,rsize,hipMemcpyHostToDevice); if(err != hipSuccess) return 6;
  err_code=gpu_get_error_count();
  bx=1+(ll-1)/MAT_MULT_TILE_DIMX; by=1+(lr-1)/MAT_MULT_TILE_DIMY; limit_cuda_blocks2d(MAX_CUDA_BLOCKS,&bx,&by);
  dim3 blcks(bx,by); dim3 thrds(MAT_MULT_TILE_DIMX,MAT_MULT_TILE_DIMY);
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_matrix_multiply_tn_r8): Running GPU kernel ..."); //debug
  gpu_matrix_multiply_tn_r8__<<<blcks,thrds>>>(ll,lr,lc,lptr,rptr,dptr);
  err=hipDeviceSynchronize(); if(err != hipSuccess) return 7;
  err=hipGetLastError();
  if(err!=hipSuccess){
   err_msg=hipGetErrorString(err);
   if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_matrix_multiply_tn_r8): Kernel error: %s\n",err_msg);
   return 8;
  }
  if(gpu_get_error_count() > err_code) return 9;
//printf("Done: %d",err); //debug
  err=hipMemcpy((void*)dmat,(void*)dptr,dsize,hipMemcpyDeviceToHost); if(err != hipSuccess) return 10;
  err=hipDeviceSynchronize(); if(err != hipSuccess) return 11;
  err_code=gpu_mem_free((void*)rptr); if(err_code != 0) return 12;
  err_code=gpu_mem_free((void*)lptr); if(err_code != 0) return 13;
  err_code=gpu_mem_free((void*)dptr); if(err_code != 0) return 14;
  err=hipDeviceSynchronize(); if(err != hipSuccess) return 15;
 }else{
  return 16;
 }
 return 0;
}
//------------------------------------------------------------------------------------------------------
// TENSOR BLOCK INITIALIZATION (non-blocking):
__host__ int gpu_tensor_block_init_(tensBlck_t *ctens, double val, int copy_back, cudaTask_t *cuda_task)
/** ctens=val: The GPU part of <ctens> will be initialized to value <val>.
If <copy_back> = 0, no copy back to Host: One must use gpu_get_arg() explicitly: Careful! **/
{
 size_t tsize;
 int i,bx,dev_num,dev_kind,gpu_num;
 hipStream_t hip_stream;
 hipEvent_t cuda_start,cuda_output,cuda_finish;
 hipError_t err;
 if(ctens != NULL && cuda_task != NULL){
  if(ctens->elems_h != NULL && ctens->elems_d != NULL){
   if(ctens->rank >= 0 && ctens->data_kind > 0){
    err=hipGetLastError(); err=hipSuccess;
    dev_num=decode_device_id(ctens->device_id,&dev_kind);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0){
     err=hipGetDevice(&gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,1); return 1;}
     if(dev_num != gpu_num){err=hipSetDevice(dev_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,2); err=hipSetDevice(gpu_num); return 2;}}
     i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_start); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_output); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_finish); if(err != hipSuccess) i++;
     if(i != 0){
      i=cuda_task_record(cuda_task,3,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 3;
     }
     tsize=tensBlck_volume(ctens); //tensor block size (elements)
     if(tsize > 0){
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,4,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 4;
       }
      }
      bx=1+(tsize-1)/THRDS_ARRAY_INIT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(ctens->data_kind){
       case R4:
        gpu_array_init_r4__<<<bx,THRDS_ARRAY_INIT,0,hip_stream>>>(tsize,(float*)(ctens->elems_d),(float)val);
        break;
       case R8:
        gpu_array_init_r8__<<<bx,THRDS_ARRAY_INIT,0,hip_stream>>>(tsize,(double*)(ctens->elems_d),val);
        break;
       default:
        i=cuda_task_record(cuda_task,5,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 5;
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_output,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,6,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 6;
       }
      }
      if(copy_back != NO_COPY_BACK){
       err=hipMemcpyAsync(ctens->elems_h,ctens->elems_d,tsize*(ctens->data_kind),hipMemcpyDeviceToHost,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,7,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 7;
       }
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,8,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 8;
       }
      }
      i=cuda_task_record(cuda_task,0,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
      if(i!=0){i=cuda_task_finalize(cuda_task,9,dev_num); err=hipSetDevice(gpu_num); return 9;}
      if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,10,dev_num); return 10;}}
     }else{
      i=cuda_task_record(cuda_task,11,dev_num,hip_stream,cuda_start,cuda_start,cuda_output,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 11;
     }
    }else{
     i=cuda_task_finalize(cuda_task,12); return 12;
    }
   }else{
    i=cuda_task_finalize(cuda_task,13); return 13;
   }
  }else{
   i=cuda_task_finalize(cuda_task,14); return 14;
  }
 }else{
  i=cuda_task_finalize(cuda_task,15); return 15;
 }
 return 0;
}
//-------------------------------------------------------------------------------------------------------
// TENSOR BLOCK RESCALING (non-blocking):
__host__ int gpu_tensor_block_scale_(tensBlck_t *ctens, double val, int copy_back, cudaTask_t *cuda_task)
/** ctens*=val: The GPU part of <ctens> will be scaled by the scalar <val>.
If <copy_back> = 0, no copy back to Host: One must use gpu_get_arg() explicitly: Careful! **/
{
 size_t tsize;
 int i,bx,dev_num,dev_kind,gpu_num;
 hipStream_t hip_stream;
 hipEvent_t cuda_start,cuda_comput,cuda_output,cuda_finish;
 hipError_t err;
 if(ctens != NULL && cuda_task != NULL){
  if(ctens->elems_h != NULL && ctens->elems_d != NULL){
   if(ctens->rank >= 0 && ctens->data_kind > 0){
    err=hipGetLastError(); err=hipSuccess;
    dev_num=decode_device_id(ctens->device_id,&dev_kind);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0){
     err=hipGetDevice(&gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,1); return 1;}
     if(dev_num != gpu_num){err=hipSetDevice(dev_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,2); err=hipSetDevice(gpu_num); return 2;}}
     i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_start); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_comput); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_output); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_finish); if(err != hipSuccess) i++;
     if(i != 0){
      i=cuda_task_record(cuda_task,3,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 3;
     }
     tsize=tensBlck_volume(ctens); //tensor block size (elements)
     if(tsize > 0){
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,4,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 4;
       }
      }
      if(ctens->device_id < 0){
       err=hipMemcpyAsync(ctens->elems_d,ctens->elems_h,tsize*(ctens->data_kind),hipMemcpyHostToDevice,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,5,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 5;
       }
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_comput,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,6,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 6;
       }
      }
      bx=1+(tsize-1)/THRDS_ARRAY_SCALE; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(ctens->data_kind){
       case R4:
        gpu_array_scale_r4__<<<bx,THRDS_ARRAY_SCALE,0,hip_stream>>>(tsize,(float*)(ctens->elems_d),(float)val);
        break;
       case R8:
        gpu_array_scale_r8__<<<bx,THRDS_ARRAY_SCALE,0,hip_stream>>>(tsize,(double*)(ctens->elems_d),val);
        break;
       default:
        i=cuda_task_record(cuda_task,7,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 7;
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_output,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,8,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 8;
       }
      }
      if(copy_back != NO_COPY_BACK){
       err=hipMemcpyAsync(ctens->elems_h,ctens->elems_d,tsize*(ctens->data_kind),hipMemcpyDeviceToHost,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,9,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 9;
       }
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,10,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 10;
       }
      }
      i=cuda_task_record(cuda_task,0,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
      if(i!=0){i=cuda_task_finalize(cuda_task,11,dev_num); err=hipSetDevice(gpu_num); return 11;}
      if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,12,dev_num); return 12;}}
     }else{
      i=cuda_task_record(cuda_task,13,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 13;
     }
    }else{
     i=cuda_task_finalize(cuda_task,14); return 14;
    }
   }else{
    i=cuda_task_finalize(cuda_task,15); return 15;
   }
  }else{
   i=cuda_task_finalize(cuda_task,16); return 16;
  }
 }else{
  i=cuda_task_finalize(cuda_task,17); return 17;
 }
 return 0;
}
//----------------------------------------------------------------------------------------
// TENSOR ADDITION [DLF] (non-blocking):
__host__ int gpu_tensor_block_add_dlf_(tensBlck_t *ctens0, tensBlck_t *ctens1, double val,
                                       int copy_back, cudaTask_t *cuda_task)
/** ctens0+=ctens1*val:
If <copy_back> = 0, no copy back to Host: One must use gpu_get_arg() explicitly: Careful! **/
{
 size_t tsize;
 int i,bx,dev_num,dev_kind,gpu_num;
 hipStream_t hip_stream;
 hipEvent_t cuda_start,cuda_comput,cuda_output,cuda_finish;
 hipError_t err;
 if(ctens0 != NULL && ctens1 != NULL && cuda_task != NULL){
  if(ctens0->elems_h != NULL && ctens0->elems_d != NULL && ctens1->elems_h != NULL && ctens1->elems_d != NULL){
   if(ctens0->rank >= 0 && ctens1->rank == ctens0->rank && ctens0->data_kind > 0 && ctens1->data_kind == ctens0->data_kind){
    for(i=0;i<ctens0->rank;i++){if(ctens0->dims_h[i] != ctens1->dims_h[i]){bx=cuda_task_finalize(cuda_task,1); return 1;}}
    err=hipGetLastError(); err=hipSuccess;
    dev_num=decode_device_id(ctens0->device_id,&dev_kind); i=decode_device_id(ctens1->device_id,&bx);
    if(dev_kind == DEV_NVIDIA_GPU && dev_num >= 0 && bx == dev_kind && i == dev_num){
     err=hipGetDevice(&gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,2); return 2;}
     if(dev_num != gpu_num){err=hipSetDevice(dev_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,3); err=hipSetDevice(gpu_num); return 3;}}
     i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_start); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_comput); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_output); if(err != hipSuccess) i++;
     err=hipEventCreate(&cuda_finish); if(err != hipSuccess) i++;
     if(i != 0){
      i=cuda_task_record(cuda_task,4,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 4;
     }
     tsize=tensBlck_volume(ctens0); //tensor block size (elements)
     if(tsize > 0 && tsize == tensBlck_volume(ctens1)){
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,5,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 5;
       }
      }
      if(ctens0->device_id < 0){
       err=hipMemcpyAsync(ctens0->elems_d,ctens0->elems_h,tsize*(ctens0->data_kind),hipMemcpyHostToDevice,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,6,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 6;
       }
      }
      if(ctens1->device_id < 0){
       err=hipMemcpyAsync(ctens1->elems_d,ctens1->elems_h,tsize*(ctens1->data_kind),hipMemcpyHostToDevice,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,7,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 7;
       }
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_comput,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,8,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 8;
       }
      }
      bx=1+(tsize-1)/THRDS_ARRAY_ADD; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(ctens0->data_kind){
       case R4:
        gpu_array_add_r4__<<<bx,THRDS_ARRAY_ADD,0,hip_stream>>>(tsize,(float*)(ctens0->elems_d),
                                                                       (float*)(ctens1->elems_d),(float)val);
        break;
       case R8:
        gpu_array_add_r8__<<<bx,THRDS_ARRAY_ADD,0,hip_stream>>>(tsize,(double*)(ctens0->elems_d),
                                                                       (double*)(ctens1->elems_d),val);
        break;
       default:
        i=cuda_task_record(cuda_task,9,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 9;
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_output,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,10,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 10;
       }
      }
      if(copy_back != NO_COPY_BACK){
       err=hipMemcpyAsync(ctens0->elems_h,ctens0->elems_d,tsize*(ctens0->data_kind),hipMemcpyDeviceToHost,hip_stream);
       if(err != hipSuccess){
        i=cuda_task_record(cuda_task,11,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 11;
       }
      }
      if(EVENT_RECORD != 0){
       err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
        i=cuda_task_record(cuda_task,12,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
        err=hipSetDevice(gpu_num); return 12;
       }
      }
      i=cuda_task_record(cuda_task,0,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
      if(i!=0){i=cuda_task_finalize(cuda_task,13,dev_num); err=hipSetDevice(gpu_num); return 13;}
      if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err != hipSuccess){i=cuda_task_finalize(cuda_task,14,dev_num); return 14;}}
     }else{
      i=cuda_task_record(cuda_task,15,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,0,NULL);
      err=hipSetDevice(gpu_num); return 15;
     }
    }else{
     i=cuda_task_finalize(cuda_task,16); return 16;
    }
   }else{
    i=cuda_task_finalize(cuda_task,17); return 17;
   }
  }else{
   i=cuda_task_finalize(cuda_task,18); return 18;
  }
 }else{
  i=cuda_task_finalize(cuda_task,19); return 19;
 }
 return 0;
}
//---------------------------------------------------------------------------------------------------
// TENSOR TRANSPOSE [DLF] (blocking):
__host__ int gpu_tensor_block_copy_dlf(const int *dim_trn, tensBlck_t *tens_in, tensBlck_t *tens_out)
/** tens_out=TRN(tens_in):
INPUT:
 # dim_trn[0,1..dim_num] - index permutation (O2N, sign-containing, numeration starts from 1);
 # tens_in - input tensor block;
OUTPUT:
 # tens_out - output tensor block;
NOTES:
 # If both arguments are scalars or tensors of volume 1, GPU will NOT be involved!
 # In the case of error, CUDA timing events will not be destroyed (when EVENT_RECORD!=0).
**/
{
 int i,j,n,dev_in,dev_out,gpu_num,cae,bx,ibus[MAX_TENSOR_RANK];
 size_t tsize;
 hipError_t err;
 const char *err_msg;
 hipEvent_t time_beg,time_end;
 float time_ms;

 if(tens_in == NULL || tens_out == NULL || dim_trn == NULL) return 1;
 if(tens_in->elems_h == NULL || tens_in->elems_d == NULL || tens_out->elems_h == NULL || tens_out->elems_d == NULL) return 2;
 if(tens_in->rank != tens_out->rank) return 3;
 if(tens_in->data_kind <= 0 || tens_in->data_kind != tens_out->data_kind) return 4;
 err=hipGetLastError(); err=hipSuccess;
 if(tens_in->rank == 0){
//0-rank tensors (scalars):
  switch(tens_in->data_kind){
   case R4:
    ((float*)(tens_out->elems_h))[0]=((float*)(tens_in->elems_h))[0];
    break;
   case R8:
    ((double*)(tens_out->elems_h))[0]=((double*)(tens_in->elems_h))[0];
    break;
   default:
    return 5;
  }
  i=tensBlck_set_absence(tens_out); //invalidate GPU copy of the output scalar
 }else if(tens_in->rank > 0){
//Non-trivial tensors (rank>0):
  n=tens_in->rank;
//DEBUG begin:
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): rank %d, data_kind %d\n",n,tens_in->data_kind);
//for(i=0;i<n;i++) printf(" %d",tens_in->dims_h[i]); printf("\n");
//for(i=0;i<n;i++) printf(" %d",tens_out->dims_h[i]); printf("\n");
//for(i=1;i<=n;i++) printf(" %d",dim_trn[i]); printf("\n");
//DEBUG end.
// Argument check:
  for(i=0;i<n;i++) ibus[i]=0;
  for(i=1;i<=n;i++){j=dim_trn[i]; if(j>=1&&j<=n){if(ibus[j-1]==0){ibus[j-1]=i;}else{return 6;}}else{return 7;}}
  for(i=0;i<n;i++){if(tens_in->dims_h[i] != tens_out->dims_h[dim_trn[1+i]-1]) return 8;}
// Get the tensor block size:
  tsize=tensBlck_volume(tens_in); //tensor block size (elements)
  if(tsize == 1){ //tensor of volume 1
   switch(tens_in->data_kind){
    case R4:
     ((float*)(tens_out->elems_h))[0]=((float*)(tens_in->elems_h))[0];
     break;
    case R8:
     ((double*)(tens_out->elems_h))[0]=((double*)(tens_in->elems_h))[0];
     break;
    default:
     return 9;
   }
   i=tensBlck_set_absence(tens_out); //invalidate GPU copy of the output tensor (volume of 1)
  }else if(tsize > 1){ //tensor of volume > 1
   i=decode_device_id(tens_in->device_id,&dev_in); j=decode_device_id(tens_out->device_id,&dev_out);
   if(dev_in == DEV_NVIDIA_GPU && dev_out == dev_in && i >= 0 && j == i){
    err=hipGetDevice(&gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); return 10;}
    err=hipSetDevice(i); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 11;}
    if(EVENT_RECORD != 0){
     err=hipEventCreate(&time_beg); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 12;}
     err=hipEventCreate(&time_end); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 13;}
    }
// Set up constant memory arguments (tensor block dimension extents, permutation):
    cae=tens_in->const_args_entry; if(cae < 0 || cae >= MAX_GPU_ARGS){err=hipSetDevice(gpu_num); return 14;}
    err=hipMemcpyToSymbol(HIP_SYMBOL(const_args_dims),(void*)((*tens_in).dims_h),sizeof(int)*n,sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice);
    if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 15;}
    err=hipMemcpyToSymbol(HIP_SYMBOL(const_args_prmn),(void*)(&dim_trn[1]),sizeof(int)*n,sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice);
    if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 16;}
//  printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): Constant argument entries: %d %d\n",cae,tens_out->const_args_entry); //debug
    if(tens_in->device_id < 0){ //check whether the input tensor argument is already in GPU memory
// Copy the input tensor block into GPU global memory:
//   printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): HostToDevice copy: %p %p %d\n",tens_in->elems_h,tens_in->elems_d,tsize); //debug
     switch(tens_in->data_kind){
      case R4:
       err=hipMemcpy(tens_in->elems_d,tens_in->elems_h,tsize*sizeof(float),hipMemcpyHostToDevice);
       break;
      case R8:
       err=hipMemcpy(tens_in->elems_d,tens_in->elems_h,tsize*sizeof(double),hipMemcpyHostToDevice);
       break;
      default:
       err=hipSetDevice(gpu_num); return 17;
     }
     if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 18;}
     i=tensBlck_set_presence(tens_in);
    }
// Transpose:
    j=gpu_get_error_count(); time_ms=0.0f;
    if(EVENT_RECORD != 0){err=hipEventRecord(time_beg); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 19;}}
    if(TRANS_SHMEM != 0){
     bx=1+(tsize-1)/THRDS_TENSOR_COPY; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(tens_in->data_kind){
      case R4:
       gpu_tensor_block_copy_dlf_r4__<<<bx,THRDS_TENSOR_COPY>>>(0,0,n,cae,(float*)(tens_in->elems_d),(float*)(tens_out->elems_d)); //shared-memory tensor transpose
       break;
      case R8:
       gpu_tensor_block_copy_dlf_r8__<<<bx,THRDS_TENSOR_COPY>>>(0,0,n,cae,(double*)(tens_in->elems_d),(double*)(tens_out->elems_d)); //shared-memory tensor transpose
       break;
      default:
       err=hipSetDevice(gpu_num); return 20;
     }
    }else{
     bx=1+(tsize-1)/THRDS_TENSOR_COPY_SCAT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(tens_in->data_kind){
      case R4:
       gpu_tensor_block_copy_scatter_dlf_r4__<<<bx,THRDS_TENSOR_COPY_SCAT>>>(0,0,n,cae,(float*)(tens_in->elems_d),(float*)(tens_out->elems_d)); //scattering tensor transpose
       break;
      case R8:
       gpu_tensor_block_copy_scatter_dlf_r8__<<<bx,THRDS_TENSOR_COPY_SCAT>>>(0,0,n,cae,(double*)(tens_in->elems_d),(double*)(tens_out->elems_d)); //scattering tensor transpose
       break;
      default:
       err=hipSetDevice(gpu_num); return 21;
     }
    }
    if(EVENT_RECORD != 0){err=hipEventRecord(time_end); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 22;}}
    err=hipDeviceSynchronize();
    if(err!=hipSuccess){
     err_msg=hipGetErrorString(err);
     if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): Sync error: %s\n",err_msg);
     err=hipSetDevice(gpu_num);
     return 23;
    }
    if(EVENT_RECORD != 0){err=hipEventElapsedTime(&time_ms,time_beg,time_end); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 24;}}
    err=hipGetLastError();
    if(err!=hipSuccess){
     err_msg=hipGetErrorString(err);
     if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): Kernel error: %s\n",err_msg);
     err=hipSetDevice(gpu_num);
     return 25;
    }
    if(gpu_get_error_count() > j){err=hipSetDevice(gpu_num); return 26;}
    if(PRINT_TIMING) printf("#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): Kernel (%d): Time %f: KT/s=%f \n",TRANS_SHMEM,time_ms/1000.0f,((float)(tsize*2))/time_ms);
    i=tensBlck_set_presence(tens_out);
// Copy the output tensor block back into the Host argument buffer:
//  printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): DeviceToHost copy: %p %p %d\n",tens_out->elems_h,tens_out->elems_d,tsize); //debug
    switch(tens_out->data_kind){
     case R4:
      err=hipMemcpy(tens_out->elems_h,tens_out->elems_d,tsize*sizeof(float),hipMemcpyDeviceToHost);
      break;
     case R8:
      err=hipMemcpy(tens_out->elems_h,tens_out->elems_d,tsize*sizeof(double),hipMemcpyDeviceToHost);
      break;
     default:
      err=hipSetDevice(gpu_num); return 27;
    }
    if(err!=hipSuccess){
     err_msg=hipGetErrorString(err);
     if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf): Copy back: %s\n",err_msg);
     err=hipSetDevice(gpu_num);
     return 28;
    }
    err=hipDeviceSynchronize(); if(err!=hipSuccess){err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 29;}
    if(EVENT_RECORD != 0){err=hipEventDestroy(time_beg); err=hipEventDestroy(time_end);} //destroy CUDA events
    err=hipSetDevice(gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); return 30;} //restore old GPU
   }else{
    return 31;
   }
  }else{
   return 32;
  }
 }else{
  return 33;
 }
 return 0;
}
//----------------------------------------------------------------------------------------------------
// TENSOR TRANSPOSE [DLF] (non-blocking):
__host__ int gpu_tensor_block_copy_dlf_(const int *dim_trn, tensBlck_t *tens_in, tensBlck_t *tens_out,
                                        int copy_back, cudaTask_t *cuda_task)
/** tens_out=TRN(tens_in):
INPUT:
 # dim_trn[0,1..dim_num] - index permutation (O2N, sign-containing, numeration starts from 1);
 # tens_in - input tensor block;
 # copy_back - 0: Output will not be copied back to Host (careful!); 1: It will.
OUTPUT:
 # tens_out - output tensor block;
NOTES:
 # For scalar tensors and tensors of volume 1, <copy_back> will always be TRUE.
**/
{
 int i,j,n,dev_in,dev_out,gpu_num,cae,bx,ibus[MAX_TENSOR_RANK];
 size_t tsize;
 hipStream_t hip_stream;
 hipEvent_t time_beg,time_comput,time_output,time_end;
 hipError_t err;
 const char *err_msg;

 if(tens_in == NULL || tens_out == NULL || cuda_task == NULL || dim_trn == NULL){bx=cuda_task_finalize(cuda_task,1); return 1;}
 if(tens_in->elems_h == NULL || tens_in->elems_d == NULL || tens_out->elems_h == NULL || tens_out->elems_d == NULL){bx=cuda_task_finalize(cuda_task,2); return 2;}
 if(tens_in->rank != tens_out->rank){bx=cuda_task_finalize(cuda_task,3); return 3;}
 if(tens_in->data_kind <= 0 || tens_in->data_kind != tens_out->data_kind){bx=cuda_task_finalize(cuda_task,4); return 4;}
 err=hipGetLastError(); err=hipSuccess;
//Trivial 0-rank tensors (scalars):
 if(tens_in->rank == 0){
  i=decode_device_id(tens_in->device_id,&dev_in); j=decode_device_id(tens_out->device_id,&dev_out);
  if(dev_in == DEV_NVIDIA_GPU && dev_out == dev_in && i >= 0 && j == i){
   dev_in=i; err=hipGetDevice(&gpu_num); if(err != hipSuccess){bx=cuda_task_finalize(cuda_task,5); return 5;}
   if(dev_in != gpu_num){err=hipSetDevice(dev_in); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,6); err=hipSetDevice(gpu_num); return 6;}}
   i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
   err=hipEventCreate(&time_beg); if(err != hipSuccess) i++;
   err=hipEventCreate(&time_end); if(err != hipSuccess) i++;
   if(i != 0){
    i=cuda_task_record(cuda_task,7,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
    err=hipSetDevice(gpu_num); return 7;
   }
   if(EVENT_RECORD != 0){
    err=hipEventRecord(time_beg,hip_stream); if(err != hipSuccess){
     i=cuda_task_record(cuda_task,8,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 8;
    }
   }
   switch(tens_in->data_kind){
    case R4:
     ((float*)(tens_out->elems_h))[0]=((float*)(tens_in->elems_h))[0];
     break;
    case R8:
     ((double*)(tens_out->elems_h))[0]=((double*)(tens_in->elems_h))[0];
     break;
    default:
     i=cuda_task_record(cuda_task,9,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 9;
   }
   err=hipMemcpyAsync(tens_out->elems_d,tens_out->elems_h,tens_out->data_kind,hipMemcpyHostToDevice,hip_stream);
   if(err != hipSuccess){
    i=cuda_task_record(cuda_task,10,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
    err=hipSetDevice(gpu_num); return 10;
   }
   if(EVENT_RECORD != 0){
    err=hipEventRecord(time_end,hip_stream); if(err != hipSuccess){
     i=cuda_task_record(cuda_task,11,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 11;
    }
   }
   i=cuda_task_record(cuda_task,0,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
   if(i!=0){bx=cuda_task_finalize(cuda_task,12,dev_in); err=hipSetDevice(gpu_num); return 12;}
   if(dev_in != gpu_num){err=hipSetDevice(gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,13,dev_in); return 13;}}
  }else{
   bx=cuda_task_finalize(cuda_task,14); return 14;
  }
//Non-trivial tensors (rank>0):
 }else if(tens_in->rank > 0){
  n=tens_in->rank;
//DEBUG begin:
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf_): rank %d, data_kind %d\n",n,tens_in->data_kind);
//for(i=0;i<n;i++) printf(" %d",tens_in->dims_h[i]); printf("\n");
//for(i=0;i<n;i++) printf(" %d",tens_out->dims_h[i]); printf("\n");
//for(i=1;i<=n;i++) printf(" %d",dim_trn[i]); printf("\n");
//DEBUG end.
// Argument check:
  for(i=0;i<n;i++) ibus[i]=0;
  for(i=1;i<=n;i++){
   j=dim_trn[i];
   if(j>=1&&j<=n){
    if(ibus[j-1]==0){ibus[j-1]=i;}else{bx=cuda_task_finalize(cuda_task,15); return 15;}
   }else{
    bx=cuda_task_finalize(cuda_task,16); return 16;
   }
  }
  for(i=0;i<n;i++){
   if(tens_in->dims_h[i] != tens_out->dims_h[dim_trn[1+i]-1]){bx=cuda_task_finalize(cuda_task,17); return 17;}
  }
// Get the tensor block size:
  tsize=tensBlck_volume(tens_in); //tensor block size (elements)
  if(tsize == 1){ //tensor of volume 1
   i=decode_device_id(tens_in->device_id,&dev_in); j=decode_device_id(tens_out->device_id,&dev_out);
   if(dev_in == DEV_NVIDIA_GPU && dev_out == dev_in && i >= 0 && j == i){
    dev_in=i; err=hipGetDevice(&gpu_num); if(err != hipSuccess){bx=cuda_task_finalize(cuda_task,18); return 18;}
    if(dev_in != gpu_num){err=hipSetDevice(dev_in); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,19); err=hipSetDevice(gpu_num); return 19;}}
    i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
    err=hipEventCreate(&time_beg); if(err != hipSuccess) i++;
    err=hipEventCreate(&time_end); if(err != hipSuccess) i++;
    if(i != 0){
     i=cuda_task_record(cuda_task,20,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 20;
    }
    if(EVENT_RECORD != 0){
     err=hipEventRecord(time_beg,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,21,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 21;
     }
    }
    switch(tens_in->data_kind){
     case R4:
      ((float*)(tens_out->elems_h))[0]=((float*)(tens_in->elems_h))[0];
      break;
     case R8:
      ((double*)(tens_out->elems_h))[0]=((double*)(tens_in->elems_h))[0];
      break;
     default:
      i=cuda_task_record(cuda_task,22,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 22;
    }
    err=hipMemcpyAsync(tens_out->elems_d,tens_out->elems_h,tens_out->data_kind,hipMemcpyHostToDevice,hip_stream);
    if(err != hipSuccess){
     i=cuda_task_record(cuda_task,23,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 23;
    }
    if(EVENT_RECORD != 0){
     err=hipEventRecord(time_end,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,24,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 24;
     }
    }
    i=cuda_task_record(cuda_task,0,dev_in,hip_stream,time_beg,time_end,time_end,time_end,0,NULL);
    if(i!=0){bx=cuda_task_finalize(cuda_task,25,dev_in); err=hipSetDevice(gpu_num); return 25;}
    if(dev_in != gpu_num){err=hipSetDevice(gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,26,dev_in); return 26;}}
   }else{
    bx=cuda_task_finalize(cuda_task,27); return 27;
   }
  }else if(tsize > 1){ //tensor of volume > 1
   i=decode_device_id(tens_in->device_id,&dev_in); j=decode_device_id(tens_out->device_id,&dev_out);
   if(dev_in == DEV_NVIDIA_GPU && dev_out == dev_in && i >= 0 && j == i){
    dev_in=i; //GPU ID on which this tensor transpose will be executed (where data resides or will reside)
    err=hipGetDevice(&gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,28); return 28;}
    if(dev_in != gpu_num){err=hipSetDevice(dev_in); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,29); err=hipSetDevice(gpu_num); return 29;}}
    i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
    err=hipEventCreate(&time_beg); if(err != hipSuccess) i++;
    err=hipEventCreate(&time_comput); if(err != hipSuccess) i++;
    err=hipEventCreate(&time_output); if(err != hipSuccess) i++;
    err=hipEventCreate(&time_end); if(err != hipSuccess) i++;
    if(i != 0){
     i=cuda_task_record(cuda_task,30,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 30;
    }
    if(EVENT_RECORD != 0){
     err=hipEventRecord(time_beg,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,31,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 31;
     }
    }
// Set up constant memory arguments (tensor block dimension extents, permutation):
    cae=tens_in->const_args_entry;
    if(cae < 0 || cae >= MAX_GPU_ARGS){
     i=cuda_task_record(cuda_task,32,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 32;
    }
    err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_dims),(void*)((*tens_in).dims_h),sizeof(int)*n,sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
    if(err != hipSuccess){
     err_msg=hipGetErrorString(err); i=cuda_task_record(cuda_task,33,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 33;
    }
    err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_prmn),(void*)(&dim_trn[1]),sizeof(int)*n,sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
    if(err != hipSuccess){
     err_msg=hipGetErrorString(err); i=cuda_task_record(cuda_task,34,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 34;
    }
//  printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf_): Constant argument entries: %d %d\n",cae,tens_out->const_args_entry); //debug
    if(tens_in->device_id < 0){ //check whether the input tensor argument is already in GPU memory
// Copy the input tensor block into GPU global memory:
//   printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf_): HostToDevice copy: %p %p %d\n",tens_in->elems_h,tens_in->elems_d,tsize); //debug
     switch(tens_in->data_kind){
      case R4:
       err=hipMemcpyAsync(tens_in->elems_d,tens_in->elems_h,tsize*sizeof(float),hipMemcpyHostToDevice,hip_stream);
       break;
      case R8:
       err=hipMemcpyAsync(tens_in->elems_d,tens_in->elems_h,tsize*sizeof(double),hipMemcpyHostToDevice,hip_stream);
       break;
      default:
       i=cuda_task_record(cuda_task,35,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
       err=hipSetDevice(gpu_num); return 35;
     }
     if(err != hipSuccess){
      err_msg=hipGetErrorString(err); i=cuda_task_record(cuda_task,36,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 36;
     }
    }
    if(EVENT_RECORD != 0){
     err=hipEventRecord(time_comput,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,37,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 37;
     }
    }
// Transpose:
    j=gpu_get_error_count();
    if(TRANS_SHMEM != 0){
     bx=1+(tsize-1)/THRDS_TENSOR_COPY; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(tens_in->data_kind){
      case R4:
       gpu_tensor_block_copy_dlf_r4__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,0,n,cae,(float*)(tens_in->elems_d),(float*)(tens_out->elems_d)); //shared-memory tensor transpose
       break;
      case R8:
       gpu_tensor_block_copy_dlf_r8__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,0,n,cae,(double*)(tens_in->elems_d),(double*)(tens_out->elems_d)); //shared-memory tensor transpose
       break;
      default:
       i=cuda_task_record(cuda_task,38,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
       err=hipSetDevice(gpu_num); return 38;
     }
    }else{
     bx=1+(tsize-1)/THRDS_TENSOR_COPY_SCAT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(tens_in->data_kind){
      case R4:
       gpu_tensor_block_copy_scatter_dlf_r4__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,0,n,cae,(float*)(tens_in->elems_d),(float*)(tens_out->elems_d)); //scattering tensor transpose
       break;
      case R8:
       gpu_tensor_block_copy_scatter_dlf_r8__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,0,n,cae,(double*)(tens_in->elems_d),(double*)(tens_out->elems_d)); //scattering tensor transpose
       break;
      default:
       i=cuda_task_record(cuda_task,39,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
       err=hipSetDevice(gpu_num); return 39;
     }
    }
    if(gpu_get_error_count() > j){
     i=cuda_task_record(cuda_task,40,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
     err=hipSetDevice(gpu_num); return 40;
    }
// Copy the output tensor block back into the Host argument buffer:
    if(EVENT_RECORD != 0){
     err=hipEventRecord(time_output,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,41,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 41;
     }
    }
    if(copy_back != NO_COPY_BACK){
//   printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf_): DeviceToHost copy: %p %p %d\n",tens_out->elems_h,tens_out->elems_d,tsize); //debug
     switch(tens_out->data_kind){
      case R4:
       err=hipMemcpyAsync(tens_out->elems_h,tens_out->elems_d,tsize*sizeof(float),hipMemcpyDeviceToHost,hip_stream);
       break;
      case R8:
       err=hipMemcpyAsync(tens_out->elems_h,tens_out->elems_d,tsize*sizeof(double),hipMemcpyDeviceToHost,hip_stream);
       break;
      default:
       i=cuda_task_record(cuda_task,42,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
       err=hipSetDevice(gpu_num); return 42;
     }
     if(err != hipSuccess){
      err_msg=hipGetErrorString(err);
      if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_tensor_block_copy_dlf_): Copy back: %s\n",err_msg);
      i=cuda_task_record(cuda_task,43,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 43;
     }
    }
    if(EVENT_RECORD != 0){
     err=hipEventRecord(time_end,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,44,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
      err=hipSetDevice(gpu_num); return 44;
     }
    }
    i=cuda_task_record(cuda_task,0,dev_in,hip_stream,time_beg,time_comput,time_output,time_end,0,NULL);
    if(i!=0){bx=cuda_task_finalize(cuda_task,45,dev_in); err=hipSetDevice(gpu_num); return 45;}
    if(dev_in != gpu_num){err=hipSetDevice(gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); bx=cuda_task_finalize(cuda_task,46,dev_in); return 46;}}
   }else{
    bx=cuda_task_finalize(cuda_task,47); return 47;
   }
  }else{
   bx=cuda_task_finalize(cuda_task,48); return 48;
  }
 }else{
  bx=cuda_task_finalize(cuda_task,49); return 49;
 }
 return 0;
}
//-------------------------------------------------------------------------------------------------------------
// TENSOR CONTRACTION (non-blocking):
__host__ int gpu_tensor_block_contract_dlf_(const int *cptrn, const tensBlck_t *ltens, const tensBlck_t *rtens,
                                            tensBlck_t *dtens, int copy_back, cudaTask_t *cuda_task)
/**
dtens(:)+=ltens(:)*rtens(:)
INPUT:
 # cptrn(1:lrank+rrank) - contraction pattern: Position correspondence:
                          Uncontracted indices are positive, contracted are negative;
 # ltens - left tensor argument;
 # rtens - right tensor argument;
 # dtens - destination tensor (initialized!);
 # copy_back - 0: Output will not be copied back to Host (careful!); 1: It will.
OUTPUT:
 # dtens - modified destination tensor;
 # cuda_task - returns the relevant information on the cuda task launched.
NOTES:
 # For all scalar tensors or all tensors of volume 1, <copy_back> will always be TRUE.
**/
{
 int i,j,ncd,nlu,nru,cae,non_triv,gpu_num,dev_num,dev_kind,bx,by;
 int dprm[1+MAX_TENSOR_RANK],lprm[1+MAX_TENSOR_RANK],rprm[1+MAX_TENSOR_RANK]; //the 1st element is the sign of the permutation
 size_t dsize,lsize,rsize,lc,ll,lr;
 int scr_entry_cnt,scr_entries[MAX_SCR_ENTRY_COUNT]; //additional GPU argument buffer entries (three at most)
 void *darg,*larg,*rarg,*alpha,*beta;
 hipStream_t hip_stream;
 hipEvent_t cuda_start,cuda_comput,cuda_output,cuda_finish;
 hipError_t err;
 const char *err_msg;
#ifndef NO_BLAS
 hipblasStatus_t err_cublas;
#endif

 err=hipGetLastError(); err=hipSuccess; scr_entry_cnt=0;
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_contract_dlf_): GPU Tensor Contraction:\n"); //debug
//Check arguments:
 if(cptrn == NULL || dtens == NULL || ltens == NULL || rtens == NULL || cuda_task == NULL) return 1;
 if(ltens->elems_h == NULL || ltens->elems_d == NULL || rtens->elems_h == NULL || rtens->elems_d == NULL ||
    dtens->elems_h == NULL || dtens->elems_d == NULL){i=cuda_task_finalize(cuda_task,2); return 2;}
 if((*dtens).rank < 0 || (*dtens).rank > MAX_TENSOR_RANK ||
    (*ltens).rank < 0 || (*ltens).rank > MAX_TENSOR_RANK ||
    (*rtens).rank < 0 || (*rtens).rank > MAX_TENSOR_RANK){i=cuda_task_finalize(cuda_task,3); return 3;}
 if(!(dtens->data_kind > 0 && ltens->data_kind == dtens->data_kind && rtens->data_kind == dtens->data_kind)){i=cuda_task_finalize(cuda_task,4); return 4;}
//Check contraction pattern and dimension extent correspondence:
 for(i=0;i<dtens->rank;i++) dprm[i]=0; for(i=0;i<ltens->rank;i++) lprm[i]=0; for(i=0;i<rtens->rank;i++) rprm[i]=0;
 for(i=0;i<ltens->rank;i++){
  j=cptrn[i];
  if(j > 0){ //position in dtens
   if(j > dtens->rank){bx=cuda_task_finalize(cuda_task,5); return 5;}
   if(dtens->dims_h[j-1] != ltens->dims_h[i]){bx=cuda_task_finalize(cuda_task,6); return 6;}
   if(dprm[j-1] == 0){dprm[j-1]=1;}else{bx=cuda_task_finalize(cuda_task,7); return 7;}
  }else if(j < 0){ //position in rtens
   if(-j > rtens->rank){bx=cuda_task_finalize(cuda_task,8); return 8;}
   if(rtens->dims_h[-j-1] != ltens->dims_h[i]){bx=cuda_task_finalize(cuda_task,9); return 9;}
   if(cptrn[ltens->rank+(-j-1)] != -(i+1)){bx=cuda_task_finalize(cuda_task,10); return 10;}
   if(rprm[-j-1] == 0){rprm[-j-1]=1;}else{bx=cuda_task_finalize(cuda_task,11); return 11;}
  }else{
   bx=cuda_task_finalize(cuda_task,12); return 12;
  }
 }
 for(i=0;i<rtens->rank;i++){
  j=cptrn[ltens->rank+i];
  if(j > 0){ //position in dtens
   if(j > dtens->rank){bx=cuda_task_finalize(cuda_task,13); return 13;}
   if(dtens->dims_h[j-1] != rtens->dims_h[i]){bx=cuda_task_finalize(cuda_task,14); return 14;}
   if(dprm[j-1] == 0){dprm[j-1]=1;}else{bx=cuda_task_finalize(cuda_task,15); return 15;}
  }else if(j < 0){ //position in ltens
   if(-j > ltens->rank){bx=cuda_task_finalize(cuda_task,16); return 16;}
   if(ltens->dims_h[-j-1] != rtens->dims_h[i]){bx=cuda_task_finalize(cuda_task,17); return 17;}
   if(cptrn[-j-1] != -(i+1)){bx=cuda_task_finalize(cuda_task,18); return 18;}
   if(lprm[-j-1] == 0){lprm[-j-1]=1;}else{bx=cuda_task_finalize(cuda_task,19); return 19;}
  }else{
   bx=cuda_task_finalize(cuda_task,20); return 20;
  }
 }
 for(i=0;i<dtens->rank;i++) if(dprm[i] != 1){bx=cuda_task_finalize(cuda_task,21); return 21;}
//Switch GPUs, if needed:
 dev_num=decode_device_id(dtens->device_id,&dev_kind);
 i=decode_device_id(ltens->device_id,&bx); j=decode_device_id(rtens->device_id,&by);
 if(!(dev_kind == DEV_NVIDIA_GPU && bx == dev_kind && by == dev_kind && dev_num >= 0 && i == dev_num && j == dev_num)){i=cuda_task_finalize(cuda_task,22); return 22;}
 err=hipGetDevice(&gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); i=cuda_task_finalize(cuda_task,23); return 23;}
 if(dev_num != gpu_num){err=hipSetDevice(dev_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); i=cuda_task_finalize(cuda_task,24); err=hipSetDevice(gpu_num); return 24;}}
//Create a CUDA stream and events:
 i=0; err=hipStreamCreate(&hip_stream); if(err != hipSuccess) i++;
 err=hipEventCreate(&cuda_start); if(err != hipSuccess) i++;
 err=hipEventCreate(&cuda_comput); if(err != hipSuccess) i++;
 err=hipEventCreate(&cuda_output); if(err != hipSuccess) i++;
 err=hipEventCreate(&cuda_finish); if(err != hipSuccess) i++;
 if(i != 0){
  i=cuda_task_record(cuda_task,25,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
  err=hipSetDevice(gpu_num); return 25;
 }
//Contraction case: Multiplication of scalars:
 if((*dtens).rank == 0 && (*ltens).rank == 0 && (*rtens).rank == 0){
  if(EVENT_RECORD != 0){
   err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
    i=cuda_task_record(cuda_task,26,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
    err=hipSetDevice(gpu_num); return 26;
   }
   cuda_comput=cuda_start; cuda_output=cuda_start;
  }
  switch(dtens->data_kind){
   case R4: ((float*)(dtens->elems_h))[0]+=(((float*)(ltens->elems_h))[0])*(((float*)(rtens->elems_h))[0]); break;
   case R8: ((double*)(dtens->elems_h))[0]+=(((double*)(ltens->elems_h))[0])*(((double*)(rtens->elems_h))[0]); break;
   default:
    i=cuda_task_record(cuda_task,27,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
    err=hipSetDevice(gpu_num); return 27;
  }
  err=hipMemcpyAsync(dtens->elems_d,dtens->elems_h,dtens->data_kind,hipMemcpyHostToDevice,hip_stream);
  if(err != hipSuccess){
   i=cuda_task_record(cuda_task,28,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
   err=hipSetDevice(gpu_num); return 28;
  }
  if(EVENT_RECORD != 0){
   err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
    i=cuda_task_record(cuda_task,29,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
    err=hipSetDevice(gpu_num); return 29;
   }
  }
//Other contraction cases (tensor rescaling, full contraction, tensor product, partial contraction):
 }else{
// Get contraction permutations for all arguments:
  get_contr_permutations((*ltens).rank,(*rtens).rank,cptrn,dprm,lprm,rprm,&ncd,&nlu,&nru,&i);
  if(i != 0){
   i=cuda_task_record(cuda_task,30,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
   err=hipSetDevice(gpu_num); return 30;
  }
  for(i=0;i<dtens->rank;i++) dtens->prmn_h[i]=dprm[1+i]; //ignore permutaion sign
  for(i=0;i<ltens->rank;i++) ltens->prmn_h[i]=lprm[1+i]; //ignore permutaion sign
  for(i=0;i<rtens->rank;i++) rtens->prmn_h[i]=rprm[1+i]; //ignore permutaion sign
  dsize=tensBlck_volume(dtens); lsize=tensBlck_volume(ltens); rsize=tensBlck_volume(rtens);
  lc=1; ll=1; for(i=0;i<(*ltens).rank;i++){if(ltens->prmn_h[i] <= ncd){lc*=((*ltens).dims_h[i]);}else{ll*=((*ltens).dims_h[i]);}}
  if(lsize > 0 && rsize > 0 && dsize > 0 && dsize%ll == 0){
   lr=dsize/ll;
   if(rsize%lr == 0 && rsize/lr == lc){
/*DEBUG begin:
    printf(" Const args (d,l,r) : %d %d %d\n",dtens->const_args_entry,ltens->const_args_entry,rtens->const_args_entry); //debug
    printf(" Block sizes (d,l,r): %d %d %d\n",dsize,lsize,rsize); //debug
    printf(" Block ranks (d,l,r): %d %d %d\n",dtens->rank,ltens->rank,rtens->rank); //debug
    printf(" Contraction pattern:"); for(i=0;i<ltens->rank+rtens->rank;i++) printf(" %d",cptrn[i]); //debug
    printf("\n Contr/uncontr(l,r) : %d %d %d: %d %d %d\n",ncd,nlu,nru,lc,ll,lr); //debug
    printf(" D-permutation      :"); for(i=0;i<dtens->rank;i++) printf(" %d",dtens->prmn_h[i]); //debug
    printf("\n L-permutation      :"); for(i=0;i<ltens->rank;i++) printf(" %d",ltens->prmn_h[i]); //debug
    printf("\n R-permutation      :"); for(i=0;i<rtens->rank;i++) printf(" %d",rtens->prmn_h[i]); //debug
//DEBUG end.*/
// Record the start event:
    if(EVENT_RECORD != 0){
     err=hipEventRecord(cuda_start,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,31,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err);
      if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_tensor_block_contract_dlf_): Unable to record the start event: %s\n",err_msg);
      err=hipSetDevice(gpu_num); return 31;
     }
     if(LastTask[dev_num] != NULL){ //`This should be done atomically for thread safety
      err=hipStreamWaitEvent(hip_stream,(LastTask[dev_num])->task_comput,0); //input transfers should only begin after the previous task input transfers have completed
      if(err != hipSuccess){
       i=cuda_task_record(cuda_task,76,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err_msg=hipGetErrorString(err);
       if(VERBOSE) printf("\n#ERROR(tensor_algebra_gpu_nvidia:gpu_tensor_block_contract_dlf_): Unable to create a task dependency: %s\n",err_msg);
       err=hipSetDevice(gpu_num); return 76;
      }
     }
    }
// Copy the arguments into GPU memory, unless they are already there:
//  Left tensor argument:
    if(ltens->rank > 0){ //ignore input scalars
     cae=(*ltens).const_args_entry; if(cae < 0 || cae >= MAX_GPU_ARGS){
      i=cuda_task_record(cuda_task,32,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); return 32;
     }
     err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_dims),(void*)(ltens->dims_h),sizeof(int)*((*ltens).rank),sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
     if(err != hipSuccess){
      i=cuda_task_record(cuda_task,33,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 33;
     }
     err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_prmn),(void*)(ltens->prmn_h),sizeof(int)*((*ltens).rank),sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
     if(err != hipSuccess){
      i=cuda_task_record(cuda_task,34,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 34;
     }
     if((*ltens).device_id < 0){ //tensor argument is not in device memory: abs(.device_id) = (1 + active GPU number)
      err=hipMemcpyAsync(ltens->elems_d,ltens->elems_h,lsize*(ltens->data_kind),hipMemcpyHostToDevice,hip_stream);
      if(err != hipSuccess){
       i=cuda_task_record(cuda_task,35,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 35;
      }
     }
    }
//  Right tensor argument:
    if(rtens->rank > 0){ //ignore input scalars
     cae=(*rtens).const_args_entry; if(cae < 0 || cae >= MAX_GPU_ARGS){
      i=cuda_task_record(cuda_task,36,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 36;
     }
     err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_dims),(void*)(rtens->dims_h),sizeof(int)*((*rtens).rank),sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
     if(err != hipSuccess){
      i=cuda_task_record(cuda_task,37,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 37;
     }
     err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_prmn),(void*)(rtens->prmn_h),sizeof(int)*((*rtens).rank),sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
     if(err != hipSuccess){
      i=cuda_task_record(cuda_task,38,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 38;
     }
     if((*rtens).device_id < 0){ //tensor argument is not in device memory: abs(.device_id) = (1 + active GPU number)
      err=hipMemcpyAsync(rtens->elems_d,rtens->elems_h,rsize*(rtens->data_kind),hipMemcpyHostToDevice,hip_stream);
      if(err != hipSuccess){
       i=cuda_task_record(cuda_task,39,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 39;
      }
     }
    }
//  Destination tensor argument (copied in in all cases):
    cae=(*dtens).const_args_entry; if(cae < 0 || cae >= MAX_GPU_ARGS){
     i=cuda_task_record(cuda_task,40,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
     err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 40;
    }
    err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_dims),(void*)(dtens->dims_h),sizeof(int)*((*dtens).rank),sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
    if(err != hipSuccess){
     i=cuda_task_record(cuda_task,41,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
     err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 41;
    }
    err=hipMemcpyToSymbolAsync(HIP_SYMBOL(const_args_prmn),(void*)(dtens->prmn_h),sizeof(int)*((*dtens).rank),sizeof(int)*MAX_TENSOR_RANK*cae,hipMemcpyHostToDevice,hip_stream);
    if(err != hipSuccess){
     i=cuda_task_record(cuda_task,42,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
     err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 42;
    }
    if((*dtens).device_id < 0){ //tensor argument is not in device memory: abs(.device_id) = (1 + active GPU number)
     err=hipMemcpyAsync(dtens->elems_d,dtens->elems_h,dsize*(dtens->data_kind),hipMemcpyHostToDevice,hip_stream);
     if(err != hipSuccess){
      i=cuda_task_record(cuda_task,43,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 43;
     }
    }
// Record the event that all the input data has been copied to device:
    if(EVENT_RECORD != 0){
     err=hipEventRecord(cuda_comput,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,44,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 44;
     }
    }
// Transpose tensor arguments, if needed:
//  Left tensor argument:
    if(non_trivial_prmn((*ltens).rank,ltens->prmn_h) != 0){
     cae=ltens->const_args_entry;
     i=get_buf_entry_gpu(dev_num,lsize*(ltens->data_kind),(char**)&larg,&j);
     if(i != 0){
      j=cuda_task_record(cuda_task,45,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); if(i == TRY_LATER || i == DEVICE_UNABLE){return i;}else{return 45;}
     }
     if(scr_entry_cnt < MAX_SCR_ENTRY_COUNT){
      scr_entries[scr_entry_cnt++]=j;
     }else{
      i=cuda_task_record(cuda_task,46,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); return 46;
     }
     if(TRANS_SHMEM != 0){
      bx=1+(lsize-1)/THRDS_TENSOR_COPY; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(ltens->data_kind){
       case R4:
        gpu_tensor_block_copy_dlf_r4__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,0,(*ltens).rank,cae,(float*)(ltens->elems_d),(float*)larg);
        break;
       case R8:
        gpu_tensor_block_copy_dlf_r8__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,0,(*ltens).rank,cae,(double*)(ltens->elems_d),(double*)larg);
        break;
       default:
        i=cuda_task_record(cuda_task,47,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 47;
      }
     }else{
      bx=1+(lsize-1)/THRDS_TENSOR_COPY_SCAT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(ltens->data_kind){
       case R4:
        gpu_tensor_block_copy_scatter_dlf_r4__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,0,(*ltens).rank,cae,(float*)(ltens->elems_d),(float*)larg);
        break;
       case R8:
        gpu_tensor_block_copy_scatter_dlf_r8__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,0,(*ltens).rank,cae,(double*)(ltens->elems_d),(double*)larg);
        break;
       default:
        i=cuda_task_record(cuda_task,48,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 48;
      }
     }
    }else{
     larg=ltens->elems_d;
    }
//  Right tensor argument:
    if(non_trivial_prmn((*rtens).rank,rtens->prmn_h) != 0){
     cae=rtens->const_args_entry;
     i=get_buf_entry_gpu(dev_num,rsize*(rtens->data_kind),(char**)&rarg,&j);
     if(i != 0){
      j=cuda_task_record(cuda_task,49,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); if(i == TRY_LATER || i == DEVICE_UNABLE){return i;}else{return 49;}
     }
     if(scr_entry_cnt < MAX_SCR_ENTRY_COUNT){
      scr_entries[scr_entry_cnt++]=j;
     }else{
      i=cuda_task_record(cuda_task,50,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); return 50;
     }
     if(TRANS_SHMEM != 0){
      bx=1+(rsize-1)/THRDS_TENSOR_COPY; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(rtens->data_kind){
       case R4:
        gpu_tensor_block_copy_dlf_r4__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,0,(*rtens).rank,cae,(float*)(rtens->elems_d),(float*)rarg);
        break;
       case R8:
        gpu_tensor_block_copy_dlf_r8__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,0,(*rtens).rank,cae,(double*)(rtens->elems_d),(double*)rarg);
        break;
       default:
        i=cuda_task_record(cuda_task,51,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 51;
      }
     }else{
      bx=1+(rsize-1)/THRDS_TENSOR_COPY_SCAT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(rtens->data_kind){
       case R4:
        gpu_tensor_block_copy_scatter_dlf_r4__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,0,(*rtens).rank,cae,(float*)(rtens->elems_d),(float*)rarg);
        break;
       case R8:
        gpu_tensor_block_copy_scatter_dlf_r8__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,0,(*rtens).rank,cae,(double*)(rtens->elems_d),(double*)rarg);
        break;
       default:
        i=cuda_task_record(cuda_task,52,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 52;
      }
     }
    }else{
     rarg=rtens->elems_d;
    }
//  Destination tensor argument:
    non_triv=non_trivial_prmn((*dtens).rank,dtens->prmn_h);
    if(non_triv != 0){
     cae=dtens->const_args_entry;
     i=get_buf_entry_gpu(dev_num,dsize*(dtens->data_kind),(char**)&darg,&j);
     if(i != 0){
      j=cuda_task_record(cuda_task,53,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); if(i == TRY_LATER || i == DEVICE_UNABLE){return i;}else{return 53;}
     }
     if(scr_entry_cnt < MAX_SCR_ENTRY_COUNT){
      scr_entries[scr_entry_cnt++]=j;
     }else{
      i=cuda_task_record(cuda_task,54,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err=hipSetDevice(gpu_num); return 54;
     }
     if(TRANS_SHMEM != 0){
      bx=1+(dsize-1)/THRDS_TENSOR_COPY; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(dtens->data_kind){
       case R4:
        gpu_tensor_block_copy_dlf_r4__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,1,(*dtens).rank,cae,(float*)(dtens->elems_d),(float*)darg);
        break;
       case R8:
        gpu_tensor_block_copy_dlf_r8__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(0,1,(*dtens).rank,cae,(double*)(dtens->elems_d),(double*)darg);
        break;
       default:
        i=cuda_task_record(cuda_task,55,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 55;
      }
     }else{
      bx=1+(dsize-1)/THRDS_TENSOR_COPY_SCAT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(dtens->data_kind){
       case R4:
        gpu_tensor_block_copy_scatter_dlf_r4__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,1,(*dtens).rank,cae,(float*)(dtens->elems_d),(float*)darg);
        break;
       case R8:
        gpu_tensor_block_copy_scatter_dlf_r8__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(0,1,(*dtens).rank,cae,(double*)(dtens->elems_d),(double*)darg);
        break;
       default:
        i=cuda_task_record(cuda_task,56,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 56;
      }
     }
    }else{
     darg=dtens->elems_d;
    }
// Invoke a CUDA kernel corresponding to the current contraction case:
//  Right tensor rescaling:
    if(ltens->rank == 0){ //rtens->rank > 0
     bx=1+(rsize-1)/THRDS_ARRAY_SCALE; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(rtens->data_kind){
      case R4:
       gpu_array_scale_r4__<<<bx,THRDS_ARRAY_SCALE,0,hip_stream>>>(rsize,(float*)(rarg),((float*)(ltens->elems_h))[0]);
       break;
      case R8:
       gpu_array_scale_r8__<<<bx,THRDS_ARRAY_SCALE,0,hip_stream>>>(rsize,(double*)(rarg),((double*)(ltens->elems_h))[0]);
       break;
      default:
       i=cuda_task_record(cuda_task,57,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 57;
     }
//  Left tensor rescaling:
    }else if(rtens->rank == 0){ //ltens->rank > 0
     bx=1+(lsize-1)/THRDS_ARRAY_SCALE; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(ltens->data_kind){
      case R4:
       gpu_array_scale_r4__<<<bx,THRDS_ARRAY_SCALE,0,hip_stream>>>(lsize,(float*)(larg),((float*)(rtens->elems_h))[0]);
       break;
      case R8:
       gpu_array_scale_r8__<<<bx,THRDS_ARRAY_SCALE,0,hip_stream>>>(lsize,(double*)(larg),((double*)(rtens->elems_h))[0]);
       break;
      default:
       i=cuda_task_record(cuda_task,58,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 58;
     }
//  Full tensor contraction (via vector dot-product):
    }else if(dtens->rank == 0){ //ltens->rank > 0 && rtens->rank = ltens->rank && rsize = lsize
     bx=1+(lsize-1)/THRDS_ARRAY_SCALE; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
     switch(ltens->data_kind){
      case R4:
       gpu_array_dot_product_r4__<<<bx,THRDS_ARRAY_SCALE,THRDS_ARRAY_SCALE*sizeof(float),hip_stream>>>
                                 (lsize,(float*)larg,(float*)rarg,(float*)darg);
       break;
      case R8:
       gpu_array_dot_product_r8__<<<bx,THRDS_ARRAY_SCALE,THRDS_ARRAY_SCALE*sizeof(double),hip_stream>>>
                                 (lsize,(double*)larg,(double*)rarg,(double*)darg);
       break;
      default:
       i=cuda_task_record(cuda_task,59,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 59;
     }
//  Tensor product (no contracted indices):
    }else if(dtens->rank == ltens->rank + rtens->rank){
     bx=1+(lsize-1)/THRDS_ARRAY_PRODUCT; by=1+(rsize-1)/THRDS_ARRAY_PRODUCT;
     limit_cuda_blocks2d(MAX_CUDA_BLOCKS,&bx,&by); dim3 blcks(bx,by);
     switch(dtens->data_kind){
      case R4:
       gpu_array_product_r4__<<<blcks,THRDS_ARRAY_PRODUCT,0,hip_stream>>>
                             (lsize,(float*)larg,rsize,(float*)rarg,(float*)darg);
       break;
      case R8:
       gpu_array_product_r8__<<<blcks,THRDS_ARRAY_PRODUCT,0,hip_stream>>>
                             (lsize,(double*)larg,rsize,(double*)rarg,(double*)darg);
       break;
      default:
       i=cuda_task_record(cuda_task,60,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 60;
     }
//  Partial tensor contraction (via TN matrix multiplication):
    }else{
#ifndef NO_BLAS
     if(DISABLE_BLAS == 0){
      i=0;
      switch(dtens->data_kind){
       case R4:
        err=hipGetSymbolAddress(&alpha,HIP_SYMBOL(sgemm_alpha)); if(err != hipSuccess) i++;
        err=hipGetSymbolAddress(&beta,HIP_SYMBOL(sgemm_beta)); if(err != hipSuccess) i++;
        break;
       case R8:
        err=hipGetSymbolAddress(&alpha,HIP_SYMBOL(dgemm_alpha)); if(err != hipSuccess) i++;
        err=hipGetSymbolAddress(&beta,HIP_SYMBOL(dgemm_beta)); if(err != hipSuccess) i++;
        break;
       default:
        i++;
      }
      if(i != 0){
       i=cuda_task_record(cuda_task,61,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 61;
      }
      err_cublas=hipblasSetStream(cublas_handle[dev_num],hip_stream);
      if(err_cublas != HIPBLAS_STATUS_SUCCESS){
       i=cuda_task_record(cuda_task,62,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 62;
      }
      switch(dtens->data_kind){
       case R4:
        err_cublas=hipblasSgemm(cublas_handle[dev_num],HIPBLAS_OP_T,HIPBLAS_OP_N,(int)ll,(int)lr,(int)lc,
                    (float*)alpha,(float*)larg,(int)lc,(float*)rarg,(int)lc,(float*)beta,(float*)darg,(int)ll);
        break;
       case R8:
        err_cublas=hipblasDgemm(cublas_handle[dev_num],HIPBLAS_OP_T,HIPBLAS_OP_N,(int)ll,(int)lr,(int)lc,
                    (double*)alpha,(double*)larg,(int)lc,(double*)rarg,(int)lc,(double*)beta,(double*)darg,(int)ll);
        break;
       default:
        i=cuda_task_record(cuda_task,63,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 63;
      }
      if(err_cublas != HIPBLAS_STATUS_SUCCESS){
       i=cuda_task_record(cuda_task,64,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 64;
      }
     }else{
      bx=1+(ll-1)/MAT_MULT_TILE_DIMX; by=1+(lr-1)/MAT_MULT_TILE_DIMY; limit_cuda_blocks2d(MAX_CUDA_BLOCKS,&bx,&by);
//    printf("\n#DEBUG(): CUDA exec conf: %d %d %d %d\n",bx,by,MAT_MULT_TILE_DIMX,MAT_MULT_TILE_DIMY); //debug
      dim3 blcks(bx,by); dim3 thrds(MAT_MULT_TILE_DIMX,MAT_MULT_TILE_DIMY);
      switch(dtens->data_kind){
       case R4:
        gpu_matrix_multiply_tn_r4__<<<blcks,thrds,0,hip_stream>>>(ll,lr,lc,(float*)larg,(float*)rarg,(float*)darg);
        break;
       case R8:
        gpu_matrix_multiply_tn_r8__<<<blcks,thrds,0,hip_stream>>>(ll,lr,lc,(double*)larg,(double*)rarg,(double*)darg);
        break;
       default:
        i=cuda_task_record(cuda_task,65,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 65;
      }
     }
#else
     bx=1+(ll-1)/MAT_MULT_TILE_DIMX; by=1+(lr-1)/MAT_MULT_TILE_DIMY; limit_cuda_blocks2d(MAX_CUDA_BLOCKS,&bx,&by);
//   printf("\n#DEBUG(): CUDA exec conf: %d %d %d %d\n",bx,by,MAT_MULT_TILE_DIMX,MAT_MULT_TILE_DIMY); //debug
     dim3 blcks(bx,by); dim3 thrds(MAT_MULT_TILE_DIMX,MAT_MULT_TILE_DIMY);
     switch(dtens->data_kind){
      case R4:
       gpu_matrix_multiply_tn_r4__<<<blcks,thrds,0,hip_stream>>>(ll,lr,lc,(float*)larg,(float*)rarg,(float*)darg);
       break;
      case R8:
       gpu_matrix_multiply_tn_r8__<<<blcks,thrds,0,hip_stream>>>(ll,lr,lc,(double*)larg,(double*)rarg,(double*)darg);
       break;
      default:
       i=cuda_task_record(cuda_task,66,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
       err=hipSetDevice(gpu_num); return 66;
     }
#endif
    }
// Transpose back the destination tensor argument:
    if(non_triv != 0){
     cae=dtens->const_args_entry;
     if(TRANS_SHMEM != 0){
      bx=1+(dsize-1)/THRDS_TENSOR_COPY; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(dtens->data_kind){
       case R4:
        gpu_tensor_block_copy_dlf_r4__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(1,0,(*dtens).rank,cae,(float*)darg,(float*)(dtens->elems_d));
        break;
       case R8:
        gpu_tensor_block_copy_dlf_r8__<<<bx,THRDS_TENSOR_COPY,0,hip_stream>>>(1,0,(*dtens).rank,cae,(double*)darg,(double*)(dtens->elems_d));
        break;
       default:
        i=cuda_task_record(cuda_task,67,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 67;
      }
     }else{
      bx=1+(dsize-1)/THRDS_TENSOR_COPY_SCAT; if(bx > MAX_CUDA_BLOCKS) bx=MAX_CUDA_BLOCKS;
      switch(dtens->data_kind){
       case R4:
        gpu_tensor_block_copy_scatter_dlf_r4__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(1,0,(*dtens).rank,cae,(float*)darg,(float*)(dtens->elems_d));
        break;
       case R8:
        gpu_tensor_block_copy_scatter_dlf_r8__<<<bx,THRDS_TENSOR_COPY_SCAT,0,hip_stream>>>(1,0,(*dtens).rank,cae,(double*)darg,(double*)(dtens->elems_d));
        break;
       default:
        i=cuda_task_record(cuda_task,68,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
        err=hipSetDevice(gpu_num); return 68;
      }
     }
    }
// Record the event that the output is ready on device:
    if(EVENT_RECORD != 0){
     err=hipEventRecord(cuda_output,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,69,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 69;
     }
    }
// Copy back the destination tensor argument, if needed:
    if(copy_back != NO_COPY_BACK){
     err=hipMemcpyAsync(dtens->elems_h,dtens->elems_d,dsize*(dtens->data_kind),hipMemcpyDeviceToHost,hip_stream);
     if(err != hipSuccess){
      i=cuda_task_record(cuda_task,70,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 70;
     }
    }
    if(EVENT_RECORD != 0){
     err=hipEventRecord(cuda_finish,hip_stream); if(err != hipSuccess){
      i=cuda_task_record(cuda_task,71,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
      err_msg=hipGetErrorString(err); err=hipSetDevice(gpu_num); return 71;
     }
    }
   }else{
    i=cuda_task_record(cuda_task,72,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
    err=hipSetDevice(gpu_num); return 72;
   }
  }else{
   i=cuda_task_record(cuda_task,73,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
   err=hipSetDevice(gpu_num); return 73;
  }
 }
//Register the CUDA task:
 i=cuda_task_record(cuda_task,0,dev_num,hip_stream,cuda_start,cuda_comput,cuda_output,cuda_finish,scr_entry_cnt,scr_entries);
 if(i!=0){i=cuda_task_finalize(cuda_task,74,dev_num); err=hipSetDevice(gpu_num); return 74;}
 if(EVENT_RECORD != 0) LastTask[dev_num]=cuda_task; //record the last task for each GPU
 if(dev_num != gpu_num){err=hipSetDevice(gpu_num); if(err!=hipSuccess){err_msg=hipGetErrorString(err); i=cuda_task_finalize(cuda_task,75,dev_num); return 75;}}
//printf("\n#DEBUG(tensor_algebra_gpu_nvidia:gpu_tensor_block_contract_dlf_): Scheduled Successfully.\n"); //debug
 return 0;
}
//-------------------------------------------------------------------------------------
//CUDA KERNELS:
// SQUARED 2-NORM OF AN ARRAY (R4):`Obsolete
__global__ void gpu_array_2norm2_r4__(size_t arr_size, const float *arr, float *bnorm2)
/** Computes the squared Euclidean (Frobenius) norm of an array arr(0:arr_size-1)
INPUT:
 # arr_size - size of the array;
 # arr(0:arr_size-1) - array;
OUTPUT:
 # bnorm2[0:gridDim.x-1] - squared 2-norm of a sub-array computed by each CUDA thread block;
**/
{
 size_t i,n;
 float _thread_norm2;
 extern __shared__ float thread_norms2_r4[];

 n=gridDim.x*blockDim.x; _thread_norm2=0.0f;
 for(i=blockIdx.x*blockDim.x+threadIdx.x;i<arr_size;i+=n){_thread_norm2+=arr[i]*arr[i];}
 thread_norms2_r4[threadIdx.x]=_thread_norm2;
 __syncthreads();
 if(threadIdx.x == 0){
  bnorm2[blockIdx.x]=thread_norms2_r4[0]; for(i=1;i<blockDim.x;i++){bnorm2[blockIdx.x]+=thread_norms2_r4[i];}
 }
 __syncthreads();
 return;
}
//---------------------------------------------------------------------------------------
// SQUARED 2-NORM OF AN ARRAY (R8):`Obsolete
__global__ void gpu_array_2norm2_r8__(size_t arr_size, const double *arr, double *bnorm2)
/** Computes the squared Euclidean (Frobenius) norm of an array arr(0:arr_size-1)
INPUT:
 # arr_size - size of the array;
 # arr(0:arr_size-1) - array;
OUTPUT:
 # bnorm2[0:gridDim.x-1] - squared 2-norm of a sub-array computed by each CUDA thread block;
**/
{
 size_t i,n;
 double _thread_norm2;
 extern __shared__ double thread_norms2_r8[];

 n=gridDim.x*blockDim.x; _thread_norm2=0.0;
 for(i=blockIdx.x*blockDim.x+threadIdx.x;i<arr_size;i+=n){_thread_norm2+=arr[i]*arr[i];}
 thread_norms2_r8[threadIdx.x]=_thread_norm2;
 __syncthreads();
 if(threadIdx.x == 0){
  bnorm2[blockIdx.x]=thread_norms2_r8[0]; for(i=1;i<blockDim.x;i++){bnorm2[blockIdx.x]+=thread_norms2_r8[i];}
 }
 __syncthreads();
 return;
}
//------------------------------------------------------------------------------------
// SUM OF THE SQUARES OF ALL ARRAY ELEMENTS (R4):
__global__ void gpu_array_norm2_r8__(size_t arr_size, const float *arr, float *bnorm2)
/** Computes the squared 2-norm of array arr(0:arr_size-1)
INPUT:
 # arr_size - size of the array;
 # arr(0:arr_size-1) - array;
OUTPUT:
 # bnorm2 - squared 2-norm of the array (resides on device as well);
**/
{
 size_t i,n;
 float _thread_norm2;
 extern __shared__ float thread_norms2_r4[];

 n=gridDim.x*blockDim.x; _thread_norm2=0.0f;
 for(i=blockIdx.x*blockDim.x+threadIdx.x;i<arr_size;i+=n){_thread_norm2+=arr[i]*arr[i];}
 thread_norms2_r4[threadIdx.x]=_thread_norm2;
 __syncthreads();
 if(threadIdx.x == 0){
  _thread_norm2=thread_norms2_r4[0]; for(i=1;i<blockDim.x;i++){_thread_norm2+=thread_norms2_r4[i];}
  i=1; while(i == 1){i=atomicMax(&norm2_wr_lock,1);} //waiting for a lock to unlock, then lock
  *bnorm2+=_thread_norm2;
  __threadfence();
  i=atomicExch(&norm2_wr_lock,0); //unlock
 }
 __syncthreads();
 return;
}
//--------------------------------------------------------------------------------------
// SUM OF THE SQUARES OF ALL ARRAY ELEMENTS (R8):
__global__ void gpu_array_norm2_r8__(size_t arr_size, const double *arr, double *bnorm2)
/** Computes the squared 2-norm of array arr(0:arr_size-1)
INPUT:
 # arr_size - size of the array;
 # arr(0:arr_size-1) - array;
OUTPUT:
 # bnorm2 - squared 2-norm of the array (resides on device as well);
**/
{
 size_t i,n;
 double _thread_norm2;
 extern __shared__ double thread_norms2_r8[];

 n=gridDim.x*blockDim.x; _thread_norm2=0.0;
 for(i=blockIdx.x*blockDim.x+threadIdx.x;i<arr_size;i+=n){_thread_norm2+=arr[i]*arr[i];}
 thread_norms2_r8[threadIdx.x]=_thread_norm2;
 __syncthreads();
 if(threadIdx.x == 0){
  _thread_norm2=thread_norms2_r8[0]; for(i=1;i<blockDim.x;i++){_thread_norm2+=thread_norms2_r8[i];}
  i=1; while(i == 1){i=atomicMax(&norm2_wr_lock,1);} //waiting for a lock to unlock, then lock
  *bnorm2+=_thread_norm2;
  __threadfence();
  i=atomicExch(&norm2_wr_lock,0); //unlock
 }
 __syncthreads();
 return;
}
//----------------------------------------------------------------------
// ARRAY INITIALIZATION (R4):
__global__ void gpu_array_init_r4__(size_t tsize, float *arr, float val)
/** arr(:)=val **/
{
 size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
 size_t _gd = gridDim.x*blockDim.x;
 for(size_t l=_ti;l<tsize;l+=_gd){arr[l]=val;}
 return;
}
//------------------------------------------------------------------------
// ARRAY INITIALIZATION (R8):
__global__ void gpu_array_init_r8__(size_t tsize, double *arr, double val)
/** arr(:)=val **/
{
 size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
 size_t _gd = gridDim.x*blockDim.x;
 for(size_t l=_ti;l<tsize;l+=_gd){arr[l]=val;}
 return;
}
//-----------------------------------------------------------------------
// ARRAY RESCALING (R4):
__global__ void gpu_array_scale_r4__(size_t tsize, float *arr, float val)
/** arr(:)*=val **/
{
 size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
 size_t _gd = gridDim.x*blockDim.x;
 for(size_t l=_ti;l<tsize;l+=_gd){arr[l]*=val;}
 return;
}
//-------------------------------------------------------------------------
// ARRAY RESCALING (R8):
__global__ void gpu_array_scale_r8__(size_t tsize, double *arr, double val)
/** arr(:)*=val **/
{
 size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
 size_t _gd = gridDim.x*blockDim.x;
 for(size_t l=_ti;l<tsize;l+=_gd){arr[l]*=val;}
 return;
}
//-------------------------------------------------------------------------------------------------------------------
// ARRAY ADDITION (R4):
__global__ void gpu_array_add_r4__(size_t tsize, float* __restrict__ arr0, const float* __restrict__ arr1, float val)
/** arr0(:)+=arr1(:)*val **/
{
 size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
 size_t _gd = gridDim.x*blockDim.x;
 for(size_t l=_ti;l<tsize;l+=_gd){arr0[l]+=(arr1[l]*val);}
 return;
}
//----------------------------------------------------------------------------------------------------------------------
// ARRAY ADDITION (R8):
__global__ void gpu_array_add_r8__(size_t tsize, double* __restrict__ arr0, const double* __restrict__ arr1, double val)
/** arr0(:)+=arr1(:)*val **/
{
 size_t _ti = blockIdx.x*blockDim.x + threadIdx.x;
 size_t _gd = gridDim.x*blockDim.x;
 for(size_t l=_ti;l<tsize;l+=_gd){arr0[l]+=(arr1[l]*val);}
 return;
}
//-------------------------------------------------------------------------------------------------------------------
// ARRAY DOT-PRODUCT (R4):
__global__ void gpu_array_dot_product_r4__(size_t tsize, const float *arr1, const float *arr2, volatile float *dprod)
{
 extern __shared__ float dprs_r4[]; //volume = blockDim.x
 size_t l;
 int i,j;
 float dpr;
 dpr=0.0f; for(l=blockIdx.x*blockDim.x+threadIdx.x;l<tsize;l+=gridDim.x*blockDim.x){dpr+=arr1[l]*arr2[l];}
 dprs_r4[threadIdx.x]=dpr;
 __syncthreads();
 i=1; while(i < blockDim.x){j=threadIdx.x*(i*2); if(j+i < blockDim.x) dprs_r4[j]+=dprs_r4[j+i]; i*=2;}
 __syncthreads();
 if(threadIdx.x == 0){
  i=1; while(i == 1){i=atomicMax(&dot_product_wr_lock,1);} //waiting for a lock to unlock, then lock
  *dprod+=dprs_r4[0];
  __threadfence();
  i=atomicExch(&dot_product_wr_lock,0); //unlock
 }
 __syncthreads();
 return;
}
//----------------------------------------------------------------------------------------------------------------------
// ARRAY DOT-PRODUCT (R8):
__global__ void gpu_array_dot_product_r8__(size_t tsize, const double *arr1, const double *arr2, volatile double *dprod)
{
 extern __shared__ double dprs_r8[]; //volume = blockDim.x
 size_t l;
 int i,j;
 double dpr;
 dpr=0.0; for(l=blockIdx.x*blockDim.x+threadIdx.x;l<tsize;l+=gridDim.x*blockDim.x){dpr+=arr1[l]*arr2[l];}
 dprs_r8[threadIdx.x]=dpr;
 __syncthreads();
 i=1; while(i < blockDim.x){j=threadIdx.x*(i*2); if(j+i < blockDim.x) dprs_r8[j]+=dprs_r8[j+i]; i*=2;}
 __syncthreads();
 if(threadIdx.x == 0){
  i=1; while(i == 1){i=atomicMax(&dot_product_wr_lock,1);} //waiting for a lock to unlock, then lock
  *dprod+=dprs_r8[0];
  __threadfence();
  i=atomicExch(&dot_product_wr_lock,0); //unlock
 }
 __syncthreads();
 return;
}
//-------------------------------------------------------------------------------------------------------------
// ARRAY PRODUCT (R4):
__global__ void gpu_array_product_r4__(size_t tsize1, const float* __restrict__ arr1, size_t tsize2,
                                                      const float* __restrict__ arr2, float* __restrict__ arr0)
/** arr0[0:tsize2-1][0:tsize1-1]+=arr1[0:tsize1-1]*arr2[0:tsize2-1] **/
{
 __shared__ float lbuf[THRDS_ARRAY_PRODUCT+1],rbuf[THRDS_ARRAY_PRODUCT];
 size_t _ib,_in,_jb,_jn,_tx,_jc;
 _tx=(size_t)threadIdx.x;
// if(tsize1 >= THRDS_ARRAY_PRODUCT){ //large or medium size L
  for(_jb=blockIdx.y*THRDS_ARRAY_PRODUCT;_jb<tsize2;_jb+=gridDim.y*THRDS_ARRAY_PRODUCT){
   if(_jb+THRDS_ARRAY_PRODUCT > tsize2){_jn=tsize2-_jb;}else{_jn=THRDS_ARRAY_PRODUCT;}
   if(_tx < _jn) rbuf[_tx]=arr2[_jb+_tx];
   for(_ib=blockIdx.x*THRDS_ARRAY_PRODUCT;_ib<tsize1;_ib+=gridDim.x*THRDS_ARRAY_PRODUCT){
    if(_ib+THRDS_ARRAY_PRODUCT > tsize1){_in=tsize1-_ib;}else{_in=THRDS_ARRAY_PRODUCT;}
    if(_tx < _in) lbuf[_tx]=arr1[_ib+_tx];
    __syncthreads();
    for(_jc=0;_jc<_jn;_jc++){if(_tx < _in) arr0[(_jb+_jc)*tsize1+_ib+_tx]+=lbuf[_tx]*rbuf[_jc];}
    __syncthreads();
   }
  }
// }else{ //small size L
  //`Write
// }
 return;
}
//---------------------------------------------------------------------------------------------------------------
// ARRAY PRODUCT (R8):
__global__ void gpu_array_product_r8__(size_t tsize1, const double* __restrict__ arr1, size_t tsize2,
                                                      const double* __restrict__ arr2, double* __restrict__ arr0)
/** arr0[0:tsize2-1][0:tsize1-1]+=arr1[0:tsize1-1]*arr2[0:tsize2-1] **/
{
 __shared__ double lbuf[THRDS_ARRAY_PRODUCT+1],rbuf[THRDS_ARRAY_PRODUCT];
 size_t _ib,_in,_jb,_jn,_tx,_jc;
 _tx=(size_t)threadIdx.x;
// if(tsize1 >= THRDS_ARRAY_PRODUCT){ //large or medium size L
  for(_jb=blockIdx.y*THRDS_ARRAY_PRODUCT;_jb<tsize2;_jb+=gridDim.y*THRDS_ARRAY_PRODUCT){
   if(_jb+THRDS_ARRAY_PRODUCT > tsize2){_jn=tsize2-_jb;}else{_jn=THRDS_ARRAY_PRODUCT;}
   if(_tx < _jn) rbuf[_tx]=arr2[_jb+_tx];
   for(_ib=blockIdx.x*THRDS_ARRAY_PRODUCT;_ib<tsize1;_ib+=gridDim.x*THRDS_ARRAY_PRODUCT){
    if(_ib+THRDS_ARRAY_PRODUCT > tsize1){_in=tsize1-_ib;}else{_in=THRDS_ARRAY_PRODUCT;}
    if(_tx < _in) lbuf[_tx]=arr1[_ib+_tx];
    __syncthreads();
    for(_jc=0;_jc<_jn;_jc++){if(_tx < _in) arr0[(_jb+_jc)*tsize1+_ib+_tx]+=lbuf[_tx]*rbuf[_jc];}
    __syncthreads();
   }
  }
// }else{ //small size L
  //`Write
// }
 return;
}
//-------------------------------------------------------------------------------------------------------------
// TENSOR TRANSPOSE (R4) (shared-memory version):
__global__ void gpu_tensor_block_copy_dlf_r4__(int dmo, int drc, int dim_num, int const_args_pos,
                                               const float* __restrict__ tens_in, float* __restrict__ tens_out)
/**
Shared-memory version of tensor transpose: tens_out=TRN(tens_in):
INPUT:
 # dmo - dimension extents order (0: normal, as it is in <const_args>; not 0: permuted dimension order will be imposed);
 # drc - index permutation direction (0: normal, as it is in <const_args>; not 0: inversed permutation will be used);
 # dim_num - tensor block rank;
 # const_args_pos - entry in the __constant__ memory bank where tensor block dimension extents (const_args_dims)
                    and index permutation (const_args_prmn) are stored;
 # tens_in[0:] - input tensor;
OUTPUT:
 # tens_out[0:] - output (transposed) tensor;
NOTES:
 # Minimal CUDA execution configuration is <<<1,warpSize>>>
 # Number of threads per block must be multiple of the warpSize!
**/
{
 __shared__ float buf0[TENS_TRANSP_BUF_SIZE],val;
 __shared__ size_t base_in[MAX_TENSOR_RANK],base_out[MAX_TENSOR_RANK];
 __shared__ size_t ftb[TENS_TRANSP_TAB_SIZE],gtb[TENS_TRANSP_TAB_SIZE];
 __shared__ int htb[TENS_TRANSP_TAB_SIZE],stb[TENS_TRANSP_TAB_SIZE];
 __shared__ int dim_in[MAX_TENSOR_RANK],dim_out[MAX_TENSOR_RANK],o2n[MAX_TENSOR_RANK],n2o[MAX_TENSOR_RANK];
 __shared__ int pri[MAX_TENSOR_RANK],tmp0[MAX_TENSOR_RANK];
 __shared__ int err_code,minor,minor_in,minor_out,s1_ind,s1_ond,s1_step,s1_dim,s2_ind,s2_ond,s2_step,s2_dim,ns1,ns2;
 __shared__ size_t vol,vol_ext;
 size_t _vol,_addr_in,_addr_out,_addr,_work_piece;
 int i,j,k,l,m,n,_vol_minor,_vol_in,_vol_out,_s1,_s2;

//Determine the minor index set (only the master thread in each thread block):
 if(threadIdx.x == 0){
  err_code=0;
  if(dim_num >= 0 && dim_num <= MAX_TENSOR_RANK && blockDim.x >= warpSize && blockDim.x%warpSize == 0){
   s1_ind=dim_num+1; s2_ind=dim_num-1;
   _vol=1; for(i=0;i<dim_num;i++){
    _vol*=const_args_dims[const_args_pos][i]; if(const_args_prmn[const_args_pos][i] != i+1) s1_ind=0;
   }; vol=_vol; //total volume (number of tensor elements)
   if(s1_ind == 0){ //non-trivial permutation
// Set input/output permutations and dimension extents:
    if(drc == 0){ //normal index permutation
     for(i=0;i<dim_num;i++) o2n[i]=const_args_prmn[const_args_pos][i]-1; for(i=0;i<dim_num;i++) n2o[o2n[i]]=i;
    }else{ //inversed index permutation
     for(i=0;i<dim_num;i++) n2o[i]=const_args_prmn[const_args_pos][i]-1; for(i=0;i<dim_num;i++) o2n[n2o[i]]=i;
    }
    if(dmo == 0){ //normal dimension order
     for(i=0;i<dim_num;i++) dim_in[i]=const_args_dims[const_args_pos][i];
     for(i=0;i<dim_num;i++) dim_out[o2n[i]]=dim_in[i];
    }else{ //inversed dimension order
     for(i=0;i<dim_num;i++) dim_out[i]=const_args_dims[const_args_pos][i];
     for(i=0;i<dim_num;i++) dim_in[n2o[i]]=dim_out[i];
    }
    s1_step=dim_in[s1_ind]; s2_step=dim_in[s2_ind];
    if(_vol > TENS_TRANSP_BUF_SIZE){ //tensor block does not fit into the shared memory buffer
// Determine the input/output minor index sets and the combined minor index set:
     l=(int)(sqrt((float)TENS_TRANSP_BUF_SIZE));
     minor_in=0; _vol_in=1; for(i=0;i<dim_num;i++){j=_vol_in*dim_in[i]; if(j>l) break; minor_in++; _vol_in=j;}
     minor_out=0; _vol_out=1; for(i=0;i<dim_num;i++){j=_vol_out*dim_out[i]; if(j>l) break; minor_out++; _vol_out=j;}
     minor=minor_in; _vol_minor=_vol_in; for(i=0;i<minor_out;i++){if(n2o[i]>=minor_in){minor++; _vol_minor*=dim_out[i];}}
     m=1; _s1=0; _s2=0;
     while(_vol_minor < TENS_TRANSP_BUF_SIZE && m != 0){
      m=0;
      if(_s1 == 0){for(i=minor_in;i<dim_num;i++){if(o2n[i]<minor_out){minor_in++; _vol_in*=dim_in[i];}else{break;}}}
      if(_s2 == 0){for(i=minor_out;i<dim_num;i++){if(n2o[i]<minor_in){minor_out++; _vol_out*=dim_out[i];}else{break;}}}
      j=dim_in[minor_in]; l=dim_out[minor_out];
      if(minor_in == n2o[minor_out] && _s1+_s2 == 0){ //same candidate index to both the input and output index sets
       if(j > 1 && TENS_TRANSP_BUF_SIZE < _vol_minor*2) break;
       if(_vol_minor*j > TENS_TRANSP_BUF_SIZE){s1_ind=minor_in; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor; _s1++; _s2++;}
       minor_in++; _vol_in*=j; minor_out++; _vol_out*=j; minor++; _vol_minor*=j; m++;
      }else{ //the input and output index sets consider two different candidates
       if(_vol_minor*j*l <= TENS_TRANSP_BUF_SIZE && _s1+_s2 == 0){ //accept both, no splitting
        minor_in++; _vol_in*=j; minor_out++; _vol_out*=l; minor+=2; _vol_minor*=(j*l); m++;
       }else{ //try to accept either one of the two OR both with splitting
        if(j == 1 || l == 1){
         if(j == 1 && _s1 == 0){minor_in++; minor++; m++;}
         if(l == 1 && _s2 == 0){minor_out++; minor++; m++;}
        }else{
         if(_vol_minor*j <= TENS_TRANSP_BUF_SIZE && _vol_minor*l > TENS_TRANSP_BUF_SIZE &&
            _vol_out >= warpSize && _s1 == 0){ //accept the input index, no splitting
          minor_in++; _vol_in*=j; minor++; _vol_minor*=j; m++;
         }else if(_vol_minor*j > TENS_TRANSP_BUF_SIZE && _vol_minor*l <= TENS_TRANSP_BUF_SIZE &&
                  _vol_in >= warpSize && _s2 == 0){ //accept the output index, no splitting
          minor_out++; _vol_out*=l; minor++; _vol_minor*=l; m++;
         }else{ //splitting is unavoidable (both OR one OR none)
          if(TENS_TRANSP_BUF_SIZE >= _vol_minor*2){
           if(j >= 4 && l >= 4){ //dimension extents are large enough to be split
            if(_vol_minor*4 > TENS_TRANSP_BUF_SIZE){ //impossible to split both indices
             if(_vol_in <= _vol_out && _s1 == 0){ //split the input candidate index
              s1_ind=minor_in; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
              minor_in++; _vol_in*=j; minor++; _vol_minor*=j; _s1++; m++;
             }else{ //split the output candidate index
              if(_s2 == 0){
               s1_ind=n2o[minor_out]; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
               minor_out++; _vol_out*=l; minor++; _vol_minor*=l; _s2++; m++;
              }
             }
            }else{ //possible to split both indices
             i=(int)sqrt(((float)TENS_TRANSP_BUF_SIZE)/(float)_vol_minor); if(i < 2) i=2; //uniform splitting
             s1_step=i; s2_step=i; val=(float)_vol_out/(float)_vol_in;
             if(val < 1.0f){ //scale the initial uniform splitting to reflect the disbalance between _vol_in and _vol_out
              if(val*(float)i < 1.0f) val=1.0f/(float)i; if(val*(float)l < (float)i) val=(float)i/(float)l;
             }else{
              if(val*(float)i > (float)j) val=(float)j/(float)i; if(val > float(i)) val=(float)i;
             }
             s1_step=(int)(((float)i)*val); s2_step=(int)(((float)i)/val);
             if(s1_step >= 2 && _s1 == 0){ //&& s1_step <= dim_in[minor_in]
              s1_ind=minor_in; minor_in++; _vol_in*=j; minor++; _vol_minor*=j; _s1++; m++;
             }else{
              s1_step=dim_in[s1_ind];
             }
             if(s2_step >= 2 && _s2 == 0){ //&& s2_step <= dim_out[minor_out]
              s2_ind=n2o[minor_out]; minor_out++; _vol_out*=l; minor++; _vol_minor*=l; _s2++; m++;
             }else{
              s2_step=dim_in[s2_ind];
             }
            }
           }else if(j >= 4 && l < 4 && _s1 == 0){ //split the input candidate index
            s1_ind=minor_in; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
            minor_in++; _vol_in*=j; minor++; _vol_minor*=j; _s1++; m++;
           }else if(j < 4 && l >= 4 && _s2 == 0){ //split the output candidate index
            s1_ind=n2o[minor_out]; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
            minor_out++; _vol_out*=l; minor++; _vol_minor*=l; _s2++; m++;
           }else{ //both candidate indices have too small extent to be split: try to add one of them fully
            if(_vol_minor*j <= TENS_TRANSP_BUF_SIZE && _s1 == 0){
             minor_in++; _vol_in*=j; minor++; _vol_minor*=j; m++;
            }else if(_vol_minor*l <= TENS_TRANSP_BUF_SIZE && _s2 == 0){
             minor_out++; _vol_out*=l; minor++; _vol_minor*=l; m++;
            }
           }
          }else{ //unable to add more indices in the minor set
           break;
          }
         }
        }
       }
      }
     }
     if(s1_ind == dim_num-1 && s2_ind == dim_num-1){s2_ind=0; s2_step=dim_in[0];} //s1_ind was set while s2_ind was not
    }else{ //tensor block fits into the shared memory buffer from the beginning
     minor=dim_num; minor_in=dim_num; minor_out=dim_num; _vol_minor=_vol; _vol_in=_vol; _vol_out=_vol;
    }
// Share the tensor transpose configuration with other threads in each block:
    vol_ext=_vol/_vol_minor; s1_dim=dim_in[s1_ind]; s2_dim=dim_in[s2_ind];
// Set indexing bases (OUT:{out,in_c,ext_in}_new; IN:{in,out_c,ext_in}_old):
//  OUTPUT indexing (dim_out[], base_out[]: prioritized new numeration):
    for(i=0;i<dim_num;i++){tmp0[i]=dim_out[i];} //save output dimension extents (new numeration)
    j=0; for(i=0;i<minor_out;i++){pri[j++]=i;} //output minor index set (new numeration))
    for(i=0;i<dim_num;i++){if(o2n[i]>=minor_out) pri[j++]=o2n[i];} //{compl.input minor + external} index set (new numeration)
    j=1; for(i=0;i<dim_num;i++){dim_out[i]=j; j*=tmp0[i];} //output bases (new numeration)
    for(i=0;i<dim_num;i++){base_out[i]=dim_out[pri[i]];} //output bases (prioritized new numeration)
    for(i=0;i<dim_num;i++){dim_out[i]=tmp0[pri[i]];} //output extents (prioritized new numeration)
    for(i=0;i<dim_num;i++){if(n2o[pri[i]]==s1_ind){s1_ond=i;}else if(n2o[pri[i]]==s2_ind){s2_ond=i;}} //split indices (prioritized new numeration)
//  INPUT indexing (dim_in[], base_in[]: prioritized old numeration):
    for(i=0;i<dim_num;i++){tmp0[i]=dim_in[i];} //save input dimension extents (old numeration)
    j=0; for(i=0;i<minor_in;i++){pri[j++]=i;} //input minor index set (old numeration)
    for(i=0;i<minor_out;i++){if(n2o[i]>=minor_in) pri[j++]=n2o[i];} //compl.output minor idex set (old numeration)
    for(i=j;i<dim_num;i++){pri[i]=n2o[pri[i]];} //external index set (just convert new numbers to old ones for consistency)
    j=1; for(i=0;i<dim_num;i++){dim_in[i]=j; j*=tmp0[i];} //input bases (old numeration)
    for(i=0;i<dim_num;i++){base_in[i]=dim_in[pri[i]];} //input bases (prioritized old numeration)
    for(i=0;i<dim_num;i++){dim_in[i]=tmp0[pri[i]];} //input extents (prioritized old numeration)
    for(i=0;i<dim_num;i++){if(pri[i]==s1_ind){_s1=i;}else if(pri[i]==s2_ind){_s2=i;}} //split indices (prioritized old numeration)
    s1_ind=_s1; s2_ind=_s2;
    ns1=1+(s1_dim-1)/s1_step; //number of segments from the 1st split minor index
    ns2=1+(s2_dim-1)/s2_step; //number of segments from the 2nd split minor index
//  Index position correspondence for the minor index set (pri-new --> pri-old):
    j=0; for(i=0;i<minor_out;i++){if(n2o[i]<minor_in){pri[i]=n2o[i];}else{pri[i]=(minor_in+j); j++;}}
    j=0; for(i=0;i<minor_in;i++){if(o2n[i]<minor_out){pri[o2n[i]]=i;}else{pri[minor_out+j]=i; j++;}}
// Check tensor transpose configuration parameters:
    if(minor <= 0 || minor_in <= 0 || minor_out <= 0 || _vol <= 0 || _vol_minor <= 0) err_code+=5000; //trap
    if(s1_ind >= dim_num || s2_ind >= dim_num || s1_ond >= dim_num || s2_ond >= dim_num ||
       s1_ind == s2_ind || s1_ond == s2_ond || s1_step <= 0 || s2_step <= 0) err_code+=1000; //trap
    if((s1_step != dim_in[s1_ind] && s1_ind != minor_in-1 && s1_ond != minor_out-1) ||
       (s2_step != dim_in[s2_ind] && s2_ind != minor_in-1 && s2_ond != minor_out-1)) err_code+=500; //trap
    if((_vol_minor*s1_step*s2_step)/(s1_dim*s2_dim) > TENS_TRANSP_BUF_SIZE) err_code+=100; //trap
   } //endif: non-trivial permutation
  }else{
   err_code=1+2*blockDim.x%warpSize;
  }
 } //endif: Master thread.
#ifdef DEBUG_GPU
//DEBUG RECORD begin:
 if(blockIdx.x == 0 && threadIdx.x == 0){
  j=0; gpu_debug_dump[j++]=dim_num;
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_dims[const_args_pos][i];
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_prmn[const_args_pos][i];
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_in[i];
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_out[i];
  gpu_debug_dump[j++]=vol; gpu_debug_dump[j++]=vol_ext; gpu_debug_dump[j++]=vol/vol_ext;
  gpu_debug_dump[j++]=minor; gpu_debug_dump[j++]=minor_in; gpu_debug_dump[j++]=minor_out;
  gpu_debug_dump[j++]=s1_ind; gpu_debug_dump[j++]=s1_ond; gpu_debug_dump[j++]=s1_step; gpu_debug_dump[j++]=s1_dim;
  gpu_debug_dump[j++]=s2_ind; gpu_debug_dump[j++]=s2_ond; gpu_debug_dump[j++]=s2_step; gpu_debug_dump[j++]=s2_dim;
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=pri[i];
  gpu_debug_dump[j++]=err_code; gpu_debug_dump[j++]=-1;
 }
//DEBUG RECORD end.
#endif
 __syncthreads();

//Proceed:
 if(err_code == 0){
  if(s1_ind > dim_num){ //tag of a trivial permutation
// Direct copy:
   _vol=vol; j=gridDim.x*blockDim.x; i=blockIdx.x*blockDim.x+threadIdx.x; _addr_in=_vol-_vol%j;
   for(_addr=0;_addr<_addr_in;_addr+=j){
    _addr_out=_addr+i; tens_out[_addr_out]=tens_in[_addr_out];
   }
   _addr_out=_addr_in+i; if(_addr_out<_vol) tens_out[_addr_out]=tens_in[_addr_out];
  }else{ //non-trivial permutation
   l=threadIdx.x/warpSize; //l: warp number
// Distribute work accross CUDA blocks (external multi-index + splitting):
   for(_work_piece=blockIdx.x;_work_piece<vol_ext*ns1*ns2;_work_piece+=gridDim.x){ //(ns1*ns2*vol_ext) is the total number of independent tasks
    _addr=_work_piece; _addr/=vol_ext; _vol=_work_piece-_addr*vol_ext; _s2=(int)(_addr/ns1); _s1=(int)(_addr-_s2*ns1); //{_addr_ext,_s1,_s2} --> tensor subblock (CUDA block)
//  Modify dimension extents due to possible dimension splitting:
    if(threadIdx.x == 0){
     if(_s1+1 == ns1){ //last segment of the 1st split index
      j=s1_dim-_s1*s1_step; dim_in[s1_ind]=j; dim_out[s1_ond]=j;
     }else{ //internal segment of the 1st split index
      dim_in[s1_ind]=s1_step; dim_out[s1_ond]=s1_step;
     }
     if(_s2+1 == ns2){ //last segment of the 2nd split index
      j=s2_dim-_s2*s2_step; dim_in[s2_ind]=j; dim_out[s2_ond]=j;
     }else{ //internal segment of the 2nd split index
      dim_in[s2_ind]=s2_step; dim_out[s2_ond]=s2_step;
     }
     j=1; for(i=0;i<minor;i++){tmp0[i]=j; j*=dim_in[i];} //minor buffer bases (pri-old)
     for(i=0;i<minor;i++) n2o[i]=tmp0[pri[i]]; //look up table to accelerate further accesses to tmp0[]
    }
    __syncthreads();
//  Mount input/output volumes and bases:
    _vol_in=dim_in[0]; for(i=1;i<minor_in;i++){_vol_in*=dim_in[i];}
    _vol_out=dim_out[0]; for(i=1;i<minor_out;i++){_vol_out*=dim_out[i];}
    _vol_minor=_vol_out; for(i=minor_out;i<minor;i++){_vol_minor*=dim_out[i];}
    _addr_in=(_s1*s1_step)*base_in[s1_ind]+(_s2*s2_step)*base_in[s2_ind]; _addr_out=_vol;
    for(i=minor;i<dim_num;i++){_addr=_vol/dim_in[i]; _addr_in+=(_vol-_addr*dim_in[i])*base_in[i]; _vol=_addr;}
    _vol=_addr_out; _addr_out=(_s1*s1_step)*base_out[s1_ond]+(_s2*s2_step)*base_out[s2_ond];
    for(i=minor;i<dim_num;i++){_addr=_vol/dim_out[i]; _addr_out+=(_vol-_addr*dim_out[i])*base_out[i]; _vol=_addr;}
    if(_vol_out > TENS_TRANSP_TAB_SIZE || _vol_minor > _vol_in*TENS_TRANSP_TAB_SIZE ||
       _vol_minor > _vol_out*TENS_TRANSP_TAB_SIZE){
//  Algorithm 0 (slower):
//   Read the minor volume into the buffer from the input tensor block:
     _vol_minor/=_vol_in; //vol_in_c
     _s1=1+(_vol_in-1)/warpSize; //number of warps (lines) which fully cover the input volume
     _s2=blockDim.x/warpSize; //number of whole warps in a thread block (each warp treats one line)
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      m=j/_s1; _addr=_addr_in; n=m; //n: Input column number (in_c)
      for(i=minor_in;i<minor;i++){k=m/dim_in[i]; _addr+=(m-k*dim_in[i])*base_in[i]; m=k;}
//    m=(j%_s1)*warpSize+threadIdx.x%warpSize; //elemental offset in the input volume
      m=threadIdx.x+(j-n*_s1-l)*warpSize; //elemental offset in the input volume (alternative)
      if(m < _vol_in){buf0[n*_vol_in+m]=tens_in[_addr+m];}
     }
     __syncthreads();
//   Write the minor volume from the buffer into the output tensor block:
     _vol_minor=(_vol_minor*_vol_in)/_vol_out; //vol_out_c
     _s1=1+(_vol_out-1)/warpSize; //number of warps (lines) which fully cover the output volume
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      n=j/_s1; _addr=_addr_out; _vol=n; _vol_in=0; //_vol: Output column number (out_c)
//    for(i=minor_out;i<minor;i++){m=n%dim_out[i]; n/=dim_out[i]; _addr+=m*base_out[i]; _vol_in+=m*tmp0[pri[i]];}
      for(i=minor_out;i<minor;i++){k=n/dim_out[i]; m=n-k*dim_out[i]; n=k; _addr+=m*base_out[i]; _vol_in+=m*n2o[i];}
//    m=(j%_s1)*warpSize+threadIdx.x%warpSize; //elemental offset in the output volume
      m=threadIdx.x+(j-(int)_vol*_s1-l)*warpSize; //elemental offset in the output volume (alternative)
      if(m < _vol_out){
       _addr+=m;
//     for(i=0;i<minor_out;i++){_vol_in+=(m%dim_out[i])*tmp0[pri[i]]; m/=dim_out[i];}
       for(i=0;i<minor_out;i++){k=m/dim_out[i]; _vol_in+=(m-k*dim_out[i])*n2o[i]; m=k;}
       tens_out[_addr]=buf0[_vol_in];
      }
     }
     __syncthreads();
    }else{
//  Algorithm 1 (presumably faster):
//   Create per-block look-up tables:
     m=_vol_minor/_vol_in; //vol_in_c
     for(j=threadIdx.x;j<m;j+=blockDim.x){ //column number (input)
      _addr=0; _s1=j;
//    for(i=minor_in;i<minor;i++){_addr+=(_s1%dim_in[i])*base_in[i]; _s1/=dim_in[i];}
      for(i=minor_in;i<minor;i++){_s2=_s1/dim_in[i]; _addr+=(_s1-_s2*dim_in[i])*base_in[i]; _s1=_s2;}
      ftb[j]=_addr;
     }
     m=_vol_minor/_vol_out; //vol_out_c
     for(j=threadIdx.x;j<m;j+=blockDim.x){ //column number (output)
      _addr=0; _s1=j;
//    for(i=minor_out;i<minor;i++){_addr+=(_s1%dim_out[i])*base_out[i]; _s1/=dim_out[i];}
      for(i=minor_out;i<minor;i++){_s2=_s1/dim_out[i]; _addr+=(_s1-_s2*dim_out[i])*base_out[i]; _s1=_s2;}
      gtb[j]=_addr;
     }
     for(j=threadIdx.x;j<m;j+=blockDim.x){ //column number (output)
      n=0; _s1=j;
//    for(i=minor_out;i<minor;i++){n+=(_s1%dim_out[i])*n2o[i]; _s1/=dim_out[i];}
      for(i=minor_out;i<minor;i++){_s2=_s1/dim_out[i]; n+=(_s1-_s2*dim_out[i])*n2o[i]; _s1=_s2;}
      htb[j]=n;
     }
     for(j=threadIdx.x;j<_vol_out;j+=blockDim.x){
      n=0; _s1=j;
//    for(i=0;i<minor_out;i++){n+=(_s1%dim_out[i])*n2o[i]; _s1/=dim_out[i];}
      for(i=0;i<minor_out;i++){_s2=_s1/dim_out[i]; n+=(_s1-_s2*dim_out[i])*n2o[i]; _s1=_s2;}
      stb[j]=n;
     }
     __syncthreads();
//   Read the minor volume into the buffer from the input tensor block:
     _vol_minor/=_vol_in; //vol_in_c
     _s1=1+(_vol_in-1)/warpSize; //number of warps (lines) which fully cover the input volume
     _s2=blockDim.x/warpSize; //number of whole warps in a thread block (each warp treats one line)
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      m=j/_s1; n=threadIdx.x+(j-m*_s1-l)*warpSize; //m: Input column number (in_c); n: Offset in the column
      if(n < _vol_in){_addr=_addr_in+ftb[m]+n; buf0[m*_vol_in+n]=tens_in[_addr];}
     }
     __syncthreads();
//   Write the minor volume from the buffer into the output tensor block:
     _vol_minor=(_vol_minor*_vol_in)/_vol_out; //vol_out_c
     _s1=1+(_vol_out-1)/warpSize; //number of warps (lines) which fully cover the output volume
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      m=j/_s1; n=threadIdx.x+(j-m*_s1-l)*warpSize; //m: Output column number (out_c); n: Offset in the column
      if(n < _vol_out){_addr=_addr_out+gtb[m]+n; _vol_in=htb[m]+stb[n]; tens_out[_addr]=buf0[_vol_in];}
     }
     __syncthreads();
    }
   } //enddo _work_piece: independent work distribution among thread blocks
  }
 }

//Record errors if occured (for each block):
 if(threadIdx.x == 0){if(err_code != 0) i=atomicAdd(&gpu_error_count,1);}
 return;
}
//---------------------------------------------------------------------------------------------------------------
// TENSOR TRANSPOSE (R8) (shared-memory version):
__global__ void gpu_tensor_block_copy_dlf_r8__(int dmo, int drc, int dim_num, int const_args_pos,
                                               const double* __restrict__ tens_in, double* __restrict__ tens_out)
/**
Shared-memory version of tensor transpose: tens_out=TRN(tens_in):
INPUT:
 # dmo - dimension extents order (0: normal, as it is in <const_args>; not 0: permuted dimension order will be imposed);
 # drc - index permutation direction (0: normal, as it is in <const_args>; not 0: inversed permutation will be used);
 # dim_num - tensor block rank;
 # const_args_pos - entry in the __constant__ memory bank where tensor block dimension extents (const_args_dims)
                    and index permutation (const_args_prmn) are stored;
 # tens_in[0:] - input tensor;
OUTPUT:
 # tens_out[0:] - output (transposed) tensor;
NOTES:
 # Minimal CUDA execution configuration is <<<1,warpSize>>>
 # Number of threads per block must be multiple of the warpSize!
**/
{
 __shared__ double buf0[TENS_TRANSP_BUF_SIZE];
 __shared__ float val;
 __shared__ size_t base_in[MAX_TENSOR_RANK],base_out[MAX_TENSOR_RANK];
 __shared__ size_t ftb[TENS_TRANSP_TAB_SIZE],gtb[TENS_TRANSP_TAB_SIZE];
 __shared__ int htb[TENS_TRANSP_TAB_SIZE],stb[TENS_TRANSP_TAB_SIZE];
 __shared__ int dim_in[MAX_TENSOR_RANK],dim_out[MAX_TENSOR_RANK],o2n[MAX_TENSOR_RANK],n2o[MAX_TENSOR_RANK];
 __shared__ int pri[MAX_TENSOR_RANK],tmp0[MAX_TENSOR_RANK];
 __shared__ int err_code,minor,minor_in,minor_out,s1_ind,s1_ond,s1_step,s1_dim,s2_ind,s2_ond,s2_step,s2_dim,ns1,ns2;
 __shared__ size_t vol,vol_ext;
 size_t _vol,_addr_in,_addr_out,_addr,_work_piece;
 int i,j,k,l,m,n,_vol_minor,_vol_in,_vol_out,_s1,_s2;
/*
SHARED MEMORY USE =
 + TENS_TRANSP_BUF_SIZE*8
 + MAX_TENSOR_RANK*(8+8+4+4+4+4+4+4)
 + TENS_TRANSP_TAB_SIZE*(8+8+4+4)
 + 4*15 + 8*2
REGISTER USE =
 + 4*4 + 4*11 + 8*5 = 100 Bytes
*/

//Determine the minor index set (only the master thread in each thread block):
 if(threadIdx.x == 0){
  err_code=0;
  if(dim_num >= 0 && dim_num <= MAX_TENSOR_RANK && blockDim.x >= warpSize && blockDim.x%warpSize == 0){
   s1_ind=dim_num+1; s2_ind=dim_num-1;
   _vol=1; for(i=0;i<dim_num;i++){
    _vol*=const_args_dims[const_args_pos][i]; if(const_args_prmn[const_args_pos][i] != i+1) s1_ind=0;
   }; vol=_vol; //total volume (number of tensor elements)
   if(s1_ind == 0){ //non-trivial permutation
// Set input/output permutations and dimension extents:
    if(drc == 0){ //normal index permutation
     for(i=0;i<dim_num;i++) o2n[i]=const_args_prmn[const_args_pos][i]-1; for(i=0;i<dim_num;i++) n2o[o2n[i]]=i;
    }else{ //inversed index permutation
     for(i=0;i<dim_num;i++) n2o[i]=const_args_prmn[const_args_pos][i]-1; for(i=0;i<dim_num;i++) o2n[n2o[i]]=i;
    }
    if(dmo == 0){ //normal dimension order
     for(i=0;i<dim_num;i++) dim_in[i]=const_args_dims[const_args_pos][i];
     for(i=0;i<dim_num;i++) dim_out[o2n[i]]=dim_in[i];
    }else{ //inversed dimension order
     for(i=0;i<dim_num;i++) dim_out[i]=const_args_dims[const_args_pos][i];
     for(i=0;i<dim_num;i++) dim_in[n2o[i]]=dim_out[i];
    }
    s1_step=dim_in[s1_ind]; s2_step=dim_in[s2_ind];
    if(_vol > TENS_TRANSP_BUF_SIZE){ //tensor block does not fit into the shared memory buffer
// Determine the input/output minor index sets and the combined minor index set:
     l=(int)(sqrt((float)TENS_TRANSP_BUF_SIZE));
     minor_in=0; _vol_in=1; for(i=0;i<dim_num;i++){j=_vol_in*dim_in[i]; if(j>l) break; minor_in++; _vol_in=j;}
     minor_out=0; _vol_out=1; for(i=0;i<dim_num;i++){j=_vol_out*dim_out[i]; if(j>l) break; minor_out++; _vol_out=j;}
     minor=minor_in; _vol_minor=_vol_in; for(i=0;i<minor_out;i++){if(n2o[i]>=minor_in){minor++; _vol_minor*=dim_out[i];}}
     m=1; _s1=0; _s2=0;
     while(_vol_minor < TENS_TRANSP_BUF_SIZE && m != 0){
      m=0;
      if(_s1 == 0){for(i=minor_in;i<dim_num;i++){if(o2n[i]<minor_out){minor_in++; _vol_in*=dim_in[i];}else{break;}}}
      if(_s2 == 0){for(i=minor_out;i<dim_num;i++){if(n2o[i]<minor_in){minor_out++; _vol_out*=dim_out[i];}else{break;}}}
      j=dim_in[minor_in]; l=dim_out[minor_out];
      if(minor_in == n2o[minor_out] && _s1+_s2 == 0){ //same candidate index to both the input and output index sets
       if(j > 1 && TENS_TRANSP_BUF_SIZE < _vol_minor*2) break;
       if(_vol_minor*j > TENS_TRANSP_BUF_SIZE){s1_ind=minor_in; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor; _s1++; _s2++;}
       minor_in++; _vol_in*=j; minor_out++; _vol_out*=j; minor++; _vol_minor*=j; m++;
      }else{ //the input and output index sets consider two different candidates
       if(_vol_minor*j*l <= TENS_TRANSP_BUF_SIZE && _s1+_s2 == 0){ //accept both, no splitting
        minor_in++; _vol_in*=j; minor_out++; _vol_out*=l; minor+=2; _vol_minor*=(j*l); m++;
       }else{ //try to accept either one of the two OR both with splitting
        if(j == 1 || l == 1){
         if(j == 1 && _s1 == 0){minor_in++; minor++; m++;}
         if(l == 1 && _s2 == 0){minor_out++; minor++; m++;}
        }else{
         if(_vol_minor*j <= TENS_TRANSP_BUF_SIZE && _vol_minor*l > TENS_TRANSP_BUF_SIZE &&
            _vol_out >= warpSize && _s1 == 0){ //accept the input index, no splitting
          minor_in++; _vol_in*=j; minor++; _vol_minor*=j; m++;
         }else if(_vol_minor*j > TENS_TRANSP_BUF_SIZE && _vol_minor*l <= TENS_TRANSP_BUF_SIZE &&
                  _vol_in >= warpSize && _s2 == 0){ //accept the output index, no splitting
          minor_out++; _vol_out*=l; minor++; _vol_minor*=l; m++;
         }else{ //splitting is unavoidable (both OR one OR none)
          if(TENS_TRANSP_BUF_SIZE >= _vol_minor*2){
           if(j >= 4 && l >= 4){ //dimension extents are large enough to be split
            if(_vol_minor*4 > TENS_TRANSP_BUF_SIZE){ //impossible to split both indices
             if(_vol_in <= _vol_out && _s1 == 0){ //split the input candidate index
              s1_ind=minor_in; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
              minor_in++; _vol_in*=j; minor++; _vol_minor*=j; _s1++; m++;
             }else{ //split the output candidate index
              if(_s2 == 0){
               s1_ind=n2o[minor_out]; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
               minor_out++; _vol_out*=l; minor++; _vol_minor*=l; _s2++; m++;
              }
             }
            }else{ //possible to split both indices
             i=(int)sqrt(((float)TENS_TRANSP_BUF_SIZE)/(float)_vol_minor); if(i < 2) i=2; //uniform splitting
             s1_step=i; s2_step=i; val=(float)_vol_out/(float)_vol_in;
             if(val < 1.0f){ //scale the initial uniform splitting to reflect the disbalance between _vol_in and _vol_out
              if(val*(float)i < 1.0f) val=1.0f/(float)i; if(val*(float)l < (float)i) val=(float)i/(float)l;
             }else{
              if(val*(float)i > (float)j) val=(float)j/(float)i; if(val > float(i)) val=(float)i;
             }
             s1_step=(int)(((float)i)*val); s2_step=(int)(((float)i)/val);
             if(s1_step >= 2 && _s1 == 0){ //&& s1_step <= dim_in[minor_in]
              s1_ind=minor_in; minor_in++; _vol_in*=j; minor++; _vol_minor*=j; _s1++; m++;
             }else{
              s1_step=dim_in[s1_ind];
             }
             if(s2_step >= 2 && _s2 == 0){ //&& s2_step <= dim_out[minor_out]
              s2_ind=n2o[minor_out]; minor_out++; _vol_out*=l; minor++; _vol_minor*=l; _s2++; m++;
             }else{
              s2_step=dim_in[s2_ind];
             }
            }
           }else if(j >= 4 && l < 4 && _s1 == 0){ //split the input candidate index
            s1_ind=minor_in; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
            minor_in++; _vol_in*=j; minor++; _vol_minor*=j; _s1++; m++;
           }else if(j < 4 && l >= 4 && _s2 == 0){ //split the output candidate index
            s1_ind=n2o[minor_out]; s1_step=TENS_TRANSP_BUF_SIZE/_vol_minor;
            minor_out++; _vol_out*=l; minor++; _vol_minor*=l; _s2++; m++;
           }else{ //both candidate indices have too small extent to be split: try to add one of them fully
            if(_vol_minor*j <= TENS_TRANSP_BUF_SIZE && _s1 == 0){
             minor_in++; _vol_in*=j; minor++; _vol_minor*=j; m++;
            }else if(_vol_minor*l <= TENS_TRANSP_BUF_SIZE && _s2 == 0){
             minor_out++; _vol_out*=l; minor++; _vol_minor*=l; m++;
            }
           }
          }else{ //unable to add more indices in the minor set
           break;
          }
         }
        }
       }
      }
     }
     if(s1_ind == dim_num-1 && s2_ind == dim_num-1){s2_ind=0; s2_step=dim_in[0];} //s1_ind was set while s2_ind was not
    }else{ //tensor block fits into the shared memory buffer from the beginning
     minor=dim_num; minor_in=dim_num; minor_out=dim_num; _vol_minor=_vol; _vol_in=_vol; _vol_out=_vol;
    }
// Share the tensor transpose configuration with other threads in each block:
    vol_ext=_vol/_vol_minor; s1_dim=dim_in[s1_ind]; s2_dim=dim_in[s2_ind];
// Set indexing bases (OUT:{out,in_c,ext_in}_new; IN:{in,out_c,ext_in}_old):
//  OUTPUT indexing (dim_out[], base_out[]: prioritized new numeration):
    for(i=0;i<dim_num;i++){tmp0[i]=dim_out[i];} //save output dimension extents (new numeration)
    j=0; for(i=0;i<minor_out;i++){pri[j++]=i;} //output minor index set (new numeration))
    for(i=0;i<dim_num;i++){if(o2n[i]>=minor_out) pri[j++]=o2n[i];} //{compl.input minor + external} index set (new numeration)
    j=1; for(i=0;i<dim_num;i++){dim_out[i]=j; j*=tmp0[i];} //output bases (new numeration)
    for(i=0;i<dim_num;i++){base_out[i]=dim_out[pri[i]];} //output bases (prioritized new numeration)
    for(i=0;i<dim_num;i++){dim_out[i]=tmp0[pri[i]];} //output extents (prioritized new numeration)
    for(i=0;i<dim_num;i++){if(n2o[pri[i]]==s1_ind){s1_ond=i;}else if(n2o[pri[i]]==s2_ind){s2_ond=i;}} //split indices (prioritized new numeration)
//  INPUT indexing (dim_in[], base_in[]: prioritized old numeration):
    for(i=0;i<dim_num;i++){tmp0[i]=dim_in[i];} //save input dimension extents (old numeration)
    j=0; for(i=0;i<minor_in;i++){pri[j++]=i;} //input minor index set (old numeration)
    for(i=0;i<minor_out;i++){if(n2o[i]>=minor_in) pri[j++]=n2o[i];} //compl.output minor idex set (old numeration)
    for(i=j;i<dim_num;i++){pri[i]=n2o[pri[i]];} //external index set (just convert new numbers to old ones for consistency)
    j=1; for(i=0;i<dim_num;i++){dim_in[i]=j; j*=tmp0[i];} //input bases (old numeration)
    for(i=0;i<dim_num;i++){base_in[i]=dim_in[pri[i]];} //input bases (prioritized old numeration)
    for(i=0;i<dim_num;i++){dim_in[i]=tmp0[pri[i]];} //input extents (prioritized old numeration)
    for(i=0;i<dim_num;i++){if(pri[i]==s1_ind){_s1=i;}else if(pri[i]==s2_ind){_s2=i;}} //split indices (prioritized old numeration)
    s1_ind=_s1; s2_ind=_s2;
    ns1=1+(s1_dim-1)/s1_step; //number of segments from the 1st split minor index
    ns2=1+(s2_dim-1)/s2_step; //number of segments from the 2nd split minor index
//  Index position correspondence for the minor index set (pri-new --> pri-old):
    j=0; for(i=0;i<minor_out;i++){if(n2o[i]<minor_in){pri[i]=n2o[i];}else{pri[i]=(minor_in+j); j++;}}
    j=0; for(i=0;i<minor_in;i++){if(o2n[i]<minor_out){pri[o2n[i]]=i;}else{pri[minor_out+j]=i; j++;}}
// Check tensor transpose configuration parameters:
    if(minor <= 0 || minor_in <= 0 || minor_out <= 0 || _vol <= 0 || _vol_minor <= 0) err_code+=5000; //trap
    if(s1_ind >= dim_num || s2_ind >= dim_num || s1_ond >= dim_num || s2_ond >= dim_num ||
       s1_ind == s2_ind || s1_ond == s2_ond || s1_step <= 0 || s2_step <= 0) err_code+=1000; //trap
    if((s1_step != dim_in[s1_ind] && s1_ind != minor_in-1 && s1_ond != minor_out-1) ||
       (s2_step != dim_in[s2_ind] && s2_ind != minor_in-1 && s2_ond != minor_out-1)) err_code+=500; //trap
    if((_vol_minor*s1_step*s2_step)/(s1_dim*s2_dim) > TENS_TRANSP_BUF_SIZE) err_code+=100; //trap
   } //endif: non-trivial permutation
  }else{
   err_code=1+2*blockDim.x%warpSize;
  }
 } //endif: Master thread.
#ifdef DEBUG_GPU
//DEBUG RECORD begin:
 if(blockIdx.x == 0 && threadIdx.x == 0){
  j=0; gpu_debug_dump[j++]=dim_num;
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_dims[const_args_pos][i];
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_prmn[const_args_pos][i];
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_in[i];
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_out[i];
  gpu_debug_dump[j++]=vol; gpu_debug_dump[j++]=vol_ext; gpu_debug_dump[j++]=vol/vol_ext;
  gpu_debug_dump[j++]=minor; gpu_debug_dump[j++]=minor_in; gpu_debug_dump[j++]=minor_out;
  gpu_debug_dump[j++]=s1_ind; gpu_debug_dump[j++]=s1_ond; gpu_debug_dump[j++]=s1_step; gpu_debug_dump[j++]=s1_dim;
  gpu_debug_dump[j++]=s2_ind; gpu_debug_dump[j++]=s2_ond; gpu_debug_dump[j++]=s2_step; gpu_debug_dump[j++]=s2_dim;
  for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=pri[i];
  gpu_debug_dump[j++]=err_code; gpu_debug_dump[j++]=-1;
 }
//DEBUG RECORD end.
#endif
 __syncthreads();

//Proceed:
 if(err_code == 0){
  if(s1_ind > dim_num){ //tag of a trivial permutation
// Direct copy:
   _vol=vol; j=gridDim.x*blockDim.x; i=blockIdx.x*blockDim.x+threadIdx.x; _addr_in=_vol-_vol%j;
   for(_addr=0;_addr<_addr_in;_addr+=j){
    _addr_out=_addr+i; tens_out[_addr_out]=tens_in[_addr_out];
   }
   _addr_out=_addr_in+i; if(_addr_out<_vol) tens_out[_addr_out]=tens_in[_addr_out];
  }else{ //non-trivial permutation
   l=threadIdx.x/warpSize; //l: warp number
// Distribute work accross CUDA blocks (external multi-index + splitting):
   for(_work_piece=blockIdx.x;_work_piece<vol_ext*ns1*ns2;_work_piece+=gridDim.x){ //(ns1*ns2*vol_ext) is the total number of independent tasks
    _addr=_work_piece; _addr/=vol_ext; _vol=_work_piece-_addr*vol_ext; _s2=(int)(_addr/ns1); _s1=(int)(_addr-_s2*ns1); //{_addr_ext,_s1,_s2} --> tensor subblock (CUDA block)
//  Modify dimension extents due to possible dimension splitting:
    if(threadIdx.x == 0){
     if(_s1+1 == ns1){ //last segment of the 1st split index
      j=s1_dim-_s1*s1_step; dim_in[s1_ind]=j; dim_out[s1_ond]=j;
     }else{ //internal segment of the 1st split index
      dim_in[s1_ind]=s1_step; dim_out[s1_ond]=s1_step;
     }
     if(_s2+1 == ns2){ //last segment of the 2nd split index
      j=s2_dim-_s2*s2_step; dim_in[s2_ind]=j; dim_out[s2_ond]=j;
     }else{ //internal segment of the 2nd split index
      dim_in[s2_ind]=s2_step; dim_out[s2_ond]=s2_step;
     }
     j=1; for(i=0;i<minor;i++){tmp0[i]=j; j*=dim_in[i];} //minor buffer bases (pri-old)
     for(i=0;i<minor;i++) n2o[i]=tmp0[pri[i]]; //look up table to accelerate further accesses to tmp0[]
    }
    __syncthreads();
//  Mount input/output volumes and bases:
    _vol_in=dim_in[0]; for(i=1;i<minor_in;i++){_vol_in*=dim_in[i];}
    _vol_out=dim_out[0]; for(i=1;i<minor_out;i++){_vol_out*=dim_out[i];}
    _vol_minor=_vol_out; for(i=minor_out;i<minor;i++){_vol_minor*=dim_out[i];}
    _addr_in=(_s1*s1_step)*base_in[s1_ind]+(_s2*s2_step)*base_in[s2_ind]; _addr_out=_vol;
    for(i=minor;i<dim_num;i++){_addr=_vol/dim_in[i]; _addr_in+=(_vol-_addr*dim_in[i])*base_in[i]; _vol=_addr;}
    _vol=_addr_out; _addr_out=(_s1*s1_step)*base_out[s1_ond]+(_s2*s2_step)*base_out[s2_ond];
    for(i=minor;i<dim_num;i++){_addr=_vol/dim_out[i]; _addr_out+=(_vol-_addr*dim_out[i])*base_out[i]; _vol=_addr;}
    if(_vol_out > TENS_TRANSP_TAB_SIZE || _vol_minor > _vol_in*TENS_TRANSP_TAB_SIZE ||
       _vol_minor > _vol_out*TENS_TRANSP_TAB_SIZE){
//  Algorithm 0 (slower):
//   Read the minor volume into the buffer from the input tensor block:
     _vol_minor/=_vol_in; //vol_in_c
     _s1=1+(_vol_in-1)/warpSize; //number of warps (lines) which fully cover the input volume
     _s2=blockDim.x/warpSize; //number of whole warps in a thread block (each warp treats one line)
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      m=j/_s1; _addr=_addr_in; n=m; //n: Input column number (in_c)
      for(i=minor_in;i<minor;i++){k=m/dim_in[i]; _addr+=(m-k*dim_in[i])*base_in[i]; m=k;}
//    m=(j%_s1)*warpSize+threadIdx.x%warpSize; //elemental offset in the input volume
      m=threadIdx.x+(j-n*_s1-l)*warpSize; //elemental offset in the input volume (alternative)
      if(m < _vol_in){buf0[n*_vol_in+m]=tens_in[_addr+m];}
     }
     __syncthreads();
//   Write the minor volume from the buffer into the output tensor block:
     _vol_minor=(_vol_minor*_vol_in)/_vol_out; //vol_out_c
     _s1=1+(_vol_out-1)/warpSize; //number of warps (lines) which fully cover the output volume
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      n=j/_s1; _addr=_addr_out; _vol=n; _vol_in=0; //_vol: Output column number (out_c)
//    for(i=minor_out;i<minor;i++){m=n%dim_out[i]; n/=dim_out[i]; _addr+=m*base_out[i]; _vol_in+=m*tmp0[pri[i]];}
      for(i=minor_out;i<minor;i++){k=n/dim_out[i]; m=n-k*dim_out[i]; n=k; _addr+=m*base_out[i]; _vol_in+=m*n2o[i];}
//    m=(j%_s1)*warpSize+threadIdx.x%warpSize; //elemental offset in the output volume
      m=threadIdx.x+(j-(int)_vol*_s1-l)*warpSize; //elemental offset in the output volume (alternative)
      if(m < _vol_out){
       _addr+=m;
//     for(i=0;i<minor_out;i++){_vol_in+=(m%dim_out[i])*tmp0[pri[i]]; m/=dim_out[i];}
       for(i=0;i<minor_out;i++){k=m/dim_out[i]; _vol_in+=(m-k*dim_out[i])*n2o[i]; m=k;}
       tens_out[_addr]=buf0[_vol_in];
      }
     }
     __syncthreads();
    }else{
//  Algorithm 1 (presumably faster):
//   Create per-block look-up tables:
     m=_vol_minor/_vol_in; //vol_in_c
     for(j=threadIdx.x;j<m;j+=blockDim.x){ //column number (input)
      _addr=0; _s1=j;
//    for(i=minor_in;i<minor;i++){_addr+=(_s1%dim_in[i])*base_in[i]; _s1/=dim_in[i];}
      for(i=minor_in;i<minor;i++){_s2=_s1/dim_in[i]; _addr+=(_s1-_s2*dim_in[i])*base_in[i]; _s1=_s2;}
      ftb[j]=_addr;
     }
     m=_vol_minor/_vol_out; //vol_out_c
     for(j=threadIdx.x;j<m;j+=blockDim.x){ //column number (output)
      _addr=0; _s1=j;
//    for(i=minor_out;i<minor;i++){_addr+=(_s1%dim_out[i])*base_out[i]; _s1/=dim_out[i];}
      for(i=minor_out;i<minor;i++){_s2=_s1/dim_out[i]; _addr+=(_s1-_s2*dim_out[i])*base_out[i]; _s1=_s2;}
      gtb[j]=_addr;
     }
     for(j=threadIdx.x;j<m;j+=blockDim.x){ //column number (output)
      n=0; _s1=j;
//    for(i=minor_out;i<minor;i++){n+=(_s1%dim_out[i])*n2o[i]; _s1/=dim_out[i];}
      for(i=minor_out;i<minor;i++){_s2=_s1/dim_out[i]; n+=(_s1-_s2*dim_out[i])*n2o[i]; _s1=_s2;}
      htb[j]=n;
     }
     for(j=threadIdx.x;j<_vol_out;j+=blockDim.x){
      n=0; _s1=j;
//    for(i=0;i<minor_out;i++){n+=(_s1%dim_out[i])*n2o[i]; _s1/=dim_out[i];}
      for(i=0;i<minor_out;i++){_s2=_s1/dim_out[i]; n+=(_s1-_s2*dim_out[i])*n2o[i]; _s1=_s2;}
      stb[j]=n;
     }
     __syncthreads();
//   Read the minor volume into the buffer from the input tensor block:
     _vol_minor/=_vol_in; //vol_in_c
     _s1=1+(_vol_in-1)/warpSize; //number of warps (lines) which fully cover the input volume
     _s2=blockDim.x/warpSize; //number of whole warps in a thread block (each warp treats one line)
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      m=j/_s1; n=threadIdx.x+(j-m*_s1-l)*warpSize; //m: Input column number (in_c); n: Offset in the column
      if(n < _vol_in){_addr=_addr_in+ftb[m]+n; buf0[m*_vol_in+n]=tens_in[_addr];}
     }
     __syncthreads();
//   Write the minor volume from the buffer into the output tensor block:
     _vol_minor=(_vol_minor*_vol_in)/_vol_out; //vol_out_c
     _s1=1+(_vol_out-1)/warpSize; //number of warps (lines) which fully cover the output volume
     for(j=l;j<_s1*_vol_minor;j+=_s2){ //j: Line number
      m=j/_s1; n=threadIdx.x+(j-m*_s1-l)*warpSize; //m: Output column number (out_c); n: Offset in the column
      if(n < _vol_out){_addr=_addr_out+gtb[m]+n; _vol_in=htb[m]+stb[n]; tens_out[_addr]=buf0[_vol_in];}
     }
     __syncthreads();
    }
   } //enddo _work_piece: independent work distribution among thread blocks
  }
 }

//Record errors if occured (for each block):
 if(threadIdx.x == 0){if(err_code != 0) i=atomicAdd(&gpu_error_count,1);}
 return;
}
//---------------------------------------------------------------------------------------------------------------------
// TENSOR TRANSPOSE (R4) (scatter version):
__global__ void gpu_tensor_block_copy_scatter_dlf_r4__(int dmo, int drc, int dim_num, int const_args_pos,
                                                       const float* __restrict__ tens_in, float* __restrict__ tens_out)
/**
Scattering version of tensor transpose: tens_out=TRN(tens_in):
INPUT:
 # dmo - dimension extents order (0: normal, as it is in <const_args>; not 0: permuted dimension order will be imposed);
 # drc - index permutation direction (0: normal, as it is in <const_args>; not 0: inversed permutation will be used);
 # dim_num - tensor block rank;
 # const_args_pos - entry in the __constant__ memory bank where tensor block dimension extents (const_args_dims)
                    and index permutation (const_args_prmn) are stored;
 # tens_in[0:] - input tensor;
OUTPUT:
 # tens_out[0:] - output (transposed) tensor;
**/
{
 __shared__ int n2o[MAX_TENSOR_RANK];
 __shared__ size_t vol,base_in[MAX_TENSOR_RANK],base_out[MAX_TENSOR_RANK];
 int i,j,k;
 size_t _vol,_addr_in,_addr_out,_si;

 if(dim_num == 0){
  if(blockIdx.x == 0 && threadIdx.x == 0) tens_out[0]=tens_in[0];
 }else if(dim_num == 1){
  _vol=const_args_dims[const_args_pos][0];
  j=blockIdx.x*blockDim.x+threadIdx.x;
  for(_addr_in=j;_addr_in<_vol;_addr_in+=gridDim.x*blockDim.x){tens_out[_addr_in]=tens_in[_addr_in];}
 }else if(dim_num > 1){
  if(threadIdx.x == 0){
   k=0; for(i=0;i<dim_num;i++){j=const_args_prmn[const_args_pos][i]-1; n2o[j]=i; if(j!=i) k=1;}
   if(k == 0){ //trivial permutation
    n2o[0]=dim_num; //trivial permutation flag
    _vol=1; for(i=0;i<dim_num;i++){_vol*=const_args_dims[const_args_pos][i];}; vol=_vol;
   }else{ //non-trivial permutation
    if(dmo == 0){ //normal dimension order
     _vol=1; for(i=0;i<dim_num;i++){base_in[i]=_vol; _vol*=const_args_dims[const_args_pos][i];}; vol=_vol;
     if(drc == 0){ //normal index permutation
      _vol=1; for(i=0;i<dim_num;i++){k=n2o[i]; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][k];}
     }else{ //inversed index permutation
      _vol=1; for(i=0;i<dim_num;i++){
       k=const_args_prmn[const_args_pos][i]-1; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][k];
      }
     }
    }else{ //inversed dimension order
     if(drc == 0){ //normal index permutation
      _vol=1; for(i=0;i<dim_num;i++){
       k=const_args_prmn[const_args_pos][i]-1; base_in[i]=_vol; _vol*=const_args_dims[const_args_pos][k];
      }; vol=_vol;
      _vol=1; for(i=0;i<dim_num;i++){k=n2o[i]; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][i];}
     }else{ //inversed index permutation
      _vol=1; for(i=0;i<dim_num;i++){
       k=n2o[i]; base_in[i]=_vol; _vol*=const_args_dims[const_args_pos][k];
      }; vol=_vol;
      _vol=1; for(i=0;i<dim_num;i++){
       k=const_args_prmn[const_args_pos][i]-1; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][i];
      }
     }
    }
   }
  }
#ifdef DEBUG_GPU
//DEBUG RECORD begin:
  if(blockIdx.x == 0 && threadIdx.x == 0){
   j=0; gpu_debug_dump[j++]=dim_num;
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_dims[const_args_pos][i];
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_prmn[const_args_pos][i];
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_in[i];
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_out[i];
   gpu_debug_dump[j++]=vol; gpu_debug_dump[j++]=-1;
  }
//DEBUG RECORD end.
#endif
  __syncthreads();
  _vol=vol;
  if(n2o[0] >= dim_num){ //trivial permutation
   k=gridDim.x*blockDim.x; j=blockIdx.x*blockDim.x+threadIdx.x;
   for(_addr_in=j;_addr_in<_vol;_addr_in+=k){tens_out[_addr_in]=tens_in[_addr_in];}
  }else{ //non-trivial permutation
   j=blockIdx.x*blockDim.x+threadIdx.x;
   for(_addr_in=j;_addr_in<_vol;_addr_in+=gridDim.x*blockDim.x){
    _addr_out=0; _si=_addr_in; for(i=dim_num-1;i>=0;i--){_addr_out+=(_si/base_in[i])*base_out[i]; _si%=base_in[i];}
    tens_out[_addr_out]=tens_in[_addr_in];
   }
  }
 }else{ //dim_num < 0
  if(threadIdx.x == 0) i=atomicAdd(&gpu_error_count,1); //record an error (for each thread block)
 }
 return;
}
//-----------------------------------------------------------------------------------------------------------------------
// TENSOR TRANSPOSE (R8) (scatter version):
__global__ void gpu_tensor_block_copy_scatter_dlf_r8__(int dmo, int drc, int dim_num, int const_args_pos,
                                                       const double* __restrict__ tens_in, double* __restrict__ tens_out)
/**
Scattering version of tensor transpose: tens_out=TRN(tens_in):
INPUT:
 # dmo - dimension extents order (0: normal, as it is in <const_args>; not 0: permuted dimension order will be imposed);
 # drc - index permutation direction (0: normal, as it is in <const_args>; not 0: inversed permutation will be used);
 # dim_num - tensor block rank;
 # const_args_pos - entry in the __constant__ memory bank where tensor block dimension extents (const_args_dims)
                    and index permutation (const_args_prmn) are stored;
 # tens_in[0:] - input tensor;
OUTPUT:
 # tens_out[0:] - output (transposed) tensor;
**/
{
 __shared__ int n2o[MAX_TENSOR_RANK];
 __shared__ size_t vol,base_in[MAX_TENSOR_RANK],base_out[MAX_TENSOR_RANK];
 int i,j,k;
 size_t _vol,_addr_in,_addr_out,_si;

 if(dim_num == 0){
  if(blockIdx.x == 0 && threadIdx.x == 0) tens_out[0]=tens_in[0];
 }else if(dim_num == 1){
  _vol=const_args_dims[const_args_pos][0];
  j=blockIdx.x*blockDim.x+threadIdx.x;
  for(_addr_in=j;_addr_in<_vol;_addr_in+=gridDim.x*blockDim.x){tens_out[_addr_in]=tens_in[_addr_in];}
 }else if(dim_num > 1){
  if(threadIdx.x == 0){
   k=0; for(i=0;i<dim_num;i++){j=const_args_prmn[const_args_pos][i]-1; n2o[j]=i; if(j!=i) k=1;}
   if(k == 0){ //trivial permutation
    n2o[0]=dim_num; //trivial permutation flag
    _vol=1; for(i=0;i<dim_num;i++){_vol*=const_args_dims[const_args_pos][i];}; vol=_vol;
   }else{ //non-trivial permutation
    if(dmo == 0){ //normal dimension order
     _vol=1; for(i=0;i<dim_num;i++){base_in[i]=_vol; _vol*=const_args_dims[const_args_pos][i];}; vol=_vol;
     if(drc == 0){ //normal index permutation
      _vol=1; for(i=0;i<dim_num;i++){k=n2o[i]; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][k];}
     }else{ //inversed index permutation
      _vol=1; for(i=0;i<dim_num;i++){
       k=const_args_prmn[const_args_pos][i]-1; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][k];
      }
     }
    }else{ //inversed dimension order
     if(drc == 0){ //normal index permutation
      _vol=1; for(i=0;i<dim_num;i++){
       k=const_args_prmn[const_args_pos][i]-1; base_in[i]=_vol; _vol*=const_args_dims[const_args_pos][k];
      }; vol=_vol;
      _vol=1; for(i=0;i<dim_num;i++){k=n2o[i]; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][i];}
     }else{ //inversed index permutation
      _vol=1; for(i=0;i<dim_num;i++){
       k=n2o[i]; base_in[i]=_vol; _vol*=const_args_dims[const_args_pos][k];
      }; vol=_vol;
      _vol=1; for(i=0;i<dim_num;i++){
       k=const_args_prmn[const_args_pos][i]-1; base_out[k]=_vol; _vol*=const_args_dims[const_args_pos][i];
      }
     }
    }
   }
  }
#ifdef DEBUG_GPU
//DEBUG RECORD begin:
  if(blockIdx.x == 0 && threadIdx.x == 0){
   j=0; gpu_debug_dump[j++]=dim_num;
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_dims[const_args_pos][i];
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=const_args_prmn[const_args_pos][i];
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_in[i];
   for(i=0;i<dim_num;i++) gpu_debug_dump[j++]=base_out[i];
   gpu_debug_dump[j++]=vol; gpu_debug_dump[j++]=-1;
  }
//DEBUG RECORD end.
#endif
  __syncthreads();
  _vol=vol;
  if(n2o[0] >= dim_num){ //trivial permutation
   k=gridDim.x*blockDim.x; j=blockIdx.x*blockDim.x+threadIdx.x;
   for(_addr_in=j;_addr_in<_vol;_addr_in+=k){tens_out[_addr_in]=tens_in[_addr_in];}
  }else{ //non-trivial permutation
   j=blockIdx.x*blockDim.x+threadIdx.x;
   for(_addr_in=j;_addr_in<_vol;_addr_in+=gridDim.x*blockDim.x){
    _addr_out=0; _si=_addr_in; for(i=dim_num-1;i>=0;i--){_addr_out+=(_si/base_in[i])*base_out[i]; _si%=base_in[i];}
    tens_out[_addr_out]=tens_in[_addr_in];
   }
  }
 }else{ //dim_num < 0
  if(threadIdx.x == 0) i=atomicAdd(&gpu_error_count,1); //record an error (for each thread block)
 }
 return;
}
//----------------------------------------------------------------------------------------------------------
// MATRIX MULTIPLICATION (R4) (shared-memory version):
__global__ void gpu_matrix_multiply_tn_r4__(size_t ll, size_t lr, size_t lc, const float* __restrict__ arg1,
                                            const float* __restrict__ arg2, float* __restrict__ arg0)
/** arg0(0:ll-1,0:lr-1)+=arg1(0:lc-1,0:ll-1)*arg2(0:lc-1,0:lr-1)
NOTES:
 # Thread block dimensions (.x and .y) must be equal to MAT_MULT_TILE_DIM(X,Y), respectively.
**/
{
 __shared__ float buf1[MAT_MULT_TILE_DIMY+1][MAT_MULT_TILE_DIMY+1],buf2[MAT_MULT_TILE_DIMY+1][MAT_MULT_TILE_DIMY+1];
 size_t k,_col,_row,_col_base,_row_base;
 int i,j,l,m;
 float _val;

 if(lc > 0 && ll > 0 && lr > 0 && blockDim.x == MAT_MULT_TILE_DIMY && blockDim.y == MAT_MULT_TILE_DIMY){
  _val=0.0f; j=threadIdx.y; i=threadIdx.x;
  _col_base=blockIdx.y*MAT_MULT_TILE_DIMY;
  while(_col_base < lr){
   _row_base=blockIdx.x*MAT_MULT_TILE_DIMY;
   while(_row_base < ll){
    for(k=0;k<lc;k+=MAT_MULT_TILE_DIMY){
     _col=_col_base+j; _row=_row_base+j;
// Load two blocks into shared memory:
     if(k+MAT_MULT_TILE_DIMY > lc){
      m=lc-k;
      if(i < m){ //(k+i)<lc
       if(_row < ll){buf1[j][i]=arg1[_row*lc+(k+i)];} // Load a block of the 1st argument into the shared memory
       if(_col < lr){buf2[j][i]=arg2[_col*lc+(k+i)];} // Load a block of the 2nd argument into the shared memory
      }
     }else{
      m=MAT_MULT_TILE_DIMY;
      if(_row < ll){buf1[j][i]=arg1[_row*lc+(k+i)];} // Load a block of the 1st argument into the shared memory
      if(_col < lr){buf2[j][i]=arg2[_col*lc+(k+i)];} // Load a block of the 2nd argument into the shared memory
     }
     __syncthreads();
// Multiply the two blocks:
     _row=_row_base+i;
     if(_col < lr){
      if(_row < ll){
       _col=_col*ll+_row;
       for(l=0;l<m;l++){_val+=buf1[i][l]*buf2[j][l];}
       arg0[_col]+=_val; _val=0.0f;
      }
     }
     __syncthreads();
    }
    _row_base+=gridDim.x*MAT_MULT_TILE_DIMY;
   }
   _col_base+=gridDim.y*MAT_MULT_TILE_DIMY;
  }
 }else{
  if(threadIdx.x == 0 && threadIdx.y == 0) i=atomicAdd(&gpu_error_count,1); //record an error (for each thread block)
 }
 return;
}
//-----------------------------------------------------------------------------------------------------------
// MATRIX MULTIPLICATION (R8) (shared-memory version):
__global__ void gpu_matrix_multiply_tn_r8__(size_t ll, size_t lr, size_t lc, const double* __restrict__ arg1,
                                            const double* __restrict__ arg2, double* __restrict__ arg0)
/** arg0(0:ll-1,0:lr-1)+=arg1(0:lc-1,0:ll-1)*arg2(0:lc-1,0:lr-1)
NOTES:
 # Thread block dimensions (.x and .y) must be equal to MAT_MULT_TILE_DIM(X,Y), respectively.
**/
{
 __shared__ double buf1[MAT_MULT_TILE_DIMX+1][MAT_MULT_TILE_DIMX+1],buf2[MAT_MULT_TILE_DIMY+1][MAT_MULT_TILE_DIMX+1];
 size_t k,_col,_row,_col_base,_row_base;
 int i,j,l,m;
 double _val;

 if(lc > 0 && ll > 0 && lr > 0 && blockDim.x == MAT_MULT_TILE_DIMX && blockDim.y == MAT_MULT_TILE_DIMY){
  _val=0.0; j=threadIdx.y; i=threadIdx.x;
  _col_base=blockIdx.y*MAT_MULT_TILE_DIMY;
  while(_col_base < lr){
   _row_base=blockIdx.x*MAT_MULT_TILE_DIMX;
   while(_row_base < ll){
    for(k=0;k<lc;k+=MAT_MULT_TILE_DIMX){
     _col=_col_base+j; _row=_row_base+j;
// Load two blocks into shared memory:
     if(k+MAT_MULT_TILE_DIMX > lc){m=lc-k;}else{m=MAT_MULT_TILE_DIMX;}
     if(i < m){ //(k+i)<lc
      for(l=0;l<MAT_MULT_TILE_DIMX;l+=MAT_MULT_TILE_DIMY){
       if(_row < ll){buf1[l+j][i]=arg1[_row*lc+(k+i)];} // Load a block of the 1st argument into the shared memory
       _row+=MAT_MULT_TILE_DIMY;
      }
      if(_col < lr){buf2[j][i]=arg2[_col*lc+(k+i)];} // Load a block of the 2nd argument into the shared memory
     }
     __syncthreads();
// Multiply the two blocks:
     _row=_row_base+i;
     if(_col < lr){
      if(_row < ll){
       _col=_col*ll+_row;
       for(l=0;l<m;l++){_val+=buf1[i][l]*buf2[j][l];}
       arg0[_col]+=_val; _val=0.0;
      }
     }
     __syncthreads();
    }
    _row_base+=gridDim.x*MAT_MULT_TILE_DIMX;
   }
   _col_base+=gridDim.y*MAT_MULT_TILE_DIMY;
  }
 }else{
  if(threadIdx.x == 0 && threadIdx.y == 0) i=atomicAdd(&gpu_error_count,1); //record an error (for each thread block)
 }
 return;
}
//-D NO_GPU
#endif
